#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"
float*			hapticDeformationCollisionForce_D; // UNUSED
float*			hapticDeformationInterpolatePositions_D;// UNUSED
float*			hapticDeformationExternForce_D;    // �ش������������洢ÿ���������ɹ���ʩ�ӵĳͷ�����
float*			hapticDeformationExternForceTotal_D; // �ۼӹ���ʩ�ӵ������ϵ�������applyForce��ʱ����ա�
int				hapticCounter_D;// ����һ������֮֡�����˶��ٸ�������֡�ļ�������
float*			hapticDeformationPrePositions_D;	//���ڲ�ֵ����
float*			hapticDeformationPositions_D;	//�������˽�����ײ���ı��������ӣ���Ҫ��ʱͬ��
float*			hapticDeformationNormals_D;     // �������˽�����ײ���������巨��������Ҫ��ʱͬ��
float*			hapticCollisionZone_D;			//��¼������ײ����������������߶Ρ�������������ò�ͬ��Լ������

int* hapticContinuousFrameNumOfCollision_D;     // ��¼�ö��㱻����ʩ��ѹ��������֡����

int				hapticDeformationNum_D;			//����������ȫ�������嶥��������������ڲ������嶥�㡣�����еĻ�����λ�������嶥�㣬�����ӡ�
int				hapticDeformationNumMem_D;



unsigned int*	hapticIsCollide_D;
float*			hapticConstraintForce_D;
float*			hapticConstraintPoints_D;	//�洢���յ���ײ������
float*			hapticConstraintNormals_D;
float*          hapticConstraintZone_D;

float*			hapticCylinderPos_D;
float*			hapticCylinderPhysicalPos_D;
float*			hapticCylinderDir_D;
int*			hapticIndex_D;				//�洢��ײ������������

unsigned int*	hapticQueueIndex_D;
unsigned int*	hapticAuxSumArray_D;
int* haptic_collisionIndex_to_vertIndex_array_D; //��ײ�����±��Ӧ�Ķ����±�

//����1��ʹ�ñ��������ε�������ײ���
int				hapticAABBBoxNum_D;//���������ε�������ÿ�����������ζ���Ӧһ��AABB��Χ�С�
float*			hapticAABBBoxs_D;
float*			hapticTriangleNormal_D;// ���������εķ�����
int*			hapticSurfaceIndex_D; // ���������ζ���������*�����ڲ���*�����嶥���е��±ꡣ

//����2��ʹ������
int				hapticSphereNum_D;
float*			hapticSphereInfo_D;
float*			hapticSphereDirectDirection_D;	//���ָ������
float*			hapticSphereForce_D;	//���յ�����ײ��
unsigned int*	hapticSphereIsCollide_D;
float*			hapticSphereCollisionZone_D;
int*			hapticSphereindex_D;
float*			hapticSphereConstraintPoints_D;
float*          hapticSphereConstraintZone_D;
float*			hapticSphereConstraintDirection_D;  //Լ��ָ������
unsigned int*	hapticSphereTetIndex_D;
float*			hapticSphereTetCoord_D;



//������ײ����
unsigned int*	hapticSphereQueueIndex_D;
unsigned int*	hapticSphereAuxSumArray_D;

int				hapticSphereConstraintNumLeft;
float*			hapticSphereConstraintPosLeft;
float*			hapticSphereConstraintZoneLeft;
float*			hapticSphereConstraintDirectionLeft;

int				hapticSphereConstraintNumRight;
float*			hapticSphereConstraintPosRight;
float*			hapticSphereConstraintZoneRight;
float*			hapticSphereConstraintDirectionRight;
////////////////////////////////////////////////////////
// �����ײ����
unsigned int* hapticPointQueueIndex_D;
unsigned int* hapticPointAuxSumArray_D;

int		hapticPointConstraintNumLeft;
float*	hapticPointConstraintPosLeft;
float* hapticPointConstraintNormalLeft;
float*	hapticPointConstraintZoneLeft;

float* hapticVertexForceOrthogonalToTool_D;
//----------------------------------------
			  
int		hapticPointConstraintNumRight;
float*	hapticPointConstraintPosRight;
float*	hapticPointConstraintZoneRight;
float*	hapticPointConstraintDirectionRight;
///////////////////////////////////////////////////////////

float hapticCollisionStiffness_D;
int MAX_CONTINUOUS_FRAME_COUNT_D;
extern float* triVertCollisionDiag_d;
extern float*  triVertForce_d;

int runUpdateSurfacePointPosition(float dt, int point_num)
{
	int  threadNum = 512;
	int blockNum = (hapticDeformationNum_D + threadNum - 1) / threadNum;

	hapticUpdatePointPosition << <blockNum, threadNum >> > (\
		tetVertMass_d,
		hapticDeformationPositions_D,
		tetVertVelocity_d, 
		hapticDeformationExternForce_D,
		dt,
		point_num);
	hipDeviceSynchronize();
	return 0;
}

//�������˽�����ײ��⣬ͨ����ѹ���洢��ײ��Ϣ������֮��������⹤��λ�ˡ�

int runHapticCollision(float halfLength, float radius) {
	int  threadNum = 512;
	int blockNum = (hapticDeformationNum_D + threadNum - 1) / threadNum;
	//����ײ�������

	hipMemset(hapticIndex_D, -1, sizeof(int));
	
	float obj_r = 0.05f;
	float extended_radius = radius + obj_r;

	//���м�����ײ
	////hapticCalculateCCylinder << <blockNum, threadNum >> >(hapticCylinderPos_D, hapticCylinderDir_D, halfLength, radius, hapticDeformationPositions_D, hapticIsCollide_D, hapticCollisionZone_D, hapticDeformationNum_D, hapticIndex_D);

	//printf("haptic deformationNum:%d\n", hapticDeformationNum_D);
	hapticCollision_MeshCapsule << <blockNum, threadNum >> > (
		hapticCylinderPos_D, hapticCylinderDir_D, halfLength, extended_radius,
		hapticDeformationPositions_D,
		hapticDeformationNormals_D,
		hapticIsCollide_D,
		 triVertForce_d,
		triVertCollisionDiag_d, hapticCollisionStiffness_D,
		hapticCollisionZone_D,
		hapticDeformationNum_D,
		hapticIndex_D);
	//hapticCalculateMeshCylinder <<<blockNum, threadNum >>> (
	//	hapticCylinderPos_D, hapticCylinderPhysicalPos_D,
	//	hapticCylinderDir_D, halfLength, extended_radius, 
	//	hapticDeformationPositions_D, 
	//	hapticDeformationNormals_D,
	//	hapticIsCollide_D, 
	//	hapticDeformationExternForce_D,
	//	hapticCollisionZone_D, 
	//	hapticContinuousFrameNumOfCollision_D, MAX_CONTINUOUS_FRAME_COUNT_D,
	//	hapticDeformationNum_D, 
	//	hapticIndex_D);
	//hapticCalculateContinuousCylinder << <blockNum, threadNum >> > (
	//	hapticCylinderPos_D, hapticCylinderPhysicalPos_D,
	//	hapticCylinderDir_D, halfLength, radius,
	//	hapticDeformationPositions_D,
	//	hapticDeformationNormals_D,
	//	hapticIsCollide_D,
	//	hapticDeformationExternForce_D,
	//	hapticCollisionZone_D,
	//	hapticContinuousFrameNumOfCollision_D, MAX_CONTINUOUS_FRAME_COUNT_D,
	//	hapticDeformationNum_D,
	//	hapticIndex_D);

	//�õ���ײ��֮�󣬼���ǰ׺�͵õ��ڶ����е�����(�����������ǹ����ڴ��С)
	hapticCalculatePrefixSum << <blockNum, threadNum, threadNum *sizeof(unsigned int) >> > (hapticIsCollide_D, hapticQueueIndex_D, hapticAuxSumArray_D, hapticDeformationNum_D);
	//�ٸ�����������д��ײ�㵽������
	//hapticAddCollisionToQueue << <blockNum, threadNum >> > (hapticIsCollide_D, hapticDeformationPositions_D, hapticDeformationNormals_D, hapticCollisionZone_D, hapticConstraintPoints_D, hapticConstraintNormals_D, hapticConstraintZone_D, hapticQueueIndex_D, hapticAuxSumArray_D, hapticDeformationNum_D);
	hapticAddCollisionToQueue_SaveMap << <blockNum, threadNum >> > (
		hapticIsCollide_D, 
		hapticDeformationPositions_D, 
		hapticDeformationNormals_D, 
		hapticCollisionZone_D, 
		hapticConstraintPoints_D, 
		hapticConstraintNormals_D, 
		hapticConstraintZone_D, 
		hapticQueueIndex_D, 
		hapticAuxSumArray_D, 
		hapticDeformationNum_D, 
		haptic_collisionIndex_to_vertIndex_array_D);

	hipDeviceSynchronize();
	
	return 0;
}

//�������˵�������ײ���
int runHapticContinueCollision(float* start,float* end,float halfLength, float radius) {
	int  threadNum = 512;
	int blockNum = (hapticAABBBoxNum_D + threadNum - 1) / threadNum;

	//�߶ε���ײ���
	hapticCalculateContinueCylinder << <blockNum, threadNum >> >(
		start[0], start[1], start[2],
		end[0], end[1], end[2],
		hapticSurfaceIndex_D, hapticDeformationPositions_D, hapticAABBBoxs_D, hapticTriangleNormal_D, hapticAABBBoxNum_D);
	
	return 0;
}


int runHapticCollisionSphere(float halfLength, float radius) {

	int  threadNum = 512;
	int blockNum = (hapticSphereNum_D + threadNum - 1) / threadNum;


	//����ײ�������
	hipMemset(hapticSphereindex_D, -1, sizeof(int));

	//Բ���������ײ���
	hapticCalculateCylinderSphere << <blockNum, threadNum >> >(hapticCylinderPos_D, hapticCylinderDir_D, halfLength, radius, hapticSphereInfo_D,hapticSphereForce_D, hapticSphereIsCollide_D,hapticSphereCollisionZone_D,hapticSphereindex_D, hapticSphereNum_D);

	//ͬ��ʹ��ǰ׺�ͽ���ײ������õ���ײ������
	//�õ���ײ��֮�󣬼���ǰ׺�͵õ��ڶ����е�����(�����������ǹ����ڴ��С)
	hapticCalculatePrefixSum << <blockNum, threadNum, threadNum *sizeof(unsigned int) >> > (hapticSphereIsCollide_D, hapticSphereQueueIndex_D, hapticSphereAuxSumArray_D, hapticSphereNum_D);
	//�ٸ�����������д��ײ�㵽������
	hapticAddSphereCollisionToQueue << <blockNum, threadNum >> > (hapticSphereIsCollide_D, hapticSphereInfo_D, hapticSphereCollisionZone_D, hapticSphereDirectDirection_D, hapticSphereConstraintPoints_D, hapticSphereConstraintZone_D, hapticSphereConstraintDirection_D, hapticSphereQueueIndex_D, hapticSphereAuxSumArray_D, hapticSphereNum_D);

	hipDeviceSynchronize();

	return 0;
}

int runHapticCollisionSphere_Tri(float halfLength, float radius) {
	int threadNum = 512;
	int blockNum = (hapticSphereNum_D + threadNum - 1) / threadNum;

	//����ײ�������
	hipMemset(hapticSphereindex_D, -1, sizeof(int));

	//Բ���������ײ���
	hapticCalculateCylinderSphere_Tri<< <blockNum, threadNum>> >(hapticCylinderPos_D, hapticCylinderDir_D, halfLength, radius, hapticSphereInfo_D, hapticSphereForce_D, hapticSphereIsCollide_D, hapticSphereCollisionZone_D, hapticSphereindex_D, hapticSphereNum_D);

	//ͬ��ʹ��ǰ׺�ͽ���ײ������õ���ײ������
	//�õ���ײ��֮�󣬼���ǰ׺�͵õ��ڶ����е�����(�����������ǹ����ڴ��С)
	hapticCalculatePrefixSum << <blockNum, threadNum, threadNum * sizeof(unsigned int) >> > (hapticSphereIsCollide_D, hapticSphereQueueIndex_D, hapticSphereAuxSumArray_D, hapticSphereNum_D);
	//�ٸ�����������д��ײ�㵽����
	hapticAddSphereCollisionToQueue_Tri << <blockNum, threadNum >> > (hapticSphereIsCollide_D, hapticSphereInfo_D, hapticSphereCollisionZone_D, hapticSphereConstraintPoints_D, hapticSphereConstraintZone_D, hapticSphereQueueIndex_D, hapticSphereAuxSumArray_D, hapticSphereNum_D);

	hipDeviceSynchronize();
	return 0;
}
int runAccumulateExternForce(int point_num)
{
	int  threadNum = 512;
	int blockNum = (hapticDeformationNum_D + threadNum - 1) / threadNum;

	AccumulateExternForce << <blockNum, threadNum >> > (\
		hapticDeformationExternForceTotal_D,
		hapticDeformationExternForce_D,
		point_num);

	hipDeviceSynchronize();

	hapticCounter_D++;
	return 0;
}
// ���������ݻر��ζˣ���Ϊ����ʩ�ӵ�������
int runDispatchForceToTetVertex()
{
	int  threadNum = 512;
	int blockNum = (hapticSphereNum_D + threadNum - 1) / threadNum;

	dispatchForceToTetVertex << <blockNum, threadNum >> > (hapticDeformationExternForce_D, hapticVertexForceOrthogonalToTool_D, hapticIsCollide_D, hapticDeformationNum_D);
	hipDeviceSynchronize();
	return 0;
}
//�����������Ϣ���ݵ������嶥�㣨deprecated��
int runDispatchSphereToTet() {
	int  threadNum = 512;
	int blockNum = (hapticSphereNum_D + threadNum - 1) / threadNum;

	dispatchToTet << <blockNum, threadNum >> >(hapticSphereTetIndex_D,hapticSphereTetCoord_D,hapticDeformationExternForce_D,hapticSphereForce_D, hapticSphereIsCollide_D, hapticSphereNum_D);
	hipDeviceSynchronize();
	return 0;
}

// deprecated
__global__ void hapticCalculateSurfaceCylinder(
	float* cylinderPos, float* cylinderDir, float Length, float radius, 
	float* vertexPositions,
	unsigned int* isCollide, 
	float* zone, 
	int surfaceVertexNum, 
	int* index)// unfinished
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= surfaceVertexNum) return;
	int vertIndex0 = threadid * 3 + 0;
	int vertIndex1 = threadid * 3 + 1;
	int vertIndex2 = threadid * 3 + 2;

	float vert0_x = vertexPositions[vertIndex0 * 3 + 0];
	float vert0_y = vertexPositions[vertIndex0 * 3 + 1];
	float vert0_z = vertexPositions[vertIndex0 * 3 + 2];

	float vert1_x = vertexPositions[vertIndex1 * 3 + 0];
	float vert1_y = vertexPositions[vertIndex1 * 3 + 1];
	float vert1_z = vertexPositions[vertIndex1 * 3 + 2];

	float vert2_x = vertexPositions[vertIndex2 * 3 + 0];
	float vert2_y = vertexPositions[vertIndex2 * 3 + 1];
	float vert2_z = vertexPositions[vertIndex2 * 3 + 2];


}

__global__ void hapticCalculateContinuousCylinder(
	float* cylinderPos,
	float* hapticCylinderPos,
	float* cylinderDir, float halfLength, float radius,
	float* tetPositions,
	float* vertexNormals,
	unsigned int* isCollide,
	float* vertexForce, // �ӹ���ָ����ײ��ķ�����
	float* zone,
	int* continuousFrameCounter, int max_continuous_frame,
	int vertexNum,
	int* index)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;

	float nx = vertexNormals[indexX];
	float ny = vertexNormals[indexY];
	float nz = vertexNormals[indexZ];
	float len_normal = sqrt(nx * nx + ny * ny + nz * nz);
	bool isOnSurface;
	if (len_normal < 0.1)
		isOnSurface = false;
	else
	{
		isOnSurface = true;
		nx /= len_normal;
		ny /= len_normal;
		nz /= len_normal;
	}


	//if (len_normal < 0.1)// ������Ϊ0���õ�Ϊ�����ڲ��Ķ��㣬��������ײ����������ļ��㡣
	//	return;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];
	__shared__ float hapticCylinderTip[3];

	hapticCylinderTip[0] = hapticCylinderPos[0];
	hapticCylinderTip[1] = hapticCylinderPos[1];
	hapticCylinderTip[2] = hapticCylinderPos[2];

	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = tetPositions[indexX] - cylinder0[0];
	float dy = tetPositions[indexY] - cylinder0[1];
	float dz = tetPositions[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength; // t����ײ���ڹ����ϵİٷֱ�λ�ã����Ϊ0��β��Ϊ1

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// ���������ϵ� �Ӵ��������ͶӰ��->������λ��
	// ���Ӵ����ڹ��߸��ϵ�ʱ�򣬸�������ֱ�ڹ��������ߣ��ӹ��������ϵ�ͶӰ��ָ��Ӵ��㡣
	// ���Ӵ����ڹ��߼�˵�ʱ����������ӹ�������ļ��ָ��Ӵ��㡣
	dx = tetPositions[indexX] - cylinder0[0] - t * cylinderd[0];
	dy = tetPositions[indexY] - cylinder0[1] - t * cylinderd[1];
	dz = tetPositions[indexZ] - cylinder0[2] - t * cylinderd[2];

	float sqr_distance = dx * dx + dy * dy + dz * dz;
	float distance = sqrt(sqr_distance);
	dx /= distance; dy /= distance; dz /= distance;
	// ���������⹤�������ϵ�ͶӰ��
	float p0[3] = {
		cylinder0[0] + t * cylinderd[0],
		cylinder0[1] + t * cylinderd[1],
		cylinder0[2] + t * cylinderd[2] };
	// �����������������ϵ�ͶӰ��
	float p1[3] = {
		hapticCylinderTip[0] + t * cylinderd[0],
		hapticCylinderTip[1] + t * cylinderd[1],
		hapticCylinderTip[2] + t * cylinderd[2] };
	// �����⹤���ϵ�ͶӰ��ָ����������ͶӰ�������
	float v[3] = { p1[0] - p0[0], p1[1] - p0[1] ,p1[2] - p0[2] };
	float gh_distance = sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);

	const float GH_DISTANCE_THREASHOLD = 0.25;
	if (gh_distance < GH_DISTANCE_THREASHOLD)// ֱ����ͼ��λ������ײ���
	{
		if (distance < radius)//�������Ľ����Ҫд����ײ�������ڼ������⹤��λ��
		{
			if (isOnSurface)
			{
				// ���㹤���ڶ�����ʩ�ӵ�����������Ϊ[-nx, -ny, -nz](���涥�㷨�����ķ�����)
				atomicAdd(vertexForce + threadid * 3 + 0, -nx * (radius - distance));
				atomicAdd(vertexForce + threadid * 3 + 1, -ny * (radius - distance));
				atomicAdd(vertexForce + threadid * 3 + 2, -nz * (radius - distance));
			}
			else
			{
				float fx = dx * (radius - distance);
				float fy = dy * (radius - distance);
				float fz = dz * (radius - distance);
				vertexForce[indexX] += fx;
				vertexForce[indexY] += fy;
				vertexForce[indexZ] += fz;
				//printf("inner point collision:[%f %f %f] len: %f\n", fx, fy, fz, radius-distance);
			}

			isCollide[threadid] = 1;
			zone[threadid] = t;
			//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
			atomicAdd(index, 1);

			//// printf("���⹤�߰뾶��Χ�ڷ�����ײ�� threadid:%d counter: %d\n", threadid, continuousFrameCounter[threadid]);
			// ���߶Զ���ʩ����������ǰ�������ײ����֡����+1
			if (continuousFrameCounter[threadid] < max_continuous_frame)
			{
				continuousFrameCounter[threadid] += 1;
			}
		}
		else
		{
			// δʩ��������ǰ�������ײ����֡����-1
			if (continuousFrameCounter[threadid] > 0)
			{
				continuousFrameCounter[threadid] -= 1;
			}
		}
	}
	else
	{
		// moveDirָ���ǡ�������λ�˶��뵽���⹤��λ�˵��ƶ�������
		float moveDir[3] = { -hapticCylinderPos[0] + cylinderPos[0],
			-hapticCylinderPos[1] + cylinderPos[1],
		-hapticCylinderPos[2] + cylinderPos[2] };
		float moveDistance = sqrt(moveDir[0] * moveDir[0] + moveDir[1] * moveDir[1] + moveDir[2] * moveDir[2]);
		float point[3] = { tetPositions[indexX], tetPositions[indexY], tetPositions[indexZ] };
		float collisionNormal[3];
		float collisionPos[3];
		
		float k = 1;
		float middlePos[3] = { hapticCylinderPos[0] * k + cylinderPos[0] * (1 - k),
								hapticCylinderPos[1] * k + cylinderPos[1] * (1 - k),
								hapticCylinderPos[2] * k + cylinderPos[2] * (1 - k) };

		bool collided = hapticCylinderCollisionContinue(halfLength, radius,
			middlePos, cylinderPos, cylinderDir, point,
			collisionNormal, collisionPos);
		if (collided)
		{
			float fx = collisionPos[0]-point[0];
			float fy = collisionPos[1]-point[1];
			float fz = collisionPos[2]-point[2];
			float f_len = sqrt(fx * fx + fy * fy + fz * fz);
			vertexForce[indexX] += fx;
			vertexForce[indexY] += fy;
			vertexForce[indexZ] += fz;
			//printf("continuous: collisionPos:[%f %f %f] point[%f %f %f]\n", collisionPos[0], collisionPos[1], collisionPos[2], point[0], point[1], point[2]);
			isCollide[threadid] = 1;
			zone[threadid] = 1;
			//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
			atomicAdd(index, 1);

			// ���߶Զ���ʩ����������ǰ�������ײ����֡����+1
			if (continuousFrameCounter[threadid] < max_continuous_frame)
			{
				continuousFrameCounter[threadid] += 1;
			}
		}
		else
		{
			// δʩ��������ǰ�������ײ����֡����-1
			if (continuousFrameCounter[threadid] > 0)
			{
				continuousFrameCounter[threadid] -= 1;
			}
		}
	}
}

__device__ bool hapticCylinderCollisionContinue(
	float length, float radius,
	float* HPos, float* SPos,
	float* cylinderDir,
	float* position,
	float* collisionNormal, float* collisionPos)
{
	// moveDirָ�������߶��뵽���⹤����Ҫ�ƶ���������������λ��ָ������λ�˵�����
	float moveDir[3] = { SPos[0] - HPos[0],SPos[1] - HPos[1],SPos[2] - HPos[2] };
	float moveDistance = sqrt(moveDir[0] * moveDir[0] + moveDir[1] * moveDir[1] + moveDir[2] * moveDir[2]);
	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCross_D(cylinderDir, moveDir, normal);
	tetNormal_D(normal);


	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - HPos[0] ,position[1] - HPos[1] ,position[2] - HPos[2] };//���߼��ָ�������ײ�������
	float lineStart0[3] = { HPos[0] ,HPos[1] ,HPos[2] };// ��ǰ���߼��
	float lineStart1[3] = { SPos[0] ,SPos[1] ,SPos[2] };// ��һ֡���߼��
	float lineStart2[3] = { HPos[0] + cylinderDir[0] * length ,HPos[1] + cylinderDir[1] * length,HPos[2] + cylinderDir[2] * length };// ��ǰ֡����β��


	//����Ҫ�Ƚ���һ����ײ��⣬�������Ƿ�����ײ��������������Ҫ������ײ���


	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	// �����᣺ ���߷���cylinderDir���˶�����moveDir�����߷������˶������ųɵ�ƽ��ķ�����normal
	// ������ײ��������������ɵľֲ�����ϵ����[x, y, z] �ô������������������ϵ�£�x������ײ���ڹ�����ͶӰ��λ�ã�y�������˶������ϵ��˶�����
	// ��˹��Ԫ��[A|I] ֻʹ����֮��ļӼ���A���I��I����A�������
	float x, y, z;
	float det = tetSolveFormula_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float distance = 0.0;
	bool flag = false;
	//2.���������ҵ��������ڵ�����
	if (x > length && y > moveDistance) {
		//����㵽��ľ���
		float basePoint[3] = { SPos[0] + length * cylinderDir[0],SPos[1] + length * cylinderDir[1] , SPos[2] + length * cylinderDir[2] };
		distance = tetPointPointDistance_D(position, basePoint);
		flag = true;
	}
	else if (x > length && y < moveDistance && y>0.0) {
		//����㵽�ߵľ���
		distance = tetPointLineDistance_D(lineStart2, moveDir, position);
	}
	else if (x > length && y < 0.0) {
		distance = tetPointPointDistance_D(position, lineStart2);
	}
	else if (x > 0.0 && x < length && y > moveDistance) {
		distance = tetPointLineDistance_D(lineStart1, cylinderDir, position);
		flag = true;
	}
	else if (x > 0.0 && x < length && y < moveDistance && y>0.0) {
		//����㵽��ľ���
		distance = abs(z);
	}
	else if (x > 0.0 && x < length && y < 0.0) {
		distance = tetPointLineDistance_D(lineStart0, cylinderDir, position);
	}
	else if (x<0.0 && y > moveDistance) {
		distance = tetPointPointDistance_D(position, SPos);
		flag = true;
	}
	else if (x < 0.0 && y < moveDistance && y>0.0) {
		distance = tetPointLineDistance_D(lineStart0, moveDir, position);
	}
	else if (x < 0.0 && y < 0.0) {
		distance = tetPointPointDistance_D(position, HPos);
	}


	//3.�жϾ���
	if (distance > radius) return false;
	//if (flag) return false;

	//printf("x:%f,y:%f,z:%f\n", x, y, z);

	//4. �����������ײ�ų�λ��
	//����Ԫһ�η���,�����������Բ�����м���
	float lineDir[3] = { moveDir[0],moveDir[1], moveDir[2] };


	float v0[3] = { position[0] - lineStart0[0] ,position[1] - lineStart0[1] ,position[2] - lineStart0[2] };
	float v1[3] = { position[0] - lineStart1[0] ,position[1] - lineStart1[1] ,position[2] - lineStart1[2] };
	float v2[3] = { position[0] - lineStart2[0] ,position[1] - lineStart2[1] ,position[2] - lineStart2[2] };


	//��Բ���ཻ
	float solve00, solve01;
	float solve10, solve11;
	tetSolveInsect_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
	tetSolveInsect_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
	float solve = min(solve11, solve01);
	//tetSolveInsect_D(lineDir, cylinderDir, v1, radius, &solve00, &solve01);
	//solve = min(solve, solve01);
	//tetSolveInsect_D(lineDir, moveDir, v2, radius, &solve00, &solve01);
	//solve = min(solve, solve01);


	//�����ཻ
	float solve20, solve21;
	tetSolveInsectSphere_D(lineDir, v0, radius, &solve20, &solve21);
	solve = min(solve, solve21);
	//printf("%f\n", solve);
	//tetSolveInsectSphere_D(lineDir, v1, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, v2, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, VSubO, radius, &solve10, &solve11);
	//solve = min(solve, solve11);

	if (solve != solve) return false;
	//printf("x:%f,y:%f,z:%f, solve: %f\n",x,y,z, solve);

	//����λ�õõ������ų���λ��
	collisionPos[0] = position[0] - lineDir[0] * solve;
	collisionPos[1] = position[1] - lineDir[1] * solve;
	collisionPos[2] = position[2] - lineDir[2] * solve;
	printf("solve01 solve11 solve21:%f %f %f\np[%f %f %f] collisionP[%f %f %f]\n", solve01, solve11, solve21, position[0], position[1], position[2], collisionPos[0], collisionPos[1], collisionPos[2]);

	//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
	float projPos[3] = { collisionPos[0] - HPos[0],collisionPos[1] - HPos[1],collisionPos[2] - HPos[2] };
	float proj = tetDot_D(projPos, cylinderDir);
	projPos[0] = collisionPos[0] - HPos[0] - cylinderDir[0] * proj;
	projPos[1] = collisionPos[1] - HPos[1] - cylinderDir[1] * proj;
	projPos[2] = collisionPos[2] - HPos[2] - cylinderDir[2] * proj;

	tetNormal_D(projPos);
	collisionNormal[0] = projPos[0];
	collisionNormal[1] = projPos[1];
	collisionNormal[2] = projPos[2];

	return true;
}

// �������嶥�����ײ��⣬�������⹤����������֮��ľ���ϴ�ʱ���������⹤����������֮���ɨ��������ײ��⣬�����ӹ��߶������嶥��ʩ��ѹ���ķ�Χ��.
__global__ void hapticCalculateMeshCylinder(
	float* cylinderPos,
	float* hapticCylinderPos,
	float* cylinderDir, float halfLength, float radius, 
	float* tetPositions, 
	float* vertexNormals, 
	unsigned int* isCollide, 
	float* vertexForce, // �ӹ���ָ����ײ��ķ�����
	float* zone, 
	int* continuousFrameCounter, int max_continuous_frame,
	int vertexNum, 
	int* index)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;

	//printf("threadid:%d p[%f %f %f]\n", threadid, tetPositions[indexX], tetPositions[indexY], tetPositions[indexZ]);

	float nx = vertexNormals[indexX];
	float ny = vertexNormals[indexY];
	float nz = vertexNormals[indexZ];
	float len_normal = sqrt(nx * nx + ny * ny + nz * nz);
	bool isOnSurface;
	//if (threadid == 500)
	//{
	//	printf("p[%f %f %f], n[%f %f %f]\n", 
	//		tetPositions[indexX], tetPositions[indexY], tetPositions[indexZ],
	//		nx, ny, nz);
	//}
	if (len_normal < 0.1)
		isOnSurface = false;
	else
	{
		isOnSurface = true;
		nx /= len_normal;
		ny /= len_normal;
		nz /= len_normal;
	}
		
	
	//if (len_normal < 0.1)// ������Ϊ0���õ�Ϊ�����ڲ��Ķ��㣬��������ײ����������ļ��㡣
	//	return;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];
	__shared__ float hapticCylinderTip[3];

	hapticCylinderTip[0] = hapticCylinderPos[0];
	hapticCylinderTip[1] = hapticCylinderPos[1];
	hapticCylinderTip[2] = hapticCylinderPos[2];

	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = tetPositions[indexX] - cylinder0[0];
	float dy = tetPositions[indexY] - cylinder0[1];
	float dz = tetPositions[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength; // t����ײ���ڹ����ϵİٷֱ�λ�ã����Ϊ0��β��Ϊ1

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// ���������ϵ� �Ӵ��������ͶӰ��->������λ��
	// ���Ӵ����ڹ��߸��ϵ�ʱ�򣬸�������ֱ�ڹ��������ߣ��ӹ��������ϵ�ͶӰ��ָ��Ӵ��㡣
	// ���Ӵ����ڹ��߼�˵�ʱ����������ӹ�������ļ��ָ��Ӵ��㡣
	dx = tetPositions[indexX] - cylinder0[0] - t * cylinderd[0];
	dy = tetPositions[indexY] - cylinder0[1] - t * cylinderd[1];
	dz = tetPositions[indexZ] - cylinder0[2] - t * cylinderd[2];

	float sqr_distance = dx * dx + dy * dy + dz * dz;
	float distance = sqrt(sqr_distance);
	dx /= distance; dy /= distance; dz /= distance;
	// ���������⹤�������ϵ�ͶӰ��
	float p0[3] = {
		cylinder0[0] + t * cylinderd[0],
		cylinder0[1] + t * cylinderd[1],
		cylinder0[2] + t * cylinderd[2] };
	// �����������������ϵ�ͶӰ��
	float p1[3] = {
		hapticCylinderTip[0] + t * cylinderd[0],
		hapticCylinderTip[1] + t * cylinderd[1],
		hapticCylinderTip[2] + t * cylinderd[2] };
	// �����⹤���ϵ�ͶӰ��ָ����������ͶӰ�������
	float v[3] = { p1[0] - p0[0], p1[1] - p0[1] ,p1[2] - p0[2] };
	float gh_distance = sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);

	const float GH_DISTANCE_THREASHOLD = 0.25;
	if (gh_distance < GH_DISTANCE_THREASHOLD)// ֱ����ͼ��λ������ײ���
	{
		if (distance < radius)//�������Ľ����Ҫд����ײ�������ڼ������⹤��λ��
		{
			if (isOnSurface)
			{
				// ���㹤���ڶ�����ʩ�ӵ�����������Ϊ[-nx, -ny, -nz](���涥�㷨�����ķ�����)
				atomicAdd(vertexForce + threadid * 3 + 0, -nx * (radius - distance));
				atomicAdd(vertexForce + threadid * 3 + 1, -ny * (radius - distance));
				atomicAdd(vertexForce + threadid * 3 + 2, -nz * (radius - distance));
			}
			else
			{
				float fx = dx * (radius - distance);
				float fy = dy * (radius - distance);
				float fz = dz * (radius - distance);
				vertexForce[indexX] += fx;
				vertexForce[indexY] += fy;
				vertexForce[indexZ] += fz;
				//printf("inner point collision:[%f %f %f]\n", fx, fy, fz);
			}

			isCollide[threadid] = 1;
			zone[threadid] = t;
			//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
			atomicAdd(index, 1);

			//// printf("���⹤�߰뾶��Χ�ڷ�����ײ�� threadid:%d counter: %d\n", threadid, continuousFrameCounter[threadid]);
			// ���߶Զ���ʩ����������ǰ�������ײ����֡����+1
			if(continuousFrameCounter[threadid]<max_continuous_frame)
			{
				continuousFrameCounter[threadid] += 1;
			}
		}
		else
		{
			// δʩ��������ǰ�������ײ����֡����-1
			if (continuousFrameCounter[threadid] > 0)
			{
				continuousFrameCounter[threadid] -= 1;
			}
		}
	}
	else if(gh_distance >= GH_DISTANCE_THREASHOLD) // ���⹤����������֮�������Ƚϴ�ľ��룬��ɨ���������ײ
	{
		float normal_weight = (gh_distance - distance) / gh_distance * radius;
		if (distance < radius)//�������Ľ����Ҫд����ײ�������ڼ������⹤��λ��
		{
			if (isOnSurface)
			{
				// ���㹤���ڶ�����ʩ�ӵ�����������Ϊ[-nx, -ny, -nz](���涥�㷨�����ķ�����)
				atomicAdd(vertexForce + threadid * 3 + 0, -nx * normal_weight);
				atomicAdd(vertexForce + threadid * 3 + 1, -ny * normal_weight);
				atomicAdd(vertexForce + threadid * 3 + 2, -nz * normal_weight);
			}
			else
			{
				float fx = dx * normal_weight;
				float fy = dy * normal_weight;
				float fz = dz * normal_weight;
				vertexForce[indexX] += fx;
				vertexForce[indexY] += fy;
				vertexForce[indexZ] += fz;
				//printf("saomiaoti in radius f[%f %f %f]\n", fx, fy, fz);
			}
			isCollide[threadid] = 1;
			zone[threadid] = t;
			//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
			atomicAdd(index, 1);

			////printf("��ɨ����뾶��Χ�ڣ�threadid: %d counter:%d\n", threadid, continuousFrameCounter[threadid]);
			// ���߶Զ���ʩ����������ǰ�������ײ����֡����+1
			if (continuousFrameCounter[threadid] < max_continuous_frame)
			{
				continuousFrameCounter[threadid] += 1;
			}
		}
		else if (distance < gh_distance)
		{
			// ����ʩ��ѹ���ķ���
			float dirX = v[0] / gh_distance;
			float dirY = v[1] / gh_distance;
			float dirZ = v[2] / gh_distance;
			// ���⹤���ϵ�ͶӰ��ָ����ײ���������
			float v_g2tetPos[3] = {
				tetPositions[indexX] - cylinder0[0],
				tetPositions[indexY] - cylinder0[1],
				tetPositions[indexZ] - cylinder0[2],
			};
			float temp = v_g2tetPos[0] * dirX + v_g2tetPos[1] * dirY + v_g2tetPos[2] * dirZ;
			float k = temp / gh_distance;
			if ((k < 1) && (k > 0))// ��������gh�����ϵ�ͶӰ����gh֮��
			{
				float projectedX = p0[0] + k * v[0];
				float projectedY = p0[1] + k * v[1];
				float projectedZ = p0[2] + k * v[2];
				float m[3] = {
					tetPositions[indexX] - projectedX,
					tetPositions[indexY] - projectedY,
					tetPositions[indexZ] - projectedZ
				};
				float dis = sqrt(m[0] * m[0] + m[1] * m[1] + m[2] * m[2]);
				//printf("case 2.2, dis=%f\n", dis);
				if (dis < radius)
				{
					// ��������Χ�ڣ��Ըö���ʩ��ѹ��
					float fx = dirX * normal_weight;
					float fy = dirY * normal_weight;
					float fz = dirZ * normal_weight;

					vertexForce[indexX] += fx;
					vertexForce[indexY] += fy;
					vertexForce[indexZ] += fz;
					//printf("saomiaoti f[%f %f %f] weight: %f\n", fx, fy, fz, normal_weight);
					//// printf("�ڹ��߰뾶��Χ��ɨ�����ڣ� threadid: %d counter: %d\n", threadid, continuousFrameCounter[threadid]);
					// ����ɨ����Զ���ʩ����������ǰ�������ײ����֡����+1
					if (continuousFrameCounter[threadid] < max_continuous_frame)
					{
						continuousFrameCounter[threadid] += 1;
					}
				}
				else
				{
					// printf("��ɨ����뾶��Χ�ڵ�û����ɨ������\n");
					if (continuousFrameCounter[threadid] > 0)
					{
						continuousFrameCounter[threadid] -= 1;
					}
				}
				
			}
			else
			{
				//printf("������ɨ��������ֱ���ϵ�ͶӰ����ɨ�����߶�֮�� k: %f\n", k);
				// δʩ��������ǰ�������ײ����֡����-1
				if (continuousFrameCounter[threadid] > 0)
				{
					continuousFrameCounter[threadid] -= 1;
				}
			}
		}
		else if (distance > gh_distance)
		{
			//printf("dis: %f, gh_dis:f ��ɨ�������뾶��\n", distance, gh_distance);
			// δʩ��������ǰ�������ײ����֡����-1
			if (continuousFrameCounter[threadid] > 0)
			{
				continuousFrameCounter[threadid] -= 1;
			}
		}
	}
}

__global__ void hapticCollision_MeshCapsule(float* cylinderPos, float* cylinderDir, float halfLength, float radius,
	float* vertexPositions,
	float* vertexNormals,
	unsigned int* isCollide,
	float* vertexForce, // �ӹ���ָ����ײ��ķ�����
	float* collisionDiag, float collisionStiffness,
	float* zone,
	int vertexNum,
	int* index)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;

	float nx = vertexNormals[indexX];
	float ny = vertexNormals[indexY];
	float nz = vertexNormals[indexZ];
	float len_normal = nx * nx + ny * ny + nz * nz;
	if (len_normal < 0.1)// ������Ϊ0���õ�Ϊ�����ڲ��Ķ��㣬��������ײ����������ļ��㡣
		return;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];


	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = vertexPositions[indexX] - cylinder0[0];
	float dy = vertexPositions[indexY] - cylinder0[1];
	float dz = vertexPositions[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength; // t����ײ���ڹ����ϵİٷֱ�λ�ã����Ϊ0��β��Ϊ1

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// ���������ϵ� �Ӵ��������ͶӰ��->������λ�ã���������ֱ�ڹ���
	dx = vertexPositions[indexX] - cylinder0[0] - t * cylinderd[0];
	dy = vertexPositions[indexY] - cylinder0[1] - t * cylinderd[1];
	dz = vertexPositions[indexZ] - cylinder0[2] - t * cylinderd[2];

	float sqr_distance = dx * dx + dy * dy + dz * dz;
	if (sqr_distance > radius * radius) return;
	float distance = sqrt(sqr_distance);

	// ��λ��
	dx /= distance;
	dy /= distance;
	dz /= distance;

	// ���㷴����������Ϊ[dx, dy, dz]
	atomicAdd(vertexForce + threadid * 3 + 0, dx * (radius - distance));
	atomicAdd(vertexForce + threadid * 3 + 1, dy * (radius - distance));
	atomicAdd(vertexForce + threadid * 3 + 2, dz * (radius - distance));
	collisionDiag[indexX] += dx * dx * collisionStiffness;
	collisionDiag[indexY] += dy * dy * collisionStiffness;
	collisionDiag[indexZ] += dz * dz * collisionStiffness;

	//���ñ�־λ

	isCollide[threadid] = 1;
	zone[threadid] = t;
	//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
	atomicAdd(index, 1);
}

__global__ void hapticCalculateMeshCapsule(float* cylinderPos, float* cylinderDir, float halfLength, float radius,
	float* tetPositions,
	float* vertexNormals,
	unsigned int* isCollide,
	float* vertexForce, // �ӹ���ָ����ײ��ķ�����
	float* zone,
	int vertexNum,
	int* index)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;

	float nx = vertexNormals[indexX];
	float ny = vertexNormals[indexY];
	float nz = vertexNormals[indexZ];
	float len_normal = nx * nx + ny * ny + nz * nz;
	if (len_normal < 0.1)// ������Ϊ0���õ�Ϊ�����ڲ��Ķ��㣬��������ײ����������ļ��㡣
		return;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];


	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = tetPositions[indexX] - cylinder0[0];
	float dy = tetPositions[indexY] - cylinder0[1];
	float dz = tetPositions[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength; // t����ײ���ڹ����ϵİٷֱ�λ�ã����Ϊ0��β��Ϊ1

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// ���������ϵ� �Ӵ��������ͶӰ��->������λ�ã���������ֱ�ڹ���
	dx = tetPositions[indexX] - cylinder0[0] - t * cylinderd[0];
	dy = tetPositions[indexY] - cylinder0[1] - t * cylinderd[1];
	dz = tetPositions[indexZ] - cylinder0[2] - t * cylinderd[2];

	float sqr_distance = dx * dx + dy * dy + dz * dz;
	if (sqr_distance > radius * radius) return;
	float distance = sqrt(sqr_distance);

	// ��λ��
	dx /= distance;
	dy /= distance;
	dz /= distance;

	// ���㷴����������Ϊ[dx, dy, dz]
	atomicAdd(vertexForce + threadid * 3 + 0, dx * (radius - distance));
	atomicAdd(vertexForce + threadid * 3 + 1, dy * (radius - distance));
	atomicAdd(vertexForce + threadid * 3 + 2, dz * (radius - distance));

	//���ñ�־λ

	isCollide[threadid] = 1;
	zone[threadid] = t;
	//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
	atomicAdd(index, 1);
}

__global__ void hapticCalculateCCylinder(float* cylinderPos, float* cylinderDir, float halfLength, float radius, float* tetPositions, unsigned int* isCollide, float* zone,int vertexNum, int* index) 
//float* cylinderPos, ���߼��λ��
//float* cylinderDir, ���߷���
//float halfLength, ���߳���
//float radius, ���߰뾶
//float* tetPositions, ������λ��
//unsigned int* isCollide, �Ƿ�����ײ
//float* zone, ��ײλ��
//int vertexNum, ����������
//int* index ÿ����һ����ײ�����ֵ+1
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];


	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = tetPositions[indexX] - cylinder0[0];
	float dy = tetPositions[indexY] - cylinder0[1];
	float dz = tetPositions[indexZ] - cylinder0[2];
	float t = cylinderDir[0] *dx + cylinderDir[1] *dy + cylinderDir[2] *dz;

	t /= halfLength; // t����ײ���ڹ����ϵİٷֱ�λ�ã����Ϊ0��β��Ϊ1

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// ���������ϵ� �Ӵ��������ͶӰ��->������λ�ã���������ֱ�ڹ���
	dx = tetPositions[indexX] - cylinder0[0] - t* cylinderd[0];
	dy = tetPositions[indexY] - cylinder0[1] - t* cylinderd[1];
	dz = tetPositions[indexZ] - cylinder0[2] - t* cylinderd[2];

	float distance = dx * dx + dy * dy + dz * dz;
	if (distance > radius*radius) return;
	//���ñ�־λ
	isCollide[threadid] = 1;
	zone[threadid] = t;
	//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
	atomicAdd(index, 1);

}

//����ǰ׺��
__global__ void hapticCalculatePrefixSum(unsigned int* isCollide, unsigned int* queueIndex, unsigned int* auxArray, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	//�������������Ĵ�Сδ֪
	extern __shared__ unsigned int temp[];


	//�����ڹ����ڴ��������
	temp[threadIdx.x] = isCollide[threadid];

	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < blockDim.x)
			temp[index] += temp[index - stride];//index is alway bigger than stride
		__syncthreads();
	}
	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index + stride < blockDim.x)
			temp[index + stride] += temp[index];

	}
	__syncthreads();

	//����ÿ��block�ڵ�ǰ׺��
	queueIndex[threadid] = temp[threadIdx.x];


	//��������block�ĺ�
	if (threadid % (blockDim.x - 1) == 0 && threadid != 0) {
		auxArray[blockIdx.x] = queueIndex[threadid];
	}
}

// ������ײ���д��constraint������
// ��������
// unsigned int* isCollide, GPU�ϲ��м���ĵ���ײ���
//float* tetPositions, ������λ��
//float* tetNormals, �����巨����
//float* zone, �ڹ�������ײ�����λ��
//float* constraintPoints, �����������ײ��������λ��
//float* constraintNormals, ����� ������ײ�������巨����������������ࣩ
//float* constraintZone, ������ڹ�������ײ�����λ��
//unsigned int* queueIndex,
//unsigned int* auxArray, 
//int vertexNum
__global__ void hapticAddCollisionToQueue(unsigned int* isCollide, \
	float* tetPositions, float* tetNormals, float* zone, \
	float* constraintPoints, float* constraintNormals, float* constraintZone, \
	unsigned int* queueIndex, unsigned int* auxArray, int vertexNum) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	if (isCollide[threadid]) {
		int index = -1;
		//����index
		for (int block = 0; block < blockIdx.x; block++) {
			index += auxArray[block];
		}

		index += queueIndex[threadid];
		constraintPoints[index * 3 + 0] = tetPositions[threadid * 3 + 0];
		constraintPoints[index * 3 + 1] = tetPositions[threadid * 3 + 1];
		constraintPoints[index * 3 + 2] = tetPositions[threadid * 3 + 2];
		constraintNormals[index * 3 + 2] = tetNormals[threadid * 3 + 2];
		constraintNormals[index * 3 + 0] = tetNormals[threadid * 3 + 0];
		constraintNormals[index * 3 + 1] = tetNormals[threadid * 3 + 1];
		constraintZone[index] = zone[threadid];
	}
}

__global__ void hapticAddCollisionToQueue_SaveMap(unsigned int* isCollide, \
	float* tetPositions, float* tetNormals, float* zone, \
	float* constraintPoints, float* constraintNormals, float* constraintZone, \
	unsigned int* queueIndex, unsigned int* auxArray, int vertexNum, \
	int* collisionQueueIndex_to_vertIndex_array)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	if (isCollide[threadid]) {
		int index = -1;
		//����index
		for (int block = 0; block < blockIdx.x; block++) {
			index += auxArray[block];
		}

		index += queueIndex[threadid];
		constraintPoints[index * 3 + 0] = tetPositions[threadid * 3 + 0];
		constraintPoints[index * 3 + 1] = tetPositions[threadid * 3 + 1];
		constraintPoints[index * 3 + 2] = tetPositions[threadid * 3 + 2];
		constraintNormals[index * 3 + 2] = tetNormals[threadid * 3 + 2];
		constraintNormals[index * 3 + 0] = tetNormals[threadid * 3 + 0];
		constraintNormals[index * 3 + 1] = tetNormals[threadid * 3 + 1];
		constraintZone[index] = zone[threadid];

		int queueIndex = index;
		int vertIndex = threadid;
		collisionQueueIndex_to_vertIndex_array[index] = vertIndex;
	}
}

__global__ void hapticAddSphereCollisionToQueue(unsigned int* isCollide, float* sphereInfos, float* zone, float* directDirection, float* constraintPoints, float* constraintZone, float* constraintDirection, unsigned int* queueIndex, unsigned int* auxArray, int sphereNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= sphereNum) return;

	if (isCollide[threadid]) {
		int index = -1;
		//����index
		for (int block = 0; block < blockIdx.x; block++) {
			index += auxArray[block];
		}

		index += queueIndex[threadid];
		//����ײ�����λ�úͰ뾶���б���
		constraintPoints[index * 4 + 0] = sphereInfos[threadid * 5 + 0];
		constraintPoints[index * 4 + 1] = sphereInfos[threadid * 5 + 1];
		constraintPoints[index * 4 + 2] = sphereInfos[threadid * 5 + 2];
		constraintPoints[index * 4 + 3] = sphereInfos[threadid * 5 + 3];
		constraintZone[index] = zone[threadid];
		constraintDirection[index * 3 + 0] = directDirection[threadid * 3 + 0];
		constraintDirection[index * 3 + 1] = directDirection[threadid * 3 + 1];
		constraintDirection[index * 3 + 2] = directDirection[threadid * 3 + 2];
	}
}

//�����ݷ��õ�������--Mesh
__global__ void hapticAddSphereCollisionToQueue_Tri(unsigned int* isCollide, float* sphereInfos, float* zone, float* constraintPoints, float* constraintZone, unsigned int* queueIndex, unsigned int* auxArray, int sphereNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= sphereNum) return;

	if (isCollide[threadid]) {
		int index = -1;
		//����index
		for (int block = 0; block < blockIdx.x; block++) {
			index += auxArray[block];
		}


		index += queueIndex[threadid];
		//����ײ�����λ�úͰ뾶���б���
		constraintPoints[index * 4 + 0] = sphereInfos[threadid * 4 + 0];
		constraintPoints[index * 4 + 1] = sphereInfos[threadid * 4 + 1];
		constraintPoints[index * 4 + 2] = sphereInfos[threadid * 4 + 2];
		constraintPoints[index * 4 + 3] = sphereInfos[threadid * 4 + 3];
		constraintZone[index] = zone[threadid];
	}
}

__global__ void hapticCalculateContinueCylinder(float startx, float starty, float startz, float endx, float endy, float endz, int* index, float* positions, float* boxs, float* triangleNormal, int boxNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= boxNum) return;

	float start[3] = { startx,starty,startz };
	float end[3] = { endx,endy,endz };

	//�ȺͰ�Χ�н�����ײ���
	float p0, p1;
	bool collision = hapticLineSegAABBInsect(start, end, &p0, &p1, boxs+threadid * 6);
	if (!collision) return;

	//�������ν�����ײ���
	int index0 = index[threadid * 3 + 0];
	int index1 = index[threadid * 3 + 1];
	int index2 = index[threadid * 3 + 2];
	float pos0[3] = { positions[index0 * 3 + 0],positions[index0 * 3 + 1], positions[index0 * 3 + 2] };
	float pos1[3] = { positions[index1 * 3 + 0],positions[index1 * 3 + 1], positions[index1 * 3 + 2] };
	float pos2[3] = { positions[index2 * 3 + 0],positions[index2 * 3 + 1], positions[index2 * 3 + 2] };
	collision = hapticLineSegTriangleInsect(start, end, pos0, pos1, pos2, triangleNormal+threadid * 3 + 0, &p0);
	if (collision) {
		//printf("%d: ����ײ\n",threadid);
	}
}

//ʹ��Բ�����������ײ���
__global__ void hapticCalculateCylinderSphere(float* cylinderPos, float* cylinderDir, float halfLength, float radius,float* sphereInfos,float* sphereForce,unsigned int* isCollide,float* zone, int* index,int sphereNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= sphereNum) return;
	//printf("%d\n", threadid);

	int indexX = threadid * 5 + 0;
	int indexY = threadid * 5 + 1;
	int indexZ = threadid * 5 + 2;

	float sphereRadius = sphereInfos[threadid * 5 + 3];

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;
	//radius *= 1.2;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];


	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = sphereInfos[indexX] - cylinder0[0];
	float dy = sphereInfos[indexY] - cylinder0[1];
	float dz = sphereInfos[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength;

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	// �����ϵĽӴ���ָ��Ӵ������ĵ�������
	dx = sphereInfos[indexX] - cylinder0[0] - t* cylinderd[0];
	dy = sphereInfos[indexY] - cylinder0[1] - t* cylinderd[1];
	dz = sphereInfos[indexZ] - cylinder0[2] - t* cylinderd[2];

	float distance = dx * dx + dy * dy + dz * dz;
	if (distance > (sphereRadius+radius)*(sphereRadius+radius)) return;

	//printf("%d:����ײ\n", threadid);
	//�������յ�����
	// ��׼������
	dx /= distance;
	dy /= distance;
	dz /= distance;
	// ����Ƕ����Ⱦ����������ɱ�׼����ָ������������
	atomicAdd(sphereForce + threadid * 3 + 0, dx*(sphereRadius+radius-distance));
	atomicAdd(sphereForce + threadid * 3 + 1, dy*(sphereRadius+radius-distance));
	atomicAdd(sphereForce + threadid * 3 + 2, dz*(sphereRadius+radius-distance));

	//���ñ�־λ
	isCollide[threadid] = 1;
	zone[threadid] = t;
	//printf("sphere index = %d\n", threadid);
	//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
	atomicAdd(index, 1);
}

//Բ���������ײ--Mesh
__global__ void hapticCalculateCylinderSphere_Tri(float* cylinderPos, float* cylinderDir, float halfLength, float radius, float* sphereInfos, float* sphereForce, unsigned int* isCollide, float* zone, int* index, int sphereNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= sphereNum) return;
	//printf("%d\n", threadid);

	int indexX = threadid * 4 + 0;
	int indexY = threadid * 4 + 1;
	int indexZ = threadid * 4 + 2;

	float sphereRadius = sphereInfos[threadid * 4 + 3];

	//������ײ��־λ
	isCollide[threadid] = 0;
	zone[threadid] = -1;
	radius *= 2.0;

	__shared__ float cylinder0[3];
	__shared__ float cylinder1[3];
	__shared__ float cylinderd[3];


	cylinder0[0] = cylinderPos[0];
	cylinder0[1] = cylinderPos[1];
	cylinder0[2] = cylinderPos[2];

	cylinder1[0] = cylinderPos[0] + cylinderDir[0] * halfLength;
	cylinder1[1] = cylinderPos[1] + cylinderDir[1] * halfLength;
	cylinder1[2] = cylinderPos[2] + cylinderDir[2] * halfLength;

	cylinderd[0] = cylinder1[0] - cylinder0[0];
	cylinderd[1] = cylinder1[1] - cylinder0[1];
	cylinderd[2] = cylinder1[2] - cylinder0[2];
	float dx = sphereInfos[indexX] - cylinder0[0];
	float dy = sphereInfos[indexY] - cylinder0[1];
	float dz = sphereInfos[indexZ] - cylinder0[2];
	float t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	t /= halfLength;

	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	dx = sphereInfos[indexX] - cylinder0[0] - t * cylinderd[0];
	dy = sphereInfos[indexY] - cylinder0[1] - t * cylinderd[1];
	dz = sphereInfos[indexZ] - cylinder0[2] - t * cylinderd[2];

	float distance = dx * dx + dy * dy + dz * dz;
	if (distance > (sphereRadius + radius)*(sphereRadius + radius)) return;

	//printf("%d:����ײ\n", threadid);
	//�������յ�����
	dx /= distance;
	dy /= distance;
	dz /= distance;
	atomicAdd(sphereForce + threadid * 3 + 0, dx*(sphereRadius + radius - distance));
	atomicAdd(sphereForce + threadid * 3 + 1, dy*(sphereRadius + radius - distance));
	atomicAdd(sphereForce + threadid * 3 + 2, dz*(sphereRadius + radius - distance));

	//���ñ�־λ
	isCollide[threadid] = 1;
	zone[threadid] = t;
	//����������һ(��ײ��Ϣ�ڼ���ǰ׺�͵�ʱ��ֵ)
	atomicAdd(index, 1);
}

//�߶κ�AABB��Χ�е���
__device__ bool hapticLineSegAABBInsect(float* start, float* end,float* p0,float* p1, float* boxs) {

	float dir[3] = {end[0]-start[0],end[1]-start[1],end[3]-start[3]};


	//��ȡ��Χ�еĽ���
	float minx = boxs[0];
	float miny = boxs[1];
	float minz = boxs[2];
	float maxx = boxs[3];
	float maxy = boxs[4];
	float maxz = boxs[5];


	//��ȡ������ƽ��Ľ���
	float t0x = (minx - start[0]) / dir[0];
	float t1x = (maxx - start[0]) / dir[0];
	if (t0x > t1x) hapticSwap(&t0x, &t1x);
	float t0y = (miny - start[1]) / dir[1];
	float t1y = (maxy - start[1]) / dir[1];
	if (t0y > t1y) hapticSwap(&t0y, &t1y);
	float t0z = (minz - start[2]) / dir[2];
	float t1z = (maxz - start[2]) / dir[2];
	if (t0z > t1z) hapticSwap(&t0z, &t1z);

	//�ҵ��ཻ���ֵĵ��
	float t0 = (t0x < t0y) ? t0y : t0x;
	float t1 = (t1x < t1y) ? t1x : t1y;
	t0 = (t0 > t0z) ? t0 : t0z;
	t1 = (t1 > t1z) ? t1z : t1;

	//��ȡ�⣬����Ҫclamp��01֮��
	*p1 = t0;
	*p0 = t1;

	if (*p0 > *p1) return false;

	//����Ͱ�Χ���ཻ������Ҫ��һ���ж��Ƿ���01֮��
	*p0 = hapticClamp(*p0, 0, 1);
	*p1 = hapticClamp(*p1, 0, 1);

	//���������ͬ��Ҳ�ǲ��ཻ��
	if (abs(*p0 - *p1) < 0.0001) return false;

	return true;
}


//�߶κ������ε���
__device__ bool hapticLineSegTriangleInsect(float* start, float* end, float* pos0, float* pos1, float* pos2, float* triangleNormal,float* ans) {
	//�ȼ����ƽ��Ľ���
	float insectPoint[3];

	//��������
	float v[3] = {pos0[0]-start[0],pos0[1] - start[1], pos0[2] - start[2]};

	float dir[3] = { end[0] - start[0],end[1] - start[1],end[3] - start[3] };
	float length = sqrt( dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2] );
	dir[0] /= length;
	dir[1] /= length;
	dir[2] /= length;

	float dotNV = triangleNormal[0] * v[0] + triangleNormal[1] * v[1] + triangleNormal[2] * v[2];
	float dotND = triangleNormal[0] * dir[0] + triangleNormal[1] * dir[1] + triangleNormal[2] * dir[2];
	float t = dotNV/ dotND;


	//�����ж��Ƿ����߶��ڲ�
	if (t<0 || t> length) return false;

	float p[3] = { start[0] + t*dir[0],start[1] + t*dir[1], start[2] + t*dir[2] };

	//���жϵ��ڲ�����������,ʹ�ò�˷�
	float cross0[3];
	float v0[3] = { pos1[0] - pos0[0],pos1[1] - pos0[1], pos1[2] - pos0[2] };
	float v0p[3] = { p[0] - pos0[0],p[1] - pos0[1], p[2] - pos0[2] };
	hapticCross(v0, v0p, cross0);

	float cross1[3];
	float v1[3] = { pos2[0] - pos1[0],pos2[1] - pos1[1], pos2[2] - pos1[2] };
	float v1p[3] = { p[0] - pos1[0],p[1] - pos1[1], p[2] - pos1[2] };
	hapticCross(v1, v1p, cross1);

	float cross2[3];
	float v2[3] = { pos0[0] - pos2[0],pos0[1] - pos2[1], pos0[2] - pos2[2] };
	float v2p[3] = { p[0] - pos2[0],p[1] - pos2[1], p[2] - pos2[2] };
	hapticCross(v1, v1p, cross1);

	//������ַ���ת,��Ϊ���ཻ
	float flag = hapticDot(cross0,cross1);
	if (flag < 0) return false;
	flag = hapticDot(cross1, cross2);
	if (flag < 0) return false;

	*ans = t;
	return true;
}

// ����ײ��Ϣ����
__global__ void dispatchForceToTetVertex(
	float* externForce,
	float* vertexForce,
	unsigned int* isCollide,
	int vertexNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	if (isCollide[threadid] == 0) return;

	atomicAdd(externForce + indexX, vertexForce[indexX]);
	atomicAdd(externForce + indexY, vertexForce[indexY]);
	atomicAdd(externForce + indexZ, vertexForce[indexZ]);

	// ��ն����ϵ�������
	vertexForce[indexY]	= 0.0;
	vertexForce[indexZ]	= 0.0;
	vertexForce[indexX]	= 0.0;
}
// ����ײ���ϵ�����Ϊ����ʩ�ӵ��󶨵Ķ����ϡ�
__global__ void dispatchToTet(
	unsigned int* skeletonIndex, 
	float* skeletonCoord,
	float* externForce,
	float* sphereForce, 
	unsigned int* isCollide, 
	int sphereNum) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= sphereNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//���û���������뿪
	if (isCollide[threadid] == 0) return;

	//��������Ķ���������Ȩ��
	int tetIndex0 = skeletonIndex[threadid * 4 + 0];
	int tetIndex1 = skeletonIndex[threadid * 4 + 1];
	int tetIndex2 = skeletonIndex[threadid * 4 + 2];
	int tetIndex3 = skeletonIndex[threadid * 4 + 3];

	float weight0 = skeletonCoord[threadid * 4 + 0];
	float weight1 = skeletonCoord[threadid * 4 + 1];
	float weight2 = skeletonCoord[threadid * 4 + 2];
	float weight3 = skeletonCoord[threadid * 4 + 3];

	//
	atomicAdd(externForce + tetIndex0 * 3 + 0, sphereForce[indexX] * weight0);
	atomicAdd(externForce + tetIndex0 * 3 + 1, sphereForce[indexY] * weight0);
	atomicAdd(externForce + tetIndex0 * 3 + 2, sphereForce[indexZ] * weight0);

	atomicAdd(externForce + tetIndex1 * 3 + 0, sphereForce[indexX] * weight1);
	atomicAdd(externForce + tetIndex1 * 3 + 1, sphereForce[indexY] * weight1);
	atomicAdd(externForce + tetIndex1 * 3 + 2, sphereForce[indexZ] * weight1);

	atomicAdd(externForce + tetIndex2 * 3 + 0, sphereForce[indexX] * weight2);
	atomicAdd(externForce + tetIndex2 * 3 + 1, sphereForce[indexY] * weight2);
	atomicAdd(externForce + tetIndex2 * 3 + 2, sphereForce[indexZ] * weight2);

	atomicAdd(externForce + tetIndex3 * 3 + 0, sphereForce[indexX] * weight3);
	atomicAdd(externForce + tetIndex3 * 3 + 1, sphereForce[indexY] * weight3);
	atomicAdd(externForce + tetIndex3 * 3 + 2, sphereForce[indexZ] * weight3);

}

__global__ void hapticUpdatePointPosition(
	float* mass,
	float* position,
	float* velocity,
	float* forceFromTool,
	float dt,
	int point_num)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= point_num) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	float forceX = forceFromTool[indexX];
	float forceY = forceFromTool[indexY];
	float forceZ = forceFromTool[indexZ];
	// acceleration of points due to external force
	float a_x = forceX / mass[threadid];
	float a_y = forceY / mass[threadid];
	float a_z = forceZ / mass[threadid];

	//printf("mass in thread %d: %f\tforceX: %f\n", threadid, mass[threadid], forceX);
	if (isnan(a_x))
	{
		if (isnan(forceX))
		{
			printf("%d-nan occured in force_x\n", threadid);
		}
		else if (isnan(mass[indexX]))
			printf("nan occured in mass_x\n");
	}

	//// update velocity using force and mass
	float delta_v_x = a_x * dt;
	float delta_v_y = a_y * dt;
	float delta_v_z = a_z * dt;

	float delta_pos_x = velocity[indexX] * dt + 0.5 * a_x * dt * dt;
	float delta_pos_y = velocity[indexX] * dt + 0.5 * a_y * dt * dt;
	float delta_pos_z = velocity[indexX] * dt + 0.5 * a_z * dt * dt;

	float delta_pos_len = sqrt(delta_pos_x * delta_pos_x + delta_pos_y * delta_pos_y + delta_pos_z * delta_pos_z);
	if(delta_pos_len>1)
		printf("%d-delta_pos_len:%f\n", threadid, delta_pos_len);
	atomicAdd(position + indexX, delta_pos_x);
	atomicAdd(position + indexY, delta_pos_y);
	atomicAdd(position + indexZ, delta_pos_z);
	atomicAdd(velocity + indexX, delta_v_x);
	atomicAdd(velocity + indexY, delta_v_y);
	atomicAdd(velocity + indexZ, delta_v_z);
}

__global__ void AccumulateExternForce(float* externForceTotal, float* externForce, int point_num)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= point_num) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	externForceTotal[indexX] += externForce[indexX];
	externForceTotal[indexY] += externForce[indexY];
	externForceTotal[indexZ] += externForce[indexZ];
}

//����ɨ������ײ��⣬��������λ���ųɵ�ɨ�������ײ��⡣
__device__ bool hapticCylinderCollisionContinue(
	float length, // ���߳��� 
	float moveDistance, // ����ɨ������ֹ�����������λ��֮��ľ��� 
	float radius,// ���߰뾶
	float* cylinderPos, //ɨ�����ص�Ĺ���λ��
	float* cylinderLastPos, // ɨ�������Ĺ���λ��
	float* cylinderDir, // ���߷���
	float* moveDir, //������㵽�յ���˶�����
	float* position, // ��Ҫ������ײ���Ķ���λ��
	float* collisionNormal, //��ײ֮�󶥵㱻�ų��ķ���
	float* collisionPos // ���㱻�ų���ɨ��������λ��
)
{
	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCross_D(cylinderDir, moveDir, normal);
	tetNormal_D(normal);

	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - cylinderPos[0] ,position[1] - cylinderPos[1] ,position[2] - cylinderPos[2] };//���߼��ָ�������ײ�������
	float lineStart0[3] = { cylinderPos[0] ,cylinderPos[1] ,cylinderPos[2] };// ��ǰ���߼��
	float lineStart1[3] = { cylinderLastPos[0] ,cylinderLastPos[1] ,cylinderLastPos[2] };// ��һ֡���߼��
	float lineStart2[3] = { cylinderPos[0] + cylinderDir[0] * length ,cylinderPos[1] + cylinderDir[1] * length,cylinderPos[2] + cylinderDir[2] * length };// ��ǰ֡����β��


	//����Ҫ�Ƚ���һ����ײ��⣬�������Ƿ�����ײ��������������Ҫ������ײ���


	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	// �����᣺ ���߷���cylinderDir���˶�����moveDir�����߷������˶������ųɵ�ƽ��ķ�����normal
	// ������ײ��������������ɵľֲ�����ϵ����[x, y, z] �ô������������������ϵ�£�x������ײ���ڹ�����ͶӰ��λ�ã�y�������˶������ϵ��˶�����
	// ��˹��Ԫ��[A|I] ֻʹ����֮��ļӼ���A���I��I����A�������
	float x, y, z;
	float det = tetSolveFormula_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float distance = 0.0;
	bool flag = false;
	//2.���������ҵ��������ڵ�����
	if (x > length && y > moveDistance) {
		//����㵽��ľ���
		float basePoint[3] = { cylinderLastPos[0] + length * cylinderDir[0],cylinderLastPos[1] + length * cylinderDir[1] , cylinderLastPos[2] + length * cylinderDir[2] };
		distance = tetPointPointDistance_D(position, basePoint);
		flag = true;
	}
	else if (x > length && y < moveDistance && y>0.0) {
		//����㵽�ߵľ���
		distance = tetPointLineDistance_D(lineStart2, moveDir, position);
	}
	else if (x > length && y < 0.0) {
		distance = tetPointPointDistance_D(position, lineStart2);
	}
	else if (x > 0.0 && x < length && y > moveDistance) {
		distance = tetPointLineDistance_D(lineStart1, cylinderDir, position);
		flag = true;
	}
	else if (x > 0.0 && x < length && y < moveDistance && y>0.0) {
		//����㵽��ľ���
		distance = abs(z);
	}
	else if (x > 0.0 && x < length && y < 0.0) {
		distance = tetPointLineDistance_D(lineStart0, cylinderDir, position);
	}
	else if (x<0.0 && y > moveDistance) {
		distance = tetPointPointDistance_D(position, cylinderLastPos);
		flag = true;
	}
	else if (x < 0.0 && y < moveDistance && y>0.0) {
		distance = tetPointLineDistance_D(lineStart0, moveDir, position);
	}
	else if (x < 0.0 && y < 0.0) {
		distance = tetPointPointDistance_D(position, cylinderPos);
	}


	//3.�жϾ���
	if (distance > radius) return false;
	//if (flag) return false;

	//printf("x:%f,y:%f,z:%f\n", x, y, z);

	//4. �����������ײ�ų�λ��
	//����Ԫһ�η���,�����������Բ�����м���
	float lineDir[3] = { moveDir[0],moveDir[1], moveDir[2] };


	float v0[3] = { position[0] - lineStart0[0] ,position[1] - lineStart0[1] ,position[2] - lineStart0[2] };
	float v1[3] = { position[0] - lineStart1[0] ,position[1] - lineStart1[1] ,position[2] - lineStart1[2] };
	float v2[3] = { position[0] - lineStart2[0] ,position[1] - lineStart2[1] ,position[2] - lineStart2[2] };


	//��Բ���ཻ
	float solve00, solve01;
	float solve10, solve11;
	tetSolveInsect_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
	tetSolveInsect_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
	float solve = min(solve11, solve01);
	//tetSolveInsect_D(lineDir, cylinderDir, v1, radius, &solve00, &solve01);
	//solve = min(solve, solve01);
	//tetSolveInsect_D(lineDir, moveDir, v2, radius, &solve00, &solve01);
	//solve = min(solve, solve01);


	//�����ཻ
	float solve20, solve21;
	tetSolveInsectSphere_D(lineDir, v0, radius, &solve20, &solve21);
	solve = min(solve, solve21);
	//printf("%f\n", solve);
	//tetSolveInsectSphere_D(lineDir, v1, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, v2, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, VSubO, radius, &solve10, &solve11);
	//solve = min(solve, solve11);

	if (solve != solve) return false;
	//printf("x:%f,y:%f,z:%f, solve: %f\n",x,y,z, solve);

	//����λ�õõ������ų���λ��
	collisionPos[0] = position[0] - lineDir[0] * solve;
	collisionPos[1] = position[1] - lineDir[1] * solve;
	collisionPos[2] = position[2] - lineDir[2] * solve;

	//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
	float projPos[3] = { collisionPos[0] - cylinderPos[0],collisionPos[1] - cylinderPos[1],collisionPos[2] - cylinderPos[2] };
	float proj = tetDot_D(projPos, cylinderDir);
	projPos[0] = collisionPos[0] - cylinderPos[0] - cylinderDir[0] * proj;
	projPos[1] = collisionPos[1] - cylinderPos[1] - cylinderDir[1] * proj;
	projPos[2] = collisionPos[2] - cylinderPos[2] - cylinderDir[2] * proj;

	tetNormal_D(projPos);
	collisionNormal[0] = projPos[0];
	collisionNormal[1] = projPos[1];
	collisionNormal[2] = projPos[2];

	//printf("continue: x:%f,y:%f,z:%f,solve:%f\n", collisionPos[0], collisionPos[1], collisionPos[2],solve);
	//printf("continue: nx:%f,ny:%f,nz:%f\n", collisionNormal[0], collisionNormal[1], collisionNormal[2]);
	return true;
}

__device__ float ContactForceDecay(float distance, float original_radius, float max_radius)
// distance: distance between point and tool central axis
// original_radius: tool actual radius
// max_radius: the radius of exerting tool force
{
	float f1 = -1 / original_radius * distance + 1;
	float f2 = -1 / max_radius * distance + 1;
	float t = distance / max_radius;
	float scale = (1 - t) * f1 + t * f2;
	//printf("scale: %f\n", scale);
	return scale;
}

__device__ void hapticSwap(float* a, float* b) {
	float temp = *a;
	*a = *b;
	*b = temp;
}
__device__ float hapticClamp(float a, float min, float max) {
	return a<min ? min : (a>max ? max : a);
}

__device__ void hapticCross(float* a, float* b, float* c) {
	//��˼��������η���
	c[0] = a[1] * b[2] - b[1] * a[2];
	c[1] = a[2] * b[0] - b[2] * a[0];
	c[2] = a[0] * b[1] - b[0] * a[1];
}

__device__ float hapticDot(float* a, float* b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

