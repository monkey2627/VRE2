#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"

//���ڴ洢ָ������

int* tetShellIdx_d;

float* directDirectionMU_D;
int* directIndexMU_D;

//��ײ������
float* planeNormal_D;
float* planePos_D;

float* toolPositionAndDirection_d;
float* toolPosePrev_d;
float* radius_d;

int* hapticCollisionNum_d;
float* toolContactDeltaPos_triVert_d;
float* totalFC_d;//���⹤�����ܵĽӴ���������Ϊ3
float* totalPartial_FC_X_d;//���⹤�����ܵĽӴ����Թ���λ�õ��ݶȣ�����Ϊ9
float* totalPartial_FC_Omega_d;//���⹤�����ܵĽӴ����Թ��߳�����ݶȣ�����Ϊ9
float* totalTC_d;// ���⹤�����ܵĽӴ���Ť�أ�����Ϊ3�����ݵ��������豸����α�ʾ����
float* totalPartial_TC_X_d;// Ť���ڹ���λ�õ��ݶȣ�����Ϊ9
float* totalPartial_TC_Omega_d;// Ť���ڹ��߳��򴦵��ݶȣ�����Ϊ9

//Բ���幤��
float* cylinderShiftMU_D;
float* cylinderLastPosMU_D;
float* cylinderPosMU_D;
float* cylinderDirZMU_D;
float* cylinderDirYMU_D;
float* cylinderDirXMU_D;
float	graperHalfLengthMU_D;

//ץǯ����ײ��Ϣ
float	grapperRadiusMU_D;
float	grapperLengthMU_D;

unsigned int* CollideFlagMU_D;	//�����Ƿ�����ײ

//ץȡ
bool	firstGrabLeftMU_D;
bool	firstGrabRightMU_D;
unsigned int* isGrabLeftMU_D;
unsigned int* isGrabRigthMU_D;
unsigned int* isGrabHalfLeftMU_D;		//ץǯ�պϹ����еıպϹ�ϵ
unsigned int* isGrabHalfRightMU_D;
float* relativePositionLeftMU_D;
float* relativePositionRightMU_D;
unsigned int* CollideFlagLeftMU_D;		//��־λ�������Ƿ�������֯������ײ
unsigned int* CollideFlagRightMU_D;



//��ײԼ����
float* triVertCollisionForce_d;
float* insertionDepthMU_D;

//����Բ������ײ
__device__ float* cylinderShift;
__device__ float* cylinderLastPos;
__device__ float* cylinderPos;
__device__ float* cylinderDirZ; // Բ���ı�������z�᷽�򣬼�Բ������ķ�����ά��
__device__ float* cylinderV;
__device__ unsigned char* cylinderCollideFlag;

//ֻ���㼷ѹʱ��ָ������������ʱ������
int runcalculateToolShift(float halfLength, float radius, int cylinderIdx) {

	//ѡȡ�����ֵĹ���
	int		cylinderButton = HAPTIC_BUTTON::normal;

	////ÿ�ζ����㣬ÿ�μ����µ�
	//if (flag == 1) {
	//	hipMemset(cylinderShiftLeft_D, 0, 3 * sizeof(float));
	//	cylinderShift = cylinderShiftLeft_D;
	//	cylinderPos = cylinderPosLeft_D;
	//	cylinderDirZ = ;
	//	cylinderDirY = cylinderDirYLeft_D;
	//	cylinderDirX = cylinderDirXLeft_D;
	//	cylinderButton = cylinderButtonLeft_D;
	//	grapperUpDirZ = tetgrapperUpDirZLeft_D;
	//	grapperDownDirZ = tetgrapperDownDirZRight_D;
	//}
	//else {
	//	hipMemset(cylinderShiftRight_D, 0, 3 * sizeof(float));
	//	cylinderShift = cylinderShiftRight_D;
	//	cylinderPos = cylinderPosRight_D;
	//	cylinderDirZ = cylinderDirZRight_D;
	//	cylinderDirY = cylinderDirYRight_D;
	//	cylinderDirX = cylinderDirXRight_D;
	//	cylinderButton = cylinderButtonRight_D;
	//	grapperUpDirZ = tetgrapperUpDirZRight_D;
	//	grapperDownDirZ = tetgrapperDownDirZRight_D;
	//}
	cylinderShift = &cylinderShift_d[cylinderIdx * 3];
	hipMemset(cylinderShift, 0.0f, 3 * sizeof(float));
	cylinderPos = &cylinderPos_d[cylinderIdx * 3];
	cylinderDirZ = &cylinderDirZ_d[cylinderIdx * 3];
	cylinderV = &cylinderV_d[cylinderIdx * 3];
	cylinderCollideFlag = &toolCollideFlag_d[cylinderIdx];

	switch (cylinderButton)
	{
	case cut:
		break;
	case grab:
		break;
	case normal: 
		{
			int  threadNum = 512;
			int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
			//д��ƫ��������
			calculateToolShift << <blockNum, threadNum >> > (
				cylinderPos, cylinderDirZ, 
				tetVertNonPenetrationDir_d, 
				halfLength, radius, 
				tetVertPos_d, cylinderShift, tetVertNum_d);

			hipDeviceSynchronize();
		}
		break;
	default:
		break;
	}
	return 0;
}

//ֻ���㼷ѹʱ��ָ������������ʱ������
int runcalculateToolShiftMU(float halfLength, float radius, int cylinderIdx) {

	//ѡȡ�����ֵĹ��� ע�⣺���밴����Ҫ��
	int		cylinderButton = HAPTIC_BUTTON::normal;

	cylinderShift = &cylinderShift_d[cylinderIdx * 3];
	hipMemset(cylinderShift, 0.0f, 3 * sizeof(float));
	cylinderPos = &cylinderPos_d[cylinderIdx * 3];
	cylinderDirZ = &cylinderDirZ_d[cylinderIdx * 3];
	cylinderV = &cylinderV_d[cylinderIdx * 3];
	cylinderCollideFlag = &toolCollideFlag_d[cylinderIdx];

	switch (cylinderButton)
	{
	case cut:
		break;
	case grab:
		break;
	case normal:
	{
		int  threadNum = 512;
		int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
		//д��ƫ��������
		calculateToolShift << <blockNum, threadNum >> > (
			cylinderPos, cylinderDirZ,
			triVertNonPenetrationDir_d,
			halfLength, radius,
			triVertPos_d, cylinderShift, triVertNum_d);

		hipDeviceSynchronize();
	}
	break;
	default:
		break;
	}
	return 0;
}

__global__ void calculateToolShift(
	float* cylinderPos, float* cylinderDir,
	float* directDir,
	float halfLength, float radius,
	float* positions,
	float* cylinderShift,
	int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//����һ�����ߵ���ײ���
	//ָ��������ײ���ķ�����ָ�������ķ��򣬴Ӱ��������ָ�򶥵�
	float moveDir[3];
	moveDir[0] = directDir[indexX];
	moveDir[1] = directDir[indexY];
	moveDir[2] = directDir[indexZ];


	//ʹ��ָ�������������ײ���
	bool collision = cylinderRayCollisionDetection(cylinderPos, cylinderDir, positions[indexX], positions[indexY], positions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
	if (!collision) return;


	//�ۼӵõ�ƫ������
	atomicAdd(cylinderShift + 0, -directDir[indexX]);
	atomicAdd(cylinderShift + 1, -directDir[indexY]);
	atomicAdd(cylinderShift + 2, -directDir[indexZ]);

	//printf("ƫ����:%f,%f,%f\n", cylinderShift[0], cylinderShift[1], cylinderShift[2]);
}

int runcalculateCollisionCylinder(float halfLength, float radius, 
	float collisionStiffness, float adsorbStiffness, float frictionStiffness, 
	int idx) {
	cylinderShift = &cylinderShift_d[idx * 3];
	cylinderLastPos = &cylinderLastPos_d[idx * 3];
	cylinderPos  = &cylinderPos_d[idx * 3];
	cylinderDirZ = &cylinderDirZ_d[idx * 3];
	cylinderV = &cylinderV_d[idx * 3];
	cylinderCollideFlag = &toolCollideFlag_d[idx];

	///�˴�������Բ�������ײ
	int  threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	calculateCollisionCylinderSDF << <blockNum, threadNum >> > (
		cylinderLastPos, cylinderPos, 
		cylinderDirZ, halfLength, radius, 
		tetVertPos_d, tetVertForce_d, tetVertisCollide_d, cylinderCollideFlag, 
		tetVertCollisionDiag_d, 
		tetVertNum_d,
		collisionStiffness, tetVertCollisionForce_d, tetVertNonPenetrationDir_d, cylinderShift);

	hipDeviceSynchronize();
	return 0;
}

int runcalculateCollisionCylinderMU(float halfLength, float radius,
	float collisionStiffness, float adsorbStiffness, float frictionStiffness,
	int idx)
{
	cylinderShift = &cylinderShift_d[idx * 3];
	cylinderLastPos = &cylinderLastPos_d[idx * 3];
	cylinderPos = &cylinderPos_d[idx * 3];
	cylinderDirZ = &cylinderDirZ_d[idx * 3];
	cylinderV = &cylinderV_d[idx * 3];
	cylinderCollideFlag = &toolCollideFlag_d[idx];

	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	calculateCollisionCylinderSDF << <blockNum, threadNum >> > (
		cylinderLastPos, cylinderPos,
		cylinderDirZ, halfLength, radius,
		triVertPos_d, triVertForce_d, triVertisCollide_d, cylinderCollideFlag,
		triVertCollisionDiag_d,
		triVertNum_d,
		collisionStiffness, triVertCollisionForce_d, triVertNonPenetrationDir_d, cylinderShift);
	hipDeviceSynchronize();
	return 0;
}
//ʹ�û���SDF�ı�������ײ����㷨
__global__ void calculateCollisionCylinderSDF(
	float* cylinderLastPos, float* cylinderPos, float* cylinderDir, float halfLength, float radius, 
	float* positions, float* force, unsigned char* isCollide, unsigned char* collideFlag, 
	float* collisionDiag, 
	int vertexNum, 
	float collisionStiffness, float* collisionForce, 
	float* directDirection, float* cylinderShift) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;    //�����ڹ��������ϵ�ͶӰ
	float collisionNormal[3];   //��ײ�ų�����
	float collisionPos[3];   //��ײ�ų�λ��
	int indexX = threadid * 3 + 0;  //����λ�������������е��±�
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	float directDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };

	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}
	float radius_extent_ratio = 0;
	//ƫ��һ���뾶���ȣ���������뾶Ϊԭ����������ʵ��ƫ�ĵ�Բ��
	float newPos[3];
	newPos[0] = cylinderPos[0] + cylinderShift[0] * radius_extent_ratio * radius;
	newPos[1] = cylinderPos[1] + cylinderShift[1] * radius_extent_ratio * radius;
	newPos[2] = cylinderPos[2] + cylinderShift[2] * radius_extent_ratio * radius;

	float tetPositions[3] = { positions[indexX], positions[indexY], positions[indexZ] };  //�����嶥��λ��
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPos[0],cylinderLastPos[1] - cylinderPos[1], cylinderLastPos[2] - cylinderPos[2] };  //�����˶�������
	float proj = tetDot_D(toolMoveDir, cylinderDir);  //�����˶������������ڹ��������ϵ�ͶӰ
	float moveDistance = tetNormal_D(toolMoveDir);  //����֡����λ��֮���ֱ�Ӿ���
	//moveDistance = sqrt(moveDistance* moveDistance - proj * proj);  //����֡�����ڹ������ߴ�ֱ����ľ���
	
	radius *= (1+radius_extent_ratio);

	//ʹ�û���SDF��������ײ���
	if (moveDistance > 0.5) {  //�������֡����λ�˾����Զ����ʹ��������ײ���
		bool collisionSDF = cylinderCollisionContinueSDF(halfLength, moveDistance, radius, cylinderPos, cylinderLastPos, cylinderDir, toolMoveDir, tetPositions, directDir, collisionNormal, collisionPos);
		if (!collisionSDF) return;  //δ��ײֱ���˳�
	}
	else {  //�������֡����λ�˾���Ͻ�����ʹ����ɢ��ײ���
		bool collision = cylinderCollisionSDF(newPos, cylinderDir, tetPositions, directDir, halfLength, radius, &t, collisionNormal, collisionPos);
		if (!collision) return;  //δ��ײֱ���˳�
	}

	float deltaPos[3];
	deltaPos[0] = collisionPos[0] - positions[indexX];
	deltaPos[1] = collisionPos[1] - positions[indexY];
	deltaPos[2] = collisionPos[2] - positions[indexZ];

	//if(threadid==72990)
		//printf("threadid:%d collided, deltaPos[%f %f %f]\n", threadid, deltaPos[0], deltaPos[1], deltaPos[2]);

	float temp[3];  //Լ����Ӧ����ײ��
	temp[0] = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
	temp[1] = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
	temp[2] = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
	collisionForce[indexX] += temp[0];
	collisionForce[indexY] += temp[1];
	collisionForce[indexZ] += temp[2];

	//������
	force[indexX] += temp[0];
	force[indexY] += temp[1];
	force[indexZ] += temp[2];


	//����Խ�Ԫ�ض�Ӧ��ֵ
	collisionDiag[indexX] += collisionStiffness * collisionNormal[0] * collisionNormal[0];
	collisionDiag[indexY] += collisionStiffness * collisionNormal[1] * collisionNormal[1];
	collisionDiag[indexZ] += collisionStiffness * collisionNormal[2] * collisionNormal[2];


	//���ñ�־λ
	isCollide[threadid] = 1;
	collideFlag[0] = 1;
}

//ʹ�û���SDF�ı�������ײ����㷨
__global__ void calculateCollisionCylinderSDF(
	float* cylinderLastPos, float* cylinderPos, float* cylinderDir, float halfLength, float radius,
	float* positions, float* force, unsigned char* isCollide, unsigned char* collideFlag,
	float* collisionDiag,
	int* sortedIndices, int offset, int activeElementNum,
	float collisionStiffness, float* collisionForce,
	float* directDirection, float* cylinderShift)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[offset + threadid];
	float t = 0.0;    //�����ڹ��������ϵ�ͶӰ
	float collisionNormal[3];   //��ײ�ų�����
	float collisionPos[3];   //��ײ�ų�λ��
	int indexX = vertIdx * 3 + 0;  //����λ�������������е��±�
	int indexY = vertIdx * 3 + 1;
	int indexZ = vertIdx * 3 + 2;

	float directDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };

	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}
	float radius_extent_ratio = 0;
	//ƫ��һ���뾶���ȣ���������뾶Ϊԭ����������ʵ��ƫ�ĵ�Բ��
	float newPos[3];
	newPos[0] = cylinderPos[0] + cylinderShift[0] * radius_extent_ratio * radius;
	newPos[1] = cylinderPos[1] + cylinderShift[1] * radius_extent_ratio * radius;
	newPos[2] = cylinderPos[2] + cylinderShift[2] * radius_extent_ratio * radius;

	float tetPositions[3] = { positions[indexX], positions[indexY], positions[indexZ] };  //�����嶥��λ��
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPos[0],cylinderLastPos[1] - cylinderPos[1], cylinderLastPos[2] - cylinderPos[2] };  //�����˶�������
	float proj = tetDot_D(toolMoveDir, cylinderDir);  //�����˶������������ڹ��������ϵ�ͶӰ
	float moveDistance = tetNormal_D(toolMoveDir);  //����֡����λ��֮���ֱ�Ӿ���
	//moveDistance = sqrt(moveDistance* moveDistance - proj * proj);  //����֡�����ڹ������ߴ�ֱ����ľ���

	radius *= (1 + radius_extent_ratio);

	//ʹ�û���SDF��������ײ���
	if (moveDistance > 0.5) {  //�������֡����λ�˾����Զ����ʹ��������ײ���
		bool collisionSDF = cylinderCollisionContinueSDF(halfLength, moveDistance, radius, cylinderPos, cylinderLastPos, cylinderDir, toolMoveDir, tetPositions, directDir, collisionNormal, collisionPos);
		if (!collisionSDF) return;  //δ��ײֱ���˳�
	}
	else {  //�������֡����λ�˾���Ͻ�����ʹ����ɢ��ײ���
		bool collision = cylinderCollisionSDF(newPos, cylinderDir, tetPositions, directDir, halfLength, radius, &t, collisionNormal, collisionPos);
		if (!collision) return;  //δ��ײֱ���˳�
	}

	float deltaPos[3];
	deltaPos[0] = collisionPos[0] - positions[indexX];
	deltaPos[1] = collisionPos[1] - positions[indexY];
	deltaPos[2] = collisionPos[2] - positions[indexZ];

	float temp[3];  //Լ����Ӧ����ײ��
	temp[0] = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
	temp[1] = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
	temp[2] = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
	collisionForce[indexX] += temp[0];
	collisionForce[indexY] += temp[1];
	collisionForce[indexZ] += temp[2];

	//������
	force[indexX] += temp[0];
	force[indexY] += temp[1];
	force[indexZ] += temp[2];


	//����Խ�Ԫ�ض�Ӧ��ֵ
	collisionDiag[indexX] += collisionStiffness * collisionNormal[0] * collisionNormal[0];
	collisionDiag[indexY] += collisionStiffness * collisionNormal[1] * collisionNormal[1];
	collisionDiag[indexZ] += collisionStiffness * collisionNormal[2] * collisionNormal[2];


	//���ñ�־λ
	isCollide[vertIdx] = 1;
	*collideFlag = 1;
}

//����SDF��������ײ���
__device__ bool cylinderCollisionContinueSDF(float length, float moveDistance, float radius, 
	float* cylinderPos, float* cylinderLastPos, float* cylinderDir, float* moveDir, float* position, 
	float* directDir, float* collisionNormal, float* collisionPos) {
	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCross_D(cylinderDir, moveDir, normal);
	tetNormal_D(normal);

	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - cylinderPos[0] ,position[1] - cylinderPos[1] ,position[2] - cylinderPos[2] };
	float lineStart0[3] = { cylinderPos[0] ,cylinderPos[1] ,cylinderPos[2] };
	float lineStart1[3] = { cylinderLastPos[0] ,cylinderLastPos[1] ,cylinderLastPos[2] };
	float lineEnd1[3] = { cylinderLastPos[0] + cylinderDir[0] * length, cylinderLastPos[1] + cylinderDir[1] * length, cylinderLastPos[2] + cylinderDir[2] * length };

	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	float x, y, z;
	float det = tetSolveFormula_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float MaxDis = 0.75 * moveDistance;  //SDF��ͬ���򻮷ֵ���ֵ
	float distance = 0.0;  //�����ڲ�ͬ������������
	int flag = 0;
	//2.���������ҵ��������ڵ�����
	if (x > 0.0 && x < length && y <= 0) {  //�ڵ�ǰ֡����Բ���İ�Բ������
		//��Բ�����߾���
		distance = tetPointLineDistance_D(lineStart0, cylinderDir, position);
		flag = 1;
	}
	else if (x > 0.0 && x < length && y < moveDistance && y > 0.0) {  //��ǰ��֡��������֮���ɨ��������
		//����ľ���
		distance = abs(z);
		if (y <= MaxDis) {
			flag = 1;
		}
		else {
			flag = 3;
		}
	}
	else if (x > 0.0 && x < length && y > moveDistance) {  //����һ֡����Բ���İ�Բ������
		//��Բ�����ߵľ���
		distance = tetPointLineDistance_D(lineStart1, cylinderDir, position);
		flag = 3;
	}
	else return false;  //δ��ײ

	//3.�жϾ���
	if (distance > radius) return false;

	//�ж��ųⷽ���Ƿ��ָ���������
	/*if (flag == 1) {
		float proDir[3] = { -moveDir[0], -moveDir[1], -moveDir[2] };
		float proA = tetDot_D(proDir, directDir);
		if (proA < -0.5) {
			flag = 3;
		}
	}
	else if (flag == 3) {
		float proA = tetDot_D(moveDir, directDir);
		if (proA < -0.5) {
			flag = 1;
		}
	}*/

	//4.���ݲ�ͬ����������ͬ�÷����ų⣬�����ų�λ��
	if (flag == 1) {  //��ǰ֡����Բ���ų�
		float lineDir[3] = { moveDir[0], moveDir[1], moveDir[2] };

		float v0[3] = { position[0] - lineStart0[0], position[1] - lineStart0[1], position[2] - lineStart0[2] };

		float solve00, solve01;
		float solve10, solve11;
		tetSolveInsect_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
		tetSolveInsect_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
		float solve = min(solve11, solve01);
		//float solve = solve11;

		if (solve != solve) return false;

		//���㶥���ų���λ��
		collisionPos[0] = position[0] - lineDir[0] * solve;
		collisionPos[1] = position[1] - lineDir[1] * solve;
		collisionPos[2] = position[2] - lineDir[2] * solve;

		//���㶥�����ײ����
		float projPos[3] = { collisionPos[0] - cylinderPos[0], collisionPos[1] - cylinderPos[1], collisionPos[2] - cylinderPos[2] };
		float proj = tetDot_D(projPos, cylinderDir);
		projPos[0] = collisionPos[0] - cylinderPos[0] - cylinderDir[0] * proj;
		projPos[1] = collisionPos[1] - cylinderPos[1] - cylinderDir[1] * proj;
		projPos[2] = collisionPos[2] - cylinderPos[2] - cylinderDir[2] * proj;

		tetNormal_D(projPos);
		collisionNormal[0] = projPos[0];
		collisionNormal[1] = projPos[1];
		collisionNormal[2] = projPos[2];
	}
	else if (flag == 2) {  //ֱ����ɨ���������ų�
		if (z >= 0.0) {
			//�����ų�����
			collisionNormal[0] = normal[0];
			collisionNormal[1] = normal[1];
			collisionNormal[2] = normal[2];
		}
		else {
			//�����ų�����
			collisionNormal[0] = -normal[0];
			collisionNormal[1] = -normal[1];
			collisionNormal[2] = -normal[2];
		}

		//�����ų�λ��
		collisionPos[0] = position[0] + collisionNormal[0] * (radius - distance);
		collisionPos[1] = position[1] + collisionNormal[1] * (radius - distance);
		collisionPos[2] = position[2] + collisionNormal[2] * (radius - distance);
	}
	else if (flag == 3) {  //����һ֡����Բ���ų�
		float lineDir[3] = { -moveDir[0], -moveDir[1], -moveDir[2] };
		float cyDir[3] = { -cylinderDir[0], -cylinderDir[1], -cylinderDir[2] };

		float v0[3] = { position[0] - lineEnd1[0], position[1] - lineEnd1[1], position[2] - lineEnd1[2] };

		//��Բ����
		float solve00, solve01;
		float solve10, solve11;
		tetSolveInsect_D(lineDir, cyDir, v0, radius, &solve00, &solve01);
		tetSolveInsect_D(lineDir, lineDir, v0, radius, &solve10, &solve11);
		float solve = min(solve11, solve01);

		if (solve != solve) return false;

		//���㶥����ų�λ��
		collisionPos[0] = position[0] - lineDir[0] * solve;
		collisionPos[1] = position[1] - lineDir[1] * solve;
		collisionPos[2] = position[2] - lineDir[2] * solve;

		//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
		float projPos[3] = { collisionPos[0] - cylinderLastPos[0], collisionPos[1] - cylinderLastPos[1], collisionPos[2] - cylinderLastPos[2] };
		float proj = tetDot_D(projPos, cylinderDir);
		projPos[0] = collisionPos[0] - cylinderLastPos[0] - cylinderDir[0] * proj;
		projPos[1] = collisionPos[1] - cylinderLastPos[1] - cylinderDir[1] * proj;
		projPos[2] = collisionPos[2] - cylinderLastPos[2] - cylinderDir[2] * proj;

		tetNormal_D(projPos);
		collisionNormal[0] = projPos[0];
		collisionNormal[1] = projPos[1];
		collisionNormal[2] = projPos[2];

		/*float VSub1[3] = { position[0] - cylinderLastPos[0] ,position[1] - cylinderLastPos[1] ,position[2] - cylinderLastPos[2] };
		float proj = tetDot_D(VSub1, cylinderDir);
		float projPos[3] = { cylinderLastPos[0] + proj * cylinderDir[0], cylinderLastPos[1] + proj * cylinderDir[1], cylinderLastPos[2] + proj * cylinderDir[2] };

		//�����ų�����
		collisionNormal[0] = position[0] - projPos[0];
		collisionNormal[1] = position[1] - projPos[1];
		collisionNormal[2] = position[2] - projPos[2];
		tetNormal_D(collisionNormal);

		//�����ų�λ��
		collisionPos[0] = projPos[0] + collisionNormal[0] * radius;
		collisionPos[1] = projPos[1] + collisionNormal[1] * radius;
		collisionPos[2] = projPos[2] + collisionNormal[2] * radius;*/
	}

	return true;
}


//��Բ������ɢ��ײ��⣬��ָ������
__device__ bool cylinderCollisionSDF(float* pos, float* dir, float* vert, float* directDir, float length,
	float radius, float* t, float* collisionNormal, float* collisionPos) {
	float cylinder0x, cylinder0y, cylinder0z;
	cylinder0x = pos[0];
	cylinder0y = pos[1];
	cylinder0z = pos[2];
	float cylinder1x, cylinder1y, cylinder1z;
	cylinder1x = pos[0] + dir[0] * length;
	cylinder1y = pos[1] + dir[1] * length;
	cylinder1z = pos[2] + dir[2] * length;

	float cylinderdx = cylinder1x - cylinder0x;
	float cylinderdy = cylinder1y - cylinder0y;
	float cylinderdz = cylinder1z - cylinder0z;
	float dx = vert[0] - cylinder0x;
	float dy = vert[1] - cylinder0y;
	float dz = vert[2] - cylinder0z;
	*t = dir[0] * dx + dir[1] * dy + dir[2] * dz;

	*t /= length;

	if (*t < 0) {
		//return false;
		*t = 0;
	}
	else if (*t > 1) {
		//return false;
		*t = 1;
	}

	dx = vert[0] - cylinder0x - (*t) * cylinderdx;
	dy = vert[1] - cylinder0y - (*t) * cylinderdy;
	dz = vert[2] - cylinder0z - (*t) * cylinderdz;

	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	if (distance > radius) return false;

	//������ײ����ͶӰ�Ľ���
	float moveLength = sqrt(directDir[0] * directDir[0] + directDir[1] * directDir[1] + directDir[2] * directDir[2]);
	directDir[0] /= moveLength;
	directDir[1] /= moveLength;
	directDir[2] /= moveLength;

	collisionNormal[0] = directDir[0];
	collisionNormal[1] = directDir[1];
	collisionNormal[2] = directDir[2];

	float projectx = cylinder0x + (*t) * cylinderdx;
	float projecty = cylinder0y + (*t) * cylinderdy;
	float projectz = cylinder0z + (*t) * cylinderdz;

	//����local��,���һ��һԪ���η���
	float solution;
	float SN = (vert[0] - projectx) * (collisionNormal[0]) + (vert[1] - projecty) * (collisionNormal[1]) + (vert[2] - projectz) * (collisionNormal[2]);
	float SS = (vert[0] - projectx) * (vert[0] - projectx) + (vert[1] - projecty) * (vert[1] - projecty) + (vert[2] - projectz) * (vert[2] - projectz);
	solution = -SN + sqrt(SN * SN - SS + radius * radius);//ֻȡ����

	if (solution != solution) return false;

	collisionPos[0] = vert[0] + collisionNormal[0] * solution;
	collisionPos[1] = vert[1] + collisionNormal[1] * solution;
	collisionPos[2] = vert[2] + collisionNormal[2] * solution;

	//�ٴ���������
	dx = collisionPos[0] - projectx;
	dy = collisionPos[1] - projecty;
	dz = collisionPos[2] - projectz;
	distance = sqrt(dx * dx + dy * dy + dz * dz);
	collisionNormal[0] = dx / distance;
	collisionNormal[1] = dy / distance;
	collisionNormal[2] = dz / distance;

	return true;
}




//ʹ��������ײ���ı�������ײ����㷨
__global__ void calculateCollisionCylinderAdvance(
	float* cylinderLastPos, float* cylinderPos,
	float* cylinderDir, float* cylinderV,
	float halfLength, float radius,
	float* positions, float* velocity, float* force,
	unsigned int* isCollide,
	float* collisionDiag,
	float* volumnDiag,
	int vertexNum, float collisionStiffness, float frictionStiffness,
	float* collisionForce, float* directDir, float* cylinderShift)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}

	//ָ��������ײ���ķ���
	float moveDir[3];
	moveDir[0] = directDir[indexX];
	moveDir[1] = directDir[indexY];
	moveDir[2] = directDir[indexZ];

	float tetPosition[3] = { positions[indexX] ,positions[indexY] ,positions[indexZ] };
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPos[0],cylinderLastPos[1] - cylinderPos[1], cylinderLastPos[2] - cylinderPos[2] };
	float moveDistance = tetNormal_D(toolMoveDir);

	float ratio = 0.0f;
	float newPos[3];
	newPos[0] = cylinderPos[0] + cylinderShift[0] * ratio * radius;
	newPos[1] = cylinderPos[1] + cylinderShift[1] * ratio * radius;
	newPos[2] = cylinderPos[2] + cylinderShift[2] * ratio * radius;
	float w = moveDistance / radius;
	float enlarged_radius = radius * (1.5 - 0.5 / w);

	if (moveDistance > 0.05) {
		//�޸�ʹ��������ײ����������������ײ
		// �ж���ָ��������������ײ���
		bool collisionContinus = cylinderCollisionContinue(halfLength, moveDistance, enlarged_radius, cylinderPos, cylinderLastPos, cylinderDir, toolMoveDir, tetPosition, &t, collisionNormal, collisionPos, moveDir);
		// �޶���ָ��������������ײ���
		//bool collisionContinus = cylinderCollisionContinue_without_directDir(halfLength, moveDistance, enlarged_radius, cylinderPos, cylinderLastPos, cylinderDir, toolMoveDir, tetPosition, &t, collisionNormal, collisionPos);
		if (!collisionContinus) return;
		//printf("lianxu\n");
	}
	else {
		//ʹ��ָ�������������ײ���
		////bool collision = cylinderRayCollisionDetection(newPos, cylinderDir, triPositions[indexX], triPositions[indexY], triPositions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
		//bool collision = cylinderRayCollisionDetection(cylinderPos, cylinderDir, triPositions[indexX], triPositions[indexY], triPositions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
		float vert[3] = { positions[indexX], positions[indexY], positions[indexZ] };
		bool collision = cylinderCollision(cylinderPos, cylinderDir, vert, halfLength, radius, &t, collisionNormal, collisionPos);
		if (!collision) return;
		//printf("---lisan\n");
	}


	////ʹ��ָ�������������ײ���
	//bool collision = cylinderRayCollisionDetection(newPos, cylinderDir, triPositions[indexX], triPositions[indexY], triPositions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
	//if (!collision) return;

	float deltaPos[3];
	deltaPos[0] = collisionPos[0] - positions[indexX];
	deltaPos[1] = collisionPos[1] - positions[indexY];
	deltaPos[2] = collisionPos[2] - positions[indexZ];
	//float deltaPos_length = sqrt(deltaPos[0] * deltaPos[0] + deltaPos[1] * deltaPos[1] + deltaPos[2] * deltaPos[2]);
	//if (deltaPos_length > 1e-5)
	//{
	//	printf("thread %d deltaPos %f\tqg: [%f %f %f]\tcollision pos: [%f %f %f]\n", threadid, deltaPos_length, cylinderPos[0], cylinderPos[1], cylinderPos[2], collisionPos[0], collisionPos[1], collisionPos[2]);
	//}

	// ������ײ����Ӵ�����
	float temp[3];
	temp[0] = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
	temp[1] = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
	temp[2] = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
	collisionForce[indexX] += temp[0];
	collisionForce[indexY] += temp[1];
	collisionForce[indexZ] += temp[2];


	//����Ħ����
	float friction[3];
	friction[0] = 0.0;
	friction[1] = 0.0;
	friction[2] = 0.0;
	float frictionT[3];	//����Ħ����
	float frictionN[3];	//����Ħ����
						//��������˶����ٶȣ���������ٶ�������
	float v[3];
	v[0] = velocity[indexX] - cylinderV[0];
	v[1] = velocity[indexY] - cylinderV[1];
	v[2] = velocity[indexZ] - cylinderV[2];
	float c[3];
	c[0] = collisionStiffness * collisionNormal[0] * collisionNormal[0];
	c[1] = collisionStiffness * collisionNormal[1] * collisionNormal[1];
	c[2] = collisionStiffness * collisionNormal[2] * collisionNormal[2];
	c[0] += volumnDiag[threadid];
	c[1] += volumnDiag[threadid];
	c[2] += volumnDiag[threadid];
	//b-c(x-st)������໥������
	float relatedForce[3];
	relatedForce[0] = force[indexX] + c[0] * v[0] * 0.0009;
	relatedForce[1] = force[indexY] + c[1] * v[1] * 0.0009;
	relatedForce[2] = force[indexZ] + c[2] * v[2] * 0.0009;


	//�������������Ħ����
	float product = relatedForce[0] * collisionNormal[0]\
		+ relatedForce[1] * collisionNormal[1] \
		+ relatedForce[2] * collisionNormal[2];
	//if (product < 0) {
	//	//���㴹ֱ����
	//	frictionN[0] = -product*collisionNormal[0];
	//	frictionN[1] = -product*collisionNormal[1];
	//	frictionN[2] = -product*collisionNormal[2];

	//	//�����������
	//	relatedForce[0] -= product*collisionNormal[0];
	//	relatedForce[1] -= product*collisionNormal[1];
	//	relatedForce[2] -= product*collisionNormal[2];

	//	//�Ƿ񳬹���Ħ����ֵ
	//	float relatedlength = sqrt(relatedForce[0]* relatedForce[0]+ relatedForce[1]* relatedForce[1]+ relatedForce[2]* relatedForce[2]);
	//	
	//	float frictionNlength = sqrt(frictionN[0]* frictionN[0]+ frictionN[1]* frictionN[1]+ frictionN[2]* frictionN[2]);
	//	if (relatedlength <= frictionNlength*frictionStiffness) {
	//		
	//		frictionT[0] = -relatedForce[0];
	//		frictionT[1] = -relatedForce[1];
	//		frictionT[2] = -relatedForce[2];
	//
	//	}
	//	else {
	//		frictionT[0] = -frictionStiffness*frictionNlength*(relatedForce[0]/relatedlength);
	//		frictionT[1] = -frictionStiffness*frictionNlength*(relatedForce[1]/relatedlength);
	//		frictionT[2] = -frictionStiffness*frictionNlength*(relatedForce[2]/relatedlength);
	//	}

	//	friction[0] = frictionT[0];
	//	friction[1] = frictionT[1];
	//	friction[2] = frictionT[2];
	//}

	//������
	force[indexX] += temp[0] + friction[0];
	force[indexY] += temp[1] + friction[1];
	force[indexZ] += temp[2] + friction[2];
	//triForce[indexX] += friction[0];
	//triForce[indexY] += friction[1];
	//triForce[indexZ] += friction[2];


	//����Խ�Ԫ�ض�Ӧ��ֵ
	collisionDiag[indexX] += collisionStiffness * collisionNormal[0] * collisionNormal[0];
	collisionDiag[indexY] += collisionStiffness * collisionNormal[1] * collisionNormal[1];
	collisionDiag[indexZ] += collisionStiffness * collisionNormal[2] * collisionNormal[2];

	//���ñ�־λ
	isCollide[threadid] = 1;
}



int runcalculateCollisionSphere(float ball_radius, float p_collisionStiffness, int toolIdx, bool useClusterCollision)
{
	int  threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	// ���ɵ���ײ�������ĵ��������������ϵ�ָ������ͶӰ�㡱�ķ���һ�¡�
	calculateCollisionSphere << <blockNum, threadNum >> > (toolPositionAndDirection_d, ball_radius,
		tetVertPos_d, tetVertisCollide_d, toolIdx, toolCollideFlag_d, 
		tetVertNonPenetrationDir_d,
		tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
		p_collisionStiffness,
		tetVertNum_d);

	//// ���ɵ���ײ����ָ����������һ��
	//calculateCollisionSphereFollowDDir << <blockNum, threadNum >> > (toolPositionAndDirection_d, ball_radius,
	//	tetVertPos_d, tetVertisCollide_d,
	//	tetVertNonPenetrationDir_d,
	//	tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
	//	p_collisionStiffness,
	//	tetVertNum_d);

	calculateVec3Len << <blockNum, threadNum >> > (tetVertCollisionForce_d, tetVertCollisionForceLen_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateCollisionSphere");
	return 0;
}

int runcalculateCollisionSphereMU(float ball_radius, float collisionStiffness, int toolIdx)
{
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	calculateCollisionSphere << <blockNum, threadNum >> > (toolPositionAndDirection_d, ball_radius,
		triVertPos_d, triVertisCollide_d, toolIdx, toolCollideFlag_d,
		triVertNonPenetrationDir_d,
		triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, tetVertInsertionDepth_d,
		collisionStiffness,
		triVertNum_d);

	hipDeviceSynchronize();
	printCudaError("runcalculateCollisionSphereMU");
	return 0;
}

__global__ void calculateCollisionSphere(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, float* force, float* collisionForce,
	float* collisionDiag, float* insertionDepth, float collisionStiffness, int vertexNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid<10)
	//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid, 
	//	ballPos[0], ballPos[1], ballPos[2],
	//	p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		insertionDepth[threadid] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		force[indexX] += forcex;
		force[indexY] += forcey;
		force[indexZ] += forcez;
		collisionForce[indexX] += forcex;
		collisionForce[indexY] += forcey;
		collisionForce[indexZ] += forcez;
		float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		collisionDiag[indexX] += diagx;
		collisionDiag[indexY] += diagy;
		collisionDiag[indexZ] += diagz;
		if(threadid== LOOK_THREAD)
		{
			//printf("threadid:%d triForce[%f %f %f] diag[%f %f %f]\n", threadid,
			//	triForce[indexX], triForce[indexY], triForce[indexZ],
			//	diagx, diagy, diagz);
		}

		isCollide[threadid] = 1;
		toolCollideFlag[toolIdx] = 1;

		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], force:[%f,%f,%f]\n",
				threadid, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				force[indexX], force[indexY], force[indexZ]);
		}
		
	}
	else
	{
		insertionDepth[threadid] = 0;
	}
	return;
}
int runClearFc()
{
	printCudaError("runClearFc start");
	hipMemset(hapticCollisionNum_d, 0, sizeof(int));
	hipMemset(toolContactDeltaPos_triVert_d, 0.0f, triVertNum_d * 3 * sizeof(float));
	hipMemset(totalFC_d, 0.0f, 3 * sizeof(float));
	hipMemset(totalTC_d, 0.0f, 3 * sizeof(float));
	hipMemset(totalPartial_FC_X_d, 0.0f, 9 * sizeof(float));
	hipMemset(totalPartial_FC_Omega_d, 0.0f, 9 * sizeof(float));
	hipMemset(totalPartial_TC_X_d, 0.0f, 9 * sizeof(float));
	hipMemset(totalPartial_TC_Omega_d, 0.0f, 9 * sizeof(float));
	printCudaError("runClearFc");
	// ���ö���Ĭ��ͶӰλ��Ϊ��ǰλ��
	hipMemcpy(triVertProjectedPos_d, triVertPos_d, triVertNum_d * 3 * sizeof(float), hipMemcpyDeviceToDevice);
	// ������ײ���
	hipMemset(triVertisCollide_d, 0, triVertNum_d * sizeof(unsigned char));
	return 0;
}
int runHapticCollisionSphereForTri(float toolR, float p_collisionStiffness, float kc, int toolIdx)
{
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	hapticCollisionSphere <<<blockNum, threadNum>>>(toolPositionAndDirection_d, toolR,
		triVertPos_d, triVertisCollide_d, toolIdx, toolCollideFlag_d,
		triVertNonPenetrationDir_d,
		triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, triVertInsertionDepth_d,
		p_collisionStiffness,
		toolContactDeltaPos_triVert_d, totalFC_d, totalPartial_FC_X_d, kc,
		hapticCollisionNum_d, triVertNum_d);
	hipDeviceSynchronize();
	printCudaError("HapticCollisionSphereForTri");
	return 0;
}
int runHapticCollisionSphereForTet(float toolR, float p_collisionStiffness, float kc, int toolIdx)
{
	int  threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	hapticCollisionSphere << <blockNum, threadNum >> > (toolPositionAndDirection_d, toolR,
		tetVertPos_d, tetVertisCollide_d, toolIdx, toolCollideFlag_d,
		tetVertNonPenetrationDir_d,
		tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
		p_collisionStiffness,
		toolContactDeltaPos_triVert_d, totalFC_d, totalPartial_FC_X_d, kc,
		hapticCollisionNum_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("HapticCollisionSphereForTet");
	return 0;
}
__global__ void hapticCollisionSphere(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, float* force, float* collisionForce,
	float* collisionDiag, float* insertionDepth, float collisionStiffness,
	float* toolDeltaPos, float* F_c, float* partialFc, float k_c, int* collisionNumPtr, int vertexNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid<10)
	//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid, 
	//	ballPos[0], ballPos[1], ballPos[2],
	//	p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		insertionDepth[threadid] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		
		if (isnan(dDir[0]))
		{
			printf("threadid %d, nan in dDir, dealt as no collision, return\n", threadid);
			return;
		}
		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexX] += forcex;
		//triForce[indexY] += forcey;
		//triForce[indexZ] += forcez;
		collisionForce[indexX] += forcex;
		collisionForce[indexY] += forcey;
		collisionForce[indexZ] += forcez;
		float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		collisionDiag[indexX] += diagx;
		collisionDiag[indexY] += diagy;
		collisionDiag[indexZ] += diagz;
		
		if (threadid == LOOK_THREAD)
		{
			printf("threadid:%d force[%f %f %f] diag[%f %f %f]\n", threadid,
				collisionForce[indexX], collisionForce[indexY], collisionForce[indexZ],
				diagx, diagy, diagz);
		}
		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], force:[%f,%f,%f]\n",
				threadid, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				force[indexX], force[indexY], force[indexZ]);
		}

		float partial_F_c[9] = { -dDir[0] * dDir[0] * k_c, -dDir[0] * dDir[1] * k_c, -dDir[0] * dDir[2] * k_c,
								 -dDir[0] * dDir[1] * k_c, -dDir[1] * dDir[1] * k_c, -dDir[1] * dDir[2] * k_c,
								 -dDir[0] * dDir[2] * k_c, -dDir[1] * dDir[2] * k_c, -dDir[2] * dDir[2] * k_c };
		// toolDeltaPos ���ߵ�����˶������붥����˶������෴
		atomicAdd(toolDeltaPos + 0, -deltaPos[0]);
		atomicAdd(toolDeltaPos + 1, -deltaPos[1]);
		atomicAdd(toolDeltaPos + 2, -deltaPos[2]);
		// F_c �빤�ߵ��˶�������ͬ
		atomicAdd(F_c + 0, -deltaPos[0] * k_c);
		atomicAdd(F_c + 1, -deltaPos[1] * k_c);
		atomicAdd(F_c + 2, -deltaPos[2] * k_c);
		atomicAdd(partialFc + 0, partial_F_c[0]);
		atomicAdd(partialFc + 1, partial_F_c[1]);
		atomicAdd(partialFc + 2, partial_F_c[2]);
		atomicAdd(partialFc + 3, partial_F_c[3]);
		atomicAdd(partialFc + 4, partial_F_c[4]);
		atomicAdd(partialFc + 5, partial_F_c[5]);
		atomicAdd(partialFc + 6, partial_F_c[6]);
		atomicAdd(partialFc + 7, partial_F_c[7]);
		atomicAdd(partialFc + 8, partial_F_c[8]);
		//printf("threadid %d dDir[%f %f %f] fc[%f %f %f]\n partialFc[\n%f %f %f\n%f %f %f\n%f %f %f]\n", threadid, 
		//	dDir[0], dDir[1], dDir[2],
		//	-deltaPos[0] * k_c, -deltaPos[1] * k_c, -deltaPos[2] * k_c,
		//	partial_F_c[0], partial_F_c[1], partial_F_c[2],
		//	partial_F_c[3], partial_F_c[4], partial_F_c[5], 
		//	partial_F_c[6], partial_F_c[7], partial_F_c[8]);

		atomicAdd(collisionNumPtr, 1);

		//printf("thread%d dDir:[%f %f %f] F_c:[%f %f %f]\n", threadid, 
		//	dDir[0], dDir[1], dDir[2],
		//	- deltaPos[0] * k_c, -deltaPos[1] * k_c, -deltaPos[2] * k_c);
		isCollide[threadid] = 1;
		toolCollideFlag[toolIdx] = 1;
	}
	else
	{
		insertionDepth[threadid] = 0;
	}
	return;
}

int runHapticCollisionSphere_Merged(float toolR, float p_collisionStiffness, float kc, int toolIdx)
{
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	//hapticCollisionSphere_Merge<< <blockNum, threadNum >> > (toolPositionAndDirection_d, toolR,
	//	triVertPos_d, triVertisCollide_d, toolIdx, toolCollideFlag_d,
	//	triVertNonPenetrationDir_d,
	//	triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, triVertInsertionDepth_d,
	//	tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
	//	triVert2TetVertMapping_d,
	//	p_collisionStiffness,
	//	toolContactDeltaPos_triVert_d, totalFC_d, totalPartial_FC_X_d, kc,
	//	hapticCollisionNum_d, triVertNum_d);

	hapticCollisionSphere_Merge_with_Torque << <blockNum, threadNum >> > (toolPositionAndDirection_d, toolR,
		triVertPos_d, triVertisCollide_d, toolIdx, toolCollideFlag_d,
		triVertNonPenetrationDir_d,
		triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, triVertInsertionDepth_d,
		tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
		triVert2TetVertMapping_d,
		p_collisionStiffness,
		toolContactDeltaPos_triVert_d, 
		totalFC_d, totalPartial_FC_X_d, totalPartial_FC_Omega_d,
		totalTC_d, totalPartial_TC_X_d, totalPartial_TC_Omega_d,
		kc, hapticCollisionNum_d, triVertNum_d);

	hipDeviceSynchronize();
	printCudaError("HapticCollisionSphereMerged");
	return 0;
}

int runHapticCollisionCylinder_Merged_With_Sphere(float toolR, float param_toolLength, float p_collisionStiffness, float kc, int toolIdx, float sphere_R) {
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	float frictionStiffness = 10;
	// ��ײ���˺���
	hapticCollisionCylinder_Merge << <blockNum, threadNum >> > (
		toolPosePrev_d, toolPositionAndDirection_d,
		param_toolLength, toolR, sphere_R,
		triVertPos_d, triVertVelocity_d, triVert2TetVertMapping_d,
		triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, triVertInsertionDepth_d, triVertProjectedPos_d,
		tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
		triVertisCollide_d,
		triVertNum_d, p_collisionStiffness, frictionStiffness,
		triVertNonPenetrationDir_d, cylinderShift_d, hapticCollisionNum_d);

	//hipDeviceSynchronize();
	printCudaError("HapticCollisionCylinderMerged");
	return 0;
}

int runHapticCollisionCylinder_Merged(float toolR, float param_toolLength, float p_collisionStiffness, float kc, int toolIdx)
{
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	float frictionStiffness = 10;
	// ��ײ���˺���
	hapticCollisionCylinder_Merge << <blockNum, threadNum >> > (
		toolPosePrev_d, toolPositionAndDirection_d,
		param_toolLength, toolR, -1,
		triVertPos_d, triVertVelocity_d, triVert2TetVertMapping_d,
		triVertForce_d, triVertCollisionForce_d, triVertCollisionDiag_d, triVertInsertionDepth_d, triVertProjectedPos_d,
		tetVertForce_d, tetVertCollisionForce_d, tetVertCollisionDiag_d, tetVertInsertionDepth_d,
		triVertisCollide_d,
		triVertNum_d, p_collisionStiffness, frictionStiffness,
		triVertNonPenetrationDir_d, cylinderShift_d, hapticCollisionNum_d);

	//hipDeviceSynchronize();
	printCudaError("HapticCollisionCylinderMerged");
	return 0;
}

// sphere_R���Ϊ������Ϊû������ײ��...
__global__ void hapticCollisionCylinder_Merge(
	float* cylinderLastPos, float * cylinderPose,
	float halfLength, float radius, float sphere_r, float* triPositions,
	float* velocity, int* mapping, float* triForce,
	float* triCollisionForce, float* triCollisionDiag, float* triInsertionDepth, float* triVertProjectedPos, float* tetVertForce,
	float* tetVertCollisionForce, float* tetVertCollisionDiag, float* tetInsertionDepth, unsigned char* isCollide,
	int vertexNum,
	float collisionStiffness, float frictionStiffness, float* directDir,
	 float* cylinderShift, int* collisionNumPtr)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	//if(threadid==100)
	//	printf("dir: %f %f %f\n", cylinderPose[3], cylinderPose[4], cylinderPose[5]);
	float t = 0.0;
	float depth = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	int tetIdx0 = mapping[threadid * 2 + 0];
	int tetIdx0x = tetIdx0 * 3 + 0;
	int tetIdx0y = tetIdx0 * 3 + 1;
	int tetIdx0z = tetIdx0 * 3 + 2;
	int tetIdx1 = mapping[threadid * 2 + 1];
	int tetIdx1x = tetIdx1 * 3 + 0;
	int tetIdx1y = tetIdx1 * 3 + 1;
	int tetIdx1z = tetIdx1 * 3 + 2;

#ifdef OUTPUT_INFO
	if (threadid == 0)
	{
		printf("vert: %f %f %f cylinder pose: %f %f %f %f %f %f\n",
			tripositions[indexx], tripositions[indexy], tripositions[indexz],
			cylinderpose[0], cylinderpose[1], cylinderpose[2],
			cylinderpose[3], cylinderpose[4], cylinderpose[5]);
		printf("vert num: %d\n", vertexnum);
	}
#endif
	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}

	//ָ��������ײ���ķ���
	float moveDir[3];
	moveDir[0] = directDir[indexX];
	moveDir[1] = directDir[indexY];
	moveDir[2] = directDir[indexZ];

	float tetPosition[3] = { triPositions[indexX] ,triPositions[indexY] ,triPositions[indexZ] };
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPose[0],cylinderLastPos[1] - cylinderPose[1], cylinderLastPos[2] - cylinderPose[2] };
	float moveDistance = tetNormal_D(toolMoveDir);

	float ratio = 0.0f;
	float newPos[3];
	newPos[0] = cylinderPose[0] + cylinderShift[0] * ratio * radius;
	newPos[1] = cylinderPose[1] + cylinderShift[1] * ratio * radius;
	newPos[2] = cylinderPose[2] + cylinderShift[2] * ratio * radius;
	float w = moveDistance / radius;
	float enlarged_radius = radius * (1.5 - 0.5 / w);

	{
		// ���㶥����ײ���depth�Ͷ��㱻�ų������߱����λ��collisionPos
		float vert[3] = { triPositions[indexX], triPositions[indexY], triPositions[indexZ] };
		float distance = -1;
		bool collision = cylinderCollision_withDepth(cylinderPose, 
			vert, halfLength, radius, sphere_r,
			&t, &depth, &distance, collisionNormal, collisionPos);    // �ж�������Ƿ�������ײ. collisionPos�Ƿ�������ײ�󶥵�Ӧ��ȥ�ĵط�...
		if (!collision) return;
	}

	float deltaPos[3];
	triVertProjectedPos[indexX] = collisionPos[0];
	triVertProjectedPos[indexY] = collisionPos[1];
	triVertProjectedPos[indexZ] = collisionPos[2];
	deltaPos[0] = collisionPos[0] - triPositions[indexX];
	deltaPos[1] = collisionPos[1] - triPositions[indexY];
	deltaPos[2] = collisionPos[2] - triPositions[indexZ];
	float insertionDepth = sqrt(deltaPos[0] * deltaPos[0] + deltaPos[1] * deltaPos[1] + deltaPos[2] * deltaPos[2]);
	triInsertionDepth[threadid] = insertionDepth;    // �������..

	// ������ײ����Ӵ�����
	float temp[3];
	temp[0] = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
	temp[1] = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
	temp[2] = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);

	// �ѽӴ���ʩ�ӵ�������ײ�ı���������������������񶥵���
	triCollisionForce[indexX] += temp[0];
	triCollisionForce[indexY] += temp[1];
	triCollisionForce[indexZ] += temp[2];
	tetVertCollisionForce[tetIdx0x] += temp[0] / 2;
	tetVertCollisionForce[tetIdx0y] += temp[1] / 2;
	tetVertCollisionForce[tetIdx0z] += temp[2] / 2;
	tetVertCollisionForce[tetIdx1x] += temp[0] / 2;
	tetVertCollisionForce[tetIdx1y] += temp[1] / 2;
	tetVertCollisionForce[tetIdx1z] += temp[2] / 2;

	//����Խ�Ԫ�ض�Ӧ��ֵ
	float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
	float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
	float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
	triCollisionDiag[indexX] += diagx;
	triCollisionDiag[indexY] += diagy;
	triCollisionDiag[indexZ] += diagz;
	tetVertCollisionDiag[tetIdx0x] += diagx;
	tetVertCollisionDiag[tetIdx0y] += diagy;
	tetVertCollisionDiag[tetIdx0z] += diagz;
	tetVertCollisionDiag[tetIdx1x] += diagx;
	tetVertCollisionDiag[tetIdx1y] += diagy;
	tetVertCollisionDiag[tetIdx1z] += diagz;

	//���ñ�־λ
	isCollide[threadid] = 1;
	atomicAdd(collisionNumPtr, 1);
}
int runDeviceCalculateContact(float k_c)
{
	int  threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	
	CalculateContact<< <blockNum, threadNum >> > (triVertNonPenetrationDir_d, triVertPos_d,
		triVertProjectedPos_d, triVertInsertionDepth_d,
		toolPositionAndDirection_d, toolContactDeltaPos_triVert_d, triVertisCollide_d,
		totalFC_d, totalPartial_FC_X_d, totalPartial_FC_Omega_d,
		totalTC_d, totalPartial_TC_X_d, totalPartial_TC_Omega_d,
		k_c);
	printCudaError("runDeviceCalculateContact");
	hipDeviceSynchronize();
	return 0;
}
__global__ void CalculateContact(float* nonPenetrationDirection, float* triVertPosition, 
	float* projectedPosition, float* insertionDepth,
	float* toolPose, float* toolDeltaPos,
	unsigned char* isCollide, float* total_FC, float* totalPartial_FC_X,
	float* totalPartial_FC_Omega, float* total_TC, float* totalPartial_TC_X,
	float* totalPartial_TC_Omega, float k_c)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (isCollide[threadid] != 1) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float dDir[3] = { nonPenetrationDirection[indexX], nonPenetrationDirection[indexY], nonPenetrationDirection[indexZ] };
	
	float deltaPos[3] = { projectedPosition[indexX] - triVertPosition[indexX],projectedPosition[indexY] - triVertPosition[indexY], projectedPosition[indexZ] - triVertPosition[indexZ] };
	float d = -insertionDepth[threadid];
	float p[3] = { triVertPosition[indexX],triVertPosition[indexY],triVertPosition[indexZ] };
	float l = 1;
	float X_grasp[3] = { toolPose[0] + toolPose[3] * l,
		toolPose[1] + toolPose[4] * l,
		toolPose[2] + toolPose[5] * l };
	DeviceCalculateContact(dDir, k_c, d, p, X_grasp,
		toolDeltaPos,
		total_FC, totalPartial_FC_X, totalPartial_FC_Omega,
		total_TC, totalPartial_TC_X, totalPartial_TC_Omega, false);
}
__global__ void hapticCollisionSphere_Merge(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection,
	float* triForce, float* triCollisionForce, float* triCollisionDiag, float* triInsertionDepth,
	float* tetVertForce, float* tetVertCollisionForce, float* tetVertCollisionDiag, float* tetVertInsertionDepth,
	int* mapping,
	float collisionStiffness,
	float* toolDeltaPos, float* F_c, float* partialFc, float k_c, int* collisionNumPtr, int vertexNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid<10)

	int tetIdx0 = mapping[threadid * 2 + 0];
	int tetIdx0x = tetIdx0 * 3 + 0;
	int tetIdx0y = tetIdx0 * 3 + 1;
	int tetIdx0z = tetIdx0 * 3 + 2;
	int tetIdx1 = mapping[threadid * 2 + 1];
	int tetIdx1x = tetIdx1 * 3 + 0;
	int tetIdx1y = tetIdx1 * 3 + 1;
	int tetIdx1z = tetIdx1 * 3 + 2;

	float ori_radius = radius;
	radius *= 1.05;
	if (distance < radius)
	{
		//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid,
		//	ballPos[0], ballPos[1], ballPos[2],
		//	p[0], p[1], p[2], distance);
		triInsertionDepth[threadid] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x
		if (isnan(dDir[0]))
		{
			printf("threadid %d, nan in dDir, dealt as no collision, return\n", threadid);
			return;
		}
		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexX] += forcex;
		//triForce[indexY] += forcey;
		//triForce[indexZ] += forcez;
		triCollisionForce[indexX] += forcex;
		triCollisionForce[indexY] += forcey;
		triCollisionForce[indexZ] += forcez;
		tetVertCollisionForce[tetIdx0x] += forcex / 2;
		tetVertCollisionForce[tetIdx0y] += forcey / 2;
		tetVertCollisionForce[tetIdx0z] += forcez / 2;
		tetVertCollisionForce[tetIdx1x] += forcex / 2;
		tetVertCollisionForce[tetIdx1y] += forcey / 2;
		tetVertCollisionForce[tetIdx1z] += forcez / 2;

		float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		triCollisionDiag[indexX] += diagx;
		triCollisionDiag[indexY] += diagy;
		triCollisionDiag[indexZ] += diagz;
		tetVertCollisionDiag[tetIdx0x] += diagx;
		tetVertCollisionDiag[tetIdx0y] += diagy;
		tetVertCollisionDiag[tetIdx0z] += diagz;
		tetVertCollisionDiag[tetIdx1x] += diagx;
		tetVertCollisionDiag[tetIdx1y] += diagy;
		tetVertCollisionDiag[tetIdx1z] += diagz;

		if (threadid == LOOK_THREAD)
		{
			printf("threadid:%d force[%f %f %f] diag[%f %f %f]\n", threadid,
				triCollisionForce[indexX], triCollisionForce[indexY], triCollisionForce[indexZ],
				diagx, diagy, diagz);
		}
		float forceLen = sqrt(triForce[indexX] * triForce[indexX] + triForce[indexY] * triForce[indexY] + triForce[indexZ] * triForce[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], force:[%f,%f,%f]\n",
				threadid, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				triForce[indexX], triForce[indexY], triForce[indexZ]);
		}
		if (distance < ori_radius)
		{
			float partial_F_c[9] = { -dDir[0] * dDir[0] * k_c, -dDir[0] * dDir[1] * k_c, -dDir[0] * dDir[2] * k_c,
									 -dDir[0] * dDir[1] * k_c, -dDir[1] * dDir[1] * k_c, -dDir[1] * dDir[2] * k_c,
									 -dDir[0] * dDir[2] * k_c, -dDir[1] * dDir[2] * k_c, -dDir[2] * dDir[2] * k_c };
			// toolDeltaPos ���ߵ�����˶������붥����˶������෴
			atomicAdd(toolDeltaPos + 0, -deltaPos[0]);
			atomicAdd(toolDeltaPos + 1, -deltaPos[1]);
			atomicAdd(toolDeltaPos + 2, -deltaPos[2]);
			// F_c �빤�ߵ��˶�������ͬ
			atomicAdd(F_c + 0, -deltaPos[0] * k_c);
			atomicAdd(F_c + 1, -deltaPos[1] * k_c);
			atomicAdd(F_c + 2, -deltaPos[2] * k_c);
			atomicAdd(partialFc + 0, partial_F_c[0]);
			atomicAdd(partialFc + 1, partial_F_c[1]);
			atomicAdd(partialFc + 2, partial_F_c[2]);
			atomicAdd(partialFc + 3, partial_F_c[3]);
			atomicAdd(partialFc + 4, partial_F_c[4]);
			atomicAdd(partialFc + 5, partial_F_c[5]);
			atomicAdd(partialFc + 6, partial_F_c[6]);
			atomicAdd(partialFc + 7, partial_F_c[7]);
			atomicAdd(partialFc + 8, partial_F_c[8]);
			//printf("threadid %d dDir[%f %f %f] fc[%f %f %f]\n partialFc[\n%f %f %f\n%f %f %f\n%f %f %f]\n", threadid, 
			//	dDir[0], dDir[1], dDir[2],
			//	-deltaPos[0] * k_c, -deltaPos[1] * k_c, -deltaPos[2] * k_c,
			//	partial_F_c[0], partial_F_c[1], partial_F_c[2],
			//	partial_F_c[3], partial_F_c[4], partial_F_c[5], 
			//	partial_F_c[6], partial_F_c[7], partial_F_c[8]);
		}

		atomicAdd(collisionNumPtr, 1);

		//printf("thread%d dDir:[%f %f %f] F_c:[%f %f %f]\n", threadid, 
		//	dDir[0], dDir[1], dDir[2],
		//	- deltaPos[0] * k_c, -deltaPos[1] * k_c, -deltaPos[2] * k_c);
		isCollide[threadid] = 1;
		toolCollideFlag[toolIdx] = 1;
	}
	else
	{
		triInsertionDepth[threadid] = 0;
	}
	return;
}

__global__ void hapticCollisionSphere_Merge_with_Torque(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, 
	float * triForce, float * triCollisionForce, float * triCollisionDiag, float * triInsertionDepth,
	float* tetVertForce, float* tetVertCollisionForce, float* tetVertCollisionDiag, float* tetVertInsertionDepth, 
	int* mapping, 
	float collisionStiffness,
	float* toolDeltaPos, 
	float* total_FC, float* totalPartial_FC_X, float* totalPartial_FC_Omega,
	float* total_TC, float* totalPartial_TC_X, float* totalPartial_TC_Omega,
	float k_c, int* collisionNumPtr, int vertexNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid<10)

	int tetIdx0 = mapping[threadid * 2 + 0];
	int tetIdx0x = tetIdx0 * 3 + 0;
	int tetIdx0y = tetIdx0 * 3 + 1;
	int tetIdx0z = tetIdx0 * 3 + 2;
	int tetIdx1 = mapping[threadid * 2 + 1];
	int tetIdx1x = tetIdx1 * 3 + 0;
	int tetIdx1y = tetIdx1 * 3 + 1;
	int tetIdx1z = tetIdx1 * 3 + 2;

	float ori_radius = radius;
	radius *= 1.05;
	if (distance < radius)
	{
		//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid,
		//	ballPos[0], ballPos[1], ballPos[2],
		//	p[0], p[1], p[2], distance);
		triInsertionDepth[threadid] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x
		if (isnan(dDir[0]))
		{
			printf("threadid %d, nan in dDir, dealt as no collision, return\n", threadid);
			return;
		}
		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexX] += forcex;
		//triForce[indexY] += forcey;
		//triForce[indexZ] += forcez;
		triCollisionForce[indexX] += forcex;
		triCollisionForce[indexY] += forcey;
		triCollisionForce[indexZ] += forcez;
		tetVertCollisionForce[tetIdx0x] += forcex / 2;
		tetVertCollisionForce[tetIdx0y] += forcey / 2;
		tetVertCollisionForce[tetIdx0z] += forcez / 2;
		tetVertCollisionForce[tetIdx1x] += forcex / 2;
		tetVertCollisionForce[tetIdx1y] += forcey / 2;
		tetVertCollisionForce[tetIdx1z] += forcez / 2;

		float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		triCollisionDiag[indexX] += diagx;
		triCollisionDiag[indexY] += diagy;
		triCollisionDiag[indexZ] += diagz;
		tetVertCollisionDiag[tetIdx0x] += diagx;
		tetVertCollisionDiag[tetIdx0y] += diagy;
		tetVertCollisionDiag[tetIdx0z] += diagz;
		tetVertCollisionDiag[tetIdx1x] += diagx;
		tetVertCollisionDiag[tetIdx1y] += diagy;
		tetVertCollisionDiag[tetIdx1z] += diagz;

		if (threadid == LOOK_THREAD)
		{
			printf("threadid:%d force[%f %f %f] diag[%f %f %f]\n", threadid,
				triCollisionForce[indexX], triCollisionForce[indexY], triCollisionForce[indexZ],
				diagx, diagy, diagz);
		}
		float forceLen = sqrt(triForce[indexX] * triForce[indexX] + triForce[indexY] * triForce[indexY] + triForce[indexZ] * triForce[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("check insertion depth Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], triForce:[%f,%f,%f]\n",
				threadid, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				triForce[indexX], triForce[indexY], triForce[indexZ]);
		}
		if(distance<ori_radius)
		{
			bool verbose = true;
			//if (*collisionNumPtr == 0)
			//	verbose = true;
			DeviceCalculateContact(dDir, k_c, x, p, ballPos,
				toolDeltaPos, 
				total_FC, totalPartial_FC_X, totalPartial_FC_Omega,
				total_TC, totalPartial_TC_X, totalPartial_TC_Omega, verbose);
		}

		atomicAdd(collisionNumPtr, 1);

		//printf("thread%d dDir:[%f %f %f] F_c:[%f %f %f]\n", threadid, 
		//	dDir[0], dDir[1], dDir[2],
		//	- deltaPos[0] * k_c, -deltaPos[1] * k_c, -deltaPos[2] * k_c);
		isCollide[threadid] = 1;
		toolCollideFlag[toolIdx] = 1;
	}
	else
	{
		triInsertionDepth[threadid] = 0;
	}
	return;
}
__device__ void DeviceCalculateContact(
	float* dDir, float k_c, float depth, float* p, float * Xg_grasp,
	float* toolDeltaPos, 
	float* total_FC, float* totalPartial_FC_X, float* totalPartial_FC_Omega,
	float* total_TC, float* totalPartial_TC_X, 
	float* totalPartial_TC_Omega, bool printInfo)
{
	float point_F_c[3], point_TC[3];
	float r[3] = { p[0] - Xg_grasp[0],p[1] - Xg_grasp[1] ,p[2] - Xg_grasp[2] };

	float partialFCX[9], partialFCOmega[9];
	DeviceCalculateFC(k_c, depth, dDir, point_F_c, toolDeltaPos, total_FC);
	DeviceCalculatePartial_FC_X(dDir, k_c, partialFCX, totalPartial_FC_X);
	DeviceCalculatePartial_FC_Omega(dDir, r, k_c, depth, partialFCOmega, totalPartial_FC_Omega);

	float partialTCX[9], partialTCOmega[9];
	DeviceCalculateTC(point_F_c, r, point_TC, total_TC);
	DeviceCalculatePartial_TC_X(k_c, r, dDir, point_F_c, partialTCX, totalPartial_TC_X);
	DeviceCalculatePartial_TC_Omega(k_c, point_F_c, r, depth, dDir, partialTCOmega, totalPartial_TC_Omega);
	if (printInfo)
	{
		printf("FC: %f %f %f\n TC: %f %f %f\npartialFCX:\n%f %f %f\n%f %f %f\n%f %f %f\npartialFCOmega:\n%f %f %f\n%f %f %f\n%f %f %f\n", 
			point_F_c[0], point_F_c[1], point_F_c[2],
			point_TC[0], point_TC[1], point_TC[2],
			partialFCX[0], partialFCX[1], partialFCX[2],
			partialFCX[3], partialFCX[4], partialFCX[5],
			partialFCX[6], partialFCX[7], partialFCX[8],
			partialFCOmega[0], partialFCOmega[1], partialFCOmega[2],
			partialFCOmega[3], partialFCOmega[4], partialFCOmega[5],
			partialFCOmega[6], partialFCOmega[7], partialFCOmega[8]);
		//printf("partialTCX:\n%f %f %f\n%f %f %f\n%f %f %f\npartialTCOmega:\n%f %f %f\n%f %f %f\n%f %f %f\n----------------------------------\n",
		//	partialTCX[0], partialTCX[1], partialTCX[2],
		//	partialTCX[3], partialTCX[4], partialTCX[5],
		//	partialTCX[6], partialTCX[7], partialTCX[8],
		//	partialTCOmega[0], partialTCOmega[1], partialTCOmega[2],
		//	partialTCOmega[3], partialTCOmega[4], partialTCOmega[5],
		//	partialTCOmega[6], partialTCOmega[7], partialTCOmega[8]);
	}
}

__device__ void DeviceCalculateFC(
	float k_c, float* pointDeltaPos,
	float* F_c,
	float* toolDeltaPos, float * total_F_c)
{
	// deltaPos: ��������ײ��������λ�ơ�
	// ���ߵ����λ����deltaPos�෴
	atomicAdd(toolDeltaPos + 0, -pointDeltaPos[0]);
	atomicAdd(toolDeltaPos + 1, -pointDeltaPos[1]);
	atomicAdd(toolDeltaPos + 2, -pointDeltaPos[2]);

	// F_c ���㷽ʽ1�����ݶ���λ�Ƽ���Ӵ���
	F_c[0]= -pointDeltaPos[0] * k_c;
	F_c[1]= -pointDeltaPos[1] * k_c;
	F_c[2]= -pointDeltaPos[2] * k_c;

	// printf("calculate FC: pointDeltaPos[%f %f %f], kc:%f\n", pointDeltaPos[0], pointDeltaPos[1], pointDeltaPos[2], k_c);
	// F_c �빤�ߵ��˶�������ͬ
	atomicAdd(total_F_c + 0, F_c[0]);
	atomicAdd(total_F_c + 1, F_c[1]);
	atomicAdd(total_F_c + 2, F_c[2]);
}

__device__ void DeviceCalculateFC(
	float k_c, float d, float* dDir,
	float* F_c,
	float* toolDeltaPos, float* total_F_c)
{
	// deltaPos: ��������ײ��������λ�ơ�
	// ���ߵ����λ����deltaPos�෴
	float pointDeltaPos[3] = { dDir[0] * d, dDir[1] * d, dDir[2] * d };
	atomicAdd(toolDeltaPos + 0, -pointDeltaPos[0]);
	atomicAdd(toolDeltaPos + 1, -pointDeltaPos[1]);
	atomicAdd(toolDeltaPos + 2, -pointDeltaPos[2]);

	// F_c ���㷽ʽ2������Ƕ����ȺͶ���ָ����������Ӵ���
	F_c[0] = d * dDir[0] * k_c;
	F_c[1] = d * dDir[1] * k_c;
	F_c[2] = d * dDir[2] * k_c;
	// printf("calculate FC: pointDeltaPos[%f %f %f], kc:%f\n", pointDeltaPos[0], pointDeltaPos[1], pointDeltaPos[2], k_c);
	// F_c �빤�ߵ��˶�������ͬ
	atomicAdd(total_F_c + 0, F_c[0]);
	atomicAdd(total_F_c + 1, F_c[1]);
	atomicAdd(total_F_c + 2, F_c[2]);
}
__device__ void DeviceCalculatePartial_FC_X(
	float* dDir, float k_c, float* partialFCX, float * totalPartialFc)
{
	float partial_F_c[9] = { -dDir[0] * dDir[0] * k_c, -dDir[0] * dDir[1] * k_c, -dDir[0] * dDir[2] * k_c,
						 -dDir[0] * dDir[1] * k_c, -dDir[1] * dDir[1] * k_c, -dDir[1] * dDir[2] * k_c,
						 -dDir[0] * dDir[2] * k_c, -dDir[1] * dDir[2] * k_c, -dDir[2] * dDir[2] * k_c };
	partialFCX[0] = partial_F_c[0];
	partialFCX[1] = partial_F_c[1];
	partialFCX[2] = partial_F_c[2];
	partialFCX[3] = partial_F_c[3];
	partialFCX[4] = partial_F_c[4];
	partialFCX[5] = partial_F_c[5];
	partialFCX[6] = partial_F_c[6];
	partialFCX[7] = partial_F_c[7];
	partialFCX[8] = partial_F_c[8];
	// toolDeltaPos ���ߵ�����˶������붥����˶������෴
	atomicAdd(totalPartialFc + 0, partial_F_c[0]);
	atomicAdd(totalPartialFc + 1, partial_F_c[1]);
	atomicAdd(totalPartialFc + 2, partial_F_c[2]);
	atomicAdd(totalPartialFc + 3, partial_F_c[3]);
	atomicAdd(totalPartialFc + 4, partial_F_c[4]);
	atomicAdd(totalPartialFc + 5, partial_F_c[5]);
	atomicAdd(totalPartialFc + 6, partial_F_c[6]);
	atomicAdd(totalPartialFc + 7, partial_F_c[7]);
	atomicAdd(totalPartialFc + 8, partial_F_c[8]);
}

__device__ void DeviceCalculatePartial_FC_Omega(
	float* normal, float* r, float k_c, float depth,
	float* partial,
	float* totalPartial_FC_Omega
)
{
	float nnT[9], rTilde[9], nTilde[9], part0[9], part1[9];
	DeviceVec3MulVec3T(normal, normal, nnT);
	DeviceVec3toSkewSymmetricMatrix(r, rTilde);
	DeviceMat3MulMat3(nnT, rTilde, part0);
	DeviceScaleMulMat3(k_c, part0, part0);

	DeviceVec3toSkewSymmetricMatrix(normal, nTilde);
	DeviceScaleMulMat3(k_c * depth, nTilde, part1);

	DeviceMat3AddMat3(part0, part1, partial);
	
	DeviceMat3AtomicAddMat3(partial, totalPartial_FC_Omega);
}

__device__ void DeviceCalculatePartial_TC_X(
	float k_c, float* r, float* dDir, float* F_c,
	float* partialTCX,
	float* totalPartialTCX
)
{
	float nnt[9], r_tilde[9], part0[9], FC_tilde[9];
	DeviceVec3MulVec3T(dDir, dDir, nnt);
	DeviceVec3toSkewSymmetricMatrix(r, r_tilde);
	DeviceMat3MulMat3(r_tilde, nnt, part0);
	DeviceScaleMulMat3(-k_c, part0, part0);

	DeviceVec3toSkewSymmetricMatrix(F_c, FC_tilde);
	DeviceMat3AddMat3(part0, FC_tilde, partialTCX);

	DeviceMat3AtomicAddMat3(partialTCX, totalPartialTCX);
}

__device__ void DeviceCalculatePartial_TC_Omega(
	float k_c, float* F_c, float* r, float depth, float* dDir,
	float* partialTCOmega,
	float* totalPartialTCOmega)
{
	float part0[9], part1[9], part2[9], nnt[9];
	float FC_tilde[9], r_tilde[9], dDir_tilde[9];
	DeviceVec3toSkewSymmetricMatrix(r, r_tilde);
	DeviceVec3toSkewSymmetricMatrix(F_c, FC_tilde);
	DeviceMat3MulMat3(FC_tilde, r_tilde, part0);
	DeviceScaleMulMat3(-1, part0, part0);

	DeviceVec3MulVec3T(dDir, dDir, nnt);
	DeviceMat3MulMat3(nnt, r_tilde, part1);
	DeviceMat3MulMat3(r_tilde, part1, part1);
	DeviceScaleMulMat3(k_c, part1, part1);

	DeviceVec3toSkewSymmetricMatrix(dDir, dDir_tilde);
	DeviceMat3MulMat3(r_tilde, dDir_tilde, part2);
	DeviceScaleMulMat3(k_c * depth, part2, part2);

	DeviceMat3AddMat3(part0, part1, partialTCOmega);
	DeviceMat3AddMat3(part2, partialTCOmega, partialTCOmega);

	DeviceMat3AtomicAddMat3(partialTCOmega, totalPartialTCOmega);
}

// ���������о���
__device__ void DeviceVec3toSkewSymmetricMatrix(
	float* v, float* m)
{
	m[0] = 0; m[1] = -v[2]; m[2] = v[1];
	m[3] = v[2]; m[4] = 0; m[5] = -v[0];
	m[6] = -v[1]; m[7] = v[0]; m[8] = 0;
}
// ���������о���
__device__ void DeviceMatrixDotVec(
	float* m, float* v, float* result
)
{
	result[0] = m[0] * v[0] + m[1] * v[1] + m[2] * v[2];
	result[1] = m[3] * v[0] + m[4] * v[1] + m[5] * v[2];
	result[2] = m[6] * v[0] + m[7] * v[1] + m[8] * v[2];
	//printf("matrix:\n %f %f %f\n%f %f %f\n%f %f %f\nvector %f %f %f\nresult %f %f %f\n", m[0], m[1], m[2], m[3], m[4], m[5], m[6], m[7], m[8], v[0], v[1], v[2], result[0], result[1], result[2]);
}
// ���������о���
__device__ void DeviceMat3MulMat3(
	float* m0, float* m1, float* result)
{
	result[0] = m0[0] * m1[0] + m0[1] * m1[3] + m0[2] * m1[6];
	result[1] = m0[0] * m1[1] + m0[1] * m1[4] + m0[2] * m1[7];
	result[2] = m0[0] * m1[2] + m0[1] * m1[5] + m0[2] * m1[8];
	result[3] = m0[3] * m1[0] + m0[4] * m1[3] + m0[5] * m1[6];
	result[4] = m0[3] * m1[1] + m0[4] * m1[4] + m0[5] * m1[7];
	result[5] = m0[3] * m1[2] + m0[4] * m1[5] + m0[5] * m1[8];
	result[6] = m0[6] * m1[0] + m0[7] * m1[3] + m0[8] * m1[6];
	result[7] = m0[6] * m1[1] + m0[7] * m1[4] + m0[8] * m1[7];
	result[8] = m0[6] * m1[2] + m0[7] * m1[5] + m0[8] * m1[8];
}
// ���������о���result
__device__ void DeviceVec3MulVec3T(float* v0, float* v1, float* result)
{
	result[0] = v0[0] * v1[0];
	result[1] = v0[0] * v1[1];
	result[2] = v0[0] * v1[2];
	result[3] = v0[1] * v1[0];
	result[4] = v0[1] * v1[1];
	result[5] = v0[1] * v1[2];
	result[6] = v0[2] * v1[0];
	result[7] = v0[2] * v1[1];
	result[8] = v0[2] * v1[2];
	//printf("v0:"); PrintVec3(v0);
	//printf("v1:"); PrintVec3(v1);
	//printf("result:\n"); PrintMat3(result);
}

__device__ void PrintVec3(float* vec)
{
	printf("%f %f %f\n", vec[0], vec[1], vec[2]);
}
__device__ void PrintMat3(float* mat)
{
	printf("%f %f %f\n%f %f %f\n%f %f %f\n", mat[0], mat[1], mat[2],
		mat[3], mat[4], mat[5],
		mat[6], mat[7], mat[8]);
}
__device__ void DeviceScaleMulMat3(float s, float* m, float* result)
{
	result[0] = m[0] * s;
	result[1] = m[1] * s;
	result[2] = m[2] * s;
	result[3] = m[3] * s;
	result[4] = m[4] * s;
	result[5] = m[5] * s;
	result[6] = m[6] * s;
	result[7] = m[7] * s;
	result[8] = m[8] * s;
}

__device__ void DeviceMat3AddMat3(float* m0, float* m1, float* result)
{
	result[0] = m0[0] + m1[0];
	result[1] = m0[1] + m1[1];
	result[2] = m0[2] + m1[2];
	result[3] = m0[3] + m1[3];
	result[4] = m0[4] + m1[4];
	result[5] = m0[5] + m1[5];
	result[6] = m0[6] + m1[6];
	result[7] = m0[7] + m1[7];
	result[8] = m0[8] + m1[8];
}

__device__ void DeviceMat3AtomicAddMat3(float* m, float* result)
{
	atomicAdd(result + 0, m[0]);
	atomicAdd(result + 1, m[1]);
	atomicAdd(result + 2, m[2]);
	atomicAdd(result + 3, m[3]);
	atomicAdd(result + 4, m[4]);
	atomicAdd(result + 5, m[5]);
	atomicAdd(result + 6, m[6]);
	atomicAdd(result + 7, m[7]);
	atomicAdd(result + 8, m[8]);
}

__device__ void DeviceCalculateTC(
	float* F_c, float* r,
	float* point_TC,
	float* totalTC
)
{
	float r_tilde[9];
	DeviceVec3toSkewSymmetricMatrix(r, r_tilde);
	float TC[3];
	DeviceMatrixDotVec(r_tilde, F_c, TC);
	point_TC[0] = TC[0];
	point_TC[1] = TC[1];
	point_TC[2] = TC[2];
	atomicAdd(totalTC + 0, TC[0]);
	atomicAdd(totalTC + 1, TC[1]);
	atomicAdd(totalTC + 2, TC[2]);
}




__global__ void calculateCollisionSphere(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, float* force, float* collisionForce,
	float* collisionDiag, float* insertionDepth, float collisionStiffness,
	int* sortedTetVertIndices, int offset, int activeElementNum) 
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= activeElementNum) return;

	int tetVertIdx = sortedTetVertIndices[offset + threadid];
	int indexX = tetVertIdx * 3 + 0;
	int indexY = tetVertIdx * 3 + 1;
	int indexZ = tetVertIdx * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(tetVertIdx==93)
	//	printf("tetVertIdx %d ball[%f %f %f], p[%f %f %f] distance %f\n", tetVertIdx, 
	//		ballPos[0], ballPos[1], ballPos[2],
	//		p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		//printf("collided tetVertIdx:%d\n", tetVertIdx);
		if(tetVertIdx==64)
			printf("collided tetVertIdx:%d p[%f %f %f]\n", tetVertIdx, p[0], p[1], p[2]);
		insertionDepth[tetVertIdx] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexX] += forcex;
		//triForce[indexY] += forcey;
		//triForce[indexZ] += forcez;
		collisionForce[indexX] += forcex;
		collisionForce[indexY] += forcey;
		collisionForce[indexZ] += forcez;
		float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		collisionDiag[indexX] += diagx;
		collisionDiag[indexY] += diagy;
		collisionDiag[indexZ] += diagz;
		if (threadid == LOOK_THREAD)
		{
			//printf("threadid:%d triForce[%f %f %f] diag[%f %f %f]\n", threadid,
			//	triForce[indexX], triForce[indexY], triForce[indexZ],
			//	diagx, diagy, diagz);
		}

		isCollide[tetVertIdx] = 1;
		toolCollideFlag[toolIdx] = 1;

		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, tetVertIdx:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], triForce:[%f,%f,%f]\n",
				threadid, tetVertIdx, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				force[indexX], force[indexY], force[indexZ]);
		}

	}
	else
	{
		insertionDepth[tetVertIdx] = 0;
	}
	return;
}

__global__ void calculateCollisionSphereCluster(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, float* force, float* collisionForce,
	float* collisionDiag, float* insertionDepth, float collisionStiffness,
	int* tetIndex, int* tetVertRelatedTetInfo, int* tetVertRelatedTetIdx,
	int* sortedTetVertIndices, int offset, int activeElementNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= activeElementNum) return;

	int tetVertIdx = sortedTetVertIndices[offset + threadid];
	int indexX = tetVertIdx * 3 + 0;
	int indexY = tetVertIdx * 3 + 1;
	int indexZ = tetVertIdx * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(tetVertIdx==93)
	//	printf("tetVertIdx %d ball[%f %f %f], p[%f %f %f] distance %f\n", tetVertIdx, 
	//		ballPos[0], ballPos[1], ballPos[2],
	//		p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		//if(tetVertIdx==93)
		//	printf("collided tetVertIdx:%d p[%f %f %f]\n", tetVertIdx, p[0], p[1], p[2]);
		insertionDepth[tetVertIdx] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		collisionNormal[0] = collisionPos[0] - ballPos[0];
		collisionNormal[1] = collisionPos[1] - ballPos[1];
		collisionNormal[2] = collisionPos[2] - ballPos[2];
		float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		collisionNormal[0] /= col_len;
		collisionNormal[1] /= col_len;
		collisionNormal[2] /= col_len;

		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		//float forcex = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		//float forcey = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		//float forcez = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);

		//float diagx = collisionStiffness * collisionNormal[0] * collisionNormal[0];
		//float diagy = collisionStiffness * collisionNormal[1] * collisionNormal[1];
		//float diagz = collisionStiffness * collisionNormal[2] * collisionNormal[2];
		float forcex = collisionStiffness * deltaPos[0];
		float forcey = collisionStiffness * deltaPos[1];
		float forcez = collisionStiffness * deltaPos[2];
		float diagx = collisionStiffness * dDir[0];
		float diagy = collisionStiffness * dDir[1];
		float diagz = collisionStiffness * dDir[2];

		
		int tetInfoStart = tetVertRelatedTetInfo[tetVertIdx * 2 + 0];
		int relatedTetNum = tetVertRelatedTetInfo[tetVertIdx * 2 + 1];
		for (int i = 0; i < relatedTetNum; i++)
		{
			int idx = tetInfoStart + i;
			int tetIdx = tetVertRelatedTetIdx[idx];
			for (int v = 0; v < 4; v++)
			{
				int vIdx = tetIndex[tetIdx * 4 + v]; // ������ǰ������������ĳ�������嶥��ı��
				int iX = vIdx * 3 + 0;
				int iY = vIdx * 3 + 1;
				int iZ = vIdx * 3 + 2;
				atomicAdd(collisionForce + iX, forcex / (4 * relatedTetNum));
				atomicAdd(collisionForce + iY, forcey / (4 * relatedTetNum));
				atomicAdd(collisionForce + iZ, forcez / (4 * relatedTetNum));
				atomicAdd(collisionDiag + iX, diagx / (4 * relatedTetNum));
				atomicAdd(collisionDiag + iY, diagy / (4 * relatedTetNum));
				atomicAdd(collisionDiag + iZ, diagz / (4 * relatedTetNum));
			}
		}
		if (threadid == LOOK_THREAD)
		{
			//printf("threadid:%d triForce[%f %f %f] diag[%f %f %f]\n", threadid,
			//	triForce[indexX], triForce[indexY], triForce[indexZ],
			//	diagx, diagy, diagz);
		}

		isCollide[tetVertIdx] = 1;
		toolCollideFlag[toolIdx] = 1;

		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, tetVertIdx:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], triForce:[%f,%f,%f]\n",
				threadid, tetVertIdx, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				force[indexX], force[indexY], force[indexZ]);
		}

	}
	else
	{
		insertionDepth[tetVertIdx] = 0;
	}
	return;
}

__global__ void calculateVanillaCollisionSphere(float* ballPos, float radius,
	float* positions, unsigned char* isCollide, int toolIdx,
	unsigned char* toolCollideFlag, float* directDirection, float* force, float* collisionForce,
	float* collisionDiag, float* insertionDepth, float collisionStiffness,
	int* sortedTetVertIndices, int offset, int activeElementNum)
{
	// ��ײԼ�������Ǳ�ͶӰ���Ĺ��߱��淨�������������㹫ʽ��E=0.5w||p-q||_F^2
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= activeElementNum) return;

	int tetVertIdx = sortedTetVertIndices[offset + threadid];
	int indexX = tetVertIdx * 3 + 0;
	int indexY = tetVertIdx * 3 + 1;
	int indexZ = tetVertIdx * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid==93)
	//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid, 
	//	ballPos[0], ballPos[1], ballPos[2],
	//	p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		printf("collided tetVertIdx:%d\n", tetVertIdx);
		insertionDepth[tetVertIdx] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);

		// collisionPos = p+x*dDir
		float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		
		float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		float forcex = collisionStiffness * deltaPos[0];
		float forcey = collisionStiffness * deltaPos[1];
		float forcez = collisionStiffness * deltaPos[2];
		force[indexX] += forcex;
		force[indexY] += forcey;
		force[indexZ] += forcez;
		collisionForce[indexX] += forcex;
		collisionForce[indexY] += forcey;
		collisionForce[indexZ] += forcez;
		float diagx = 0.5*collisionStiffness*deltaPos[0];
		float diagy = 0.5*collisionStiffness*deltaPos[1];
		float diagz = 0.5*collisionStiffness*deltaPos[2];
		collisionDiag[indexX] += diagx;
		collisionDiag[indexY] += diagy;
		collisionDiag[indexZ] += diagz;

		isCollide[tetVertIdx] = 1;
		toolCollideFlag[toolIdx] = 1;

		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		if ((x1 > -1e-6) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
			printf("collision thread:%d, tetVertIdx:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], triForce:[%f,%f,%f]\n",
				threadid, tetVertIdx, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
				force[indexX], force[indexY], force[indexZ]);
		}

	}
	else
	{
		insertionDepth[tetVertIdx] = 0;
	}
	return;
}

__global__ void calculateCollisionSphereFollowDDir(float* ballPos, float radius,
	float* positions, unsigned char* isCollide,
	float* directDirection,
	float* force, float* collisionDiag, float* insertionDepth,
	float collisionStiffness, int vertexNum)
{
	// ����������嶥��֮������λ�ù�ϵ�����嶥��ָ����������ײ��⣬�Զ���ʩ��������ײԼ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("threadid:%d, vertexNum:%d\n", threadid, vertexNum);
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float d[3] = { p[0] - ballPos[0], p[1] - ballPos[1], p[2] - ballPos[2] };
	float d_square = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
	float distance = sqrt(d_square);
	//if(threadid<10)
	//printf("threadid %d ball[%f %f %f], p[%f %f %f] distance %f\n", threadid, 
	//	ballPos[0], ballPos[1], ballPos[2],
	//	p[0], p[1], p[2],distance);
	if (distance < radius)
	{
		insertionDepth[threadid] = radius - distance;
		// ����Χ�ڣ�����ָ����������������ײ���
		float dDir[3] = { directDirection[indexX], directDirection[indexY], directDirection[indexZ] };
		float collisionNormal[3] = { directDirection[indexX],directDirection[indexY] ,directDirection[indexZ] };
		// (d+x*directDir)^2==r^2 ��x

		float a = dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2];
		float b = 2 * (d[0] * dDir[0] + d[1] * dDir[1] + d[2] * dDir[2]);
		float c = d[0] * d[0] + d[1] * d[1] + d[2] * d[2] - radius * radius;
		float x0 = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		float x1 = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		float x = x1;
		//printf("dDir[%f,%f,%f],a:%f b:%f c:%f x0:%f x1:%f\n", dDir[0], dDir[1], dDir[2], a, b, c, x0, x1);
		if ((x1 > 0) && (x0 < 0))
		{
			x = x1;
		}
		else
		{
			printf("Error: x0=%f, x1=%f a:%f b:%f c:%f\n", x0, x1, a, b, c);
		}
		//// collisionPos = p+x*dDir
		//float collisionPos[3] = { p[0] + dDir[0] * x, p[1] + dDir[1] * x, p[2] + dDir[2] * x };
		//// calibrated collision normal(ʵ�����ǽ������������ĺ�����1.�������������� 2.����ָ������)
		//collisionNormal[0] = collisionPos[0] - ballPos[0];
		//collisionNormal[1] = collisionPos[1] - ballPos[1];
		//collisionNormal[2] = collisionPos[2] - ballPos[2];
		//float col_len = sqrt(collisionNormal[0] * collisionNormal[0] + collisionNormal[1] * collisionNormal[1] + collisionNormal[2] * collisionNormal[2]);

		//collisionNormal[0] /= col_len;
		//collisionNormal[1] /= col_len;
		//collisionNormal[2] /= col_len;

		//float deltaPos[3] = { x * dDir[0], x * dDir[1], x * dDir[2] };

		//triForce[indexX] += collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexY] += collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
		//triForce[indexZ] += collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
		//triCollisionDiag[indexX] += collisionStiffness * collisionNormal[0] * collisionNormal[0];
		//triCollisionDiag[indexY] += collisionStiffness * collisionNormal[1] * collisionNormal[1];
		//triCollisionDiag[indexZ] += collisionStiffness * collisionNormal[2] * collisionNormal[2];

		force[indexX] += collisionStiffness * dDir[0] * x;
		force[indexY] += collisionStiffness * dDir[1] * x;
		force[indexZ] += collisionStiffness * dDir[2] * x;
		collisionDiag[indexX] += collisionStiffness * dDir[0];
		collisionDiag[indexY] += collisionStiffness * dDir[1];
		collisionDiag[indexZ] += collisionStiffness * dDir[2];
		isCollide[threadid] = 1;

		float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);
		//printf("collision thread:%d, p:[%f,%f,%f], ball:[%f,%f,%f], x:%f dDir:[%f,%f,%f], triForce:[%f,%f,%f]\n",
		//	threadid, p[0], p[1], p[2], ballPos[0], ballPos[1], ballPos[2], x, dDir[0], dDir[1], dDir[2],
		//	triForce[indexX], triForce[indexY], triForce[indexZ]);
	}
	else
	{
		insertionDepth[threadid] = 0;
	}
	return;
}
int runcalculateCollisionSphereContinue(float* ball_pos, float* ball_pos_prev, float radius, float collisionStiffness, float adsorbStiffness, float frictionStiffness, bool useClusterCollision)
{
	float d[3] = { ball_pos[0] - ball_pos_prev[0], ball_pos[1] - ball_pos_prev[1], ball_pos[2] - ball_pos_prev[2] };
	float d_len = sqrt(d[0] * d[0] + d[1] * d[1] + d[2] * d[2]);
	if (d_len < 0.5)// λ�ƺ�С��������ɢ��ײ��⡣
	{
		//printf("collision stiffness:%f\n", collisionStiffness);
		runcalculateCollisionSphere(radius, collisionStiffness, 0, useClusterCollision);
	}
	else
	{
		printf("continue collision\n");
		// ��ǰʵ�ֲ�ʹ��toolShift����ʵ��¼��ײ��
		//runcalculateToolShift(d_len, radius, 0);
		runcalculateCollisionCylinder(d_len, radius, collisionStiffness, adsorbStiffness, frictionStiffness, 0);
	}
	return 0;
}


int runcalculateCollisionSphereContinueMU(float* ball_pos, float* ball_pos_prev, float radius, float collisionStiffness, float adsorbStiffness, float frictionStiffness)
{
	float d[3] = { ball_pos[0] - ball_pos_prev[0], ball_pos[1] - ball_pos_prev[1], ball_pos[2] - ball_pos_prev[2] };
	float d_len = sqrt(d[0] * d[0] + d[1] * d[1] + d[2] * d[2]);
	if (d_len < 0.5)// λ�ƺ�С��������ɢ��ײ��⡣
	{
		runcalculateCollisionSphereMU(radius, collisionStiffness, 0);
	}
	else
	{
		runcalculateToolShiftMU(d_len, radius, 0);
		runcalculateCollisionCylinderMU(d_len, radius, collisionStiffness, adsorbStiffness, frictionStiffness, 0);
	}
	return 0;
}