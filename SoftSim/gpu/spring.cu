#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"

//���㶥��ĳ��ٶ�
extern "C" int runcalculateSTMU(float damping, float dt) {

	//ÿ��block�е��߳���
	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	calculateSTMU << <blockNum, threadNum >> > (triVertPos_d, triVertPos_old_d, triVertPos_prev_d, triVertVelocity_d, triVertExternForce_d, 
		triVertFixed_d, triVertNum_d, gravityX_d, gravityY_d, gravityZ_d, damping, dt);

	hipDeviceSynchronize();
	printCudaError("runcalculateSTMU");
	return 0;
}


__global__ void calculateSTMU(float* positions, float* old_positions, float* prev_positions, float* velocity, 
	float* externForce, float* fixed, int vertexNum, float gravityX, float gravityY, float gravityZ, float damping, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float fixflag = fixed[threadid] > 1e8 ? 0 : 1;
	//�˶�������
	velocity[indexX] *= damping * fixflag;
	velocity[indexY] *= damping * fixflag;
	velocity[indexZ] *= damping * fixflag;
	//ʩ������
	velocity[indexX] += gravityX * dt * fixflag;
	velocity[indexY] += gravityY * dt * fixflag;
	velocity[indexZ] += gravityZ * dt * fixflag;
	//ʩ����������
	velocity[indexX] += externForce[indexX] * dt * fixflag;
	velocity[indexY] += externForce[indexY] * dt * fixflag;
	velocity[indexZ] += externForce[indexZ] * dt * fixflag;

	positions[indexX] += velocity[indexX] * dt * fixflag;
	positions[indexY] += velocity[indexY] * dt * fixflag;
	positions[indexZ] += velocity[indexZ] * dt * fixflag;


	//st
	old_positions[indexX] = positions[indexX];
	old_positions[indexY] = positions[indexY];
	old_positions[indexZ] = positions[indexZ];
	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	//��������
	externForce[indexX] = 0;
	externForce[indexY] = 0;
	externForce[indexZ] = 0;
}


extern "C" int runClearCollisionMU() {
	hipMemset(triVertForce_d, 0.0f, triVertNum_d * 3 * sizeof(float));
	hipMemset(triVertisCollide_d, 0, triVertNum_d * sizeof(unsigned char));
	hipMemset(triVertCollisionDiag_d, 0.0f, triVertNum_d * 3 * sizeof(float));
	hipMemset(triVertCollisionForce_d, 0.0f, triVertNum_d * 3 * sizeof(float));
	hipMemset(triVertInsertionDepth_d, 0.0f, triVertNum_d * sizeof(float));
	printCudaError("runClearCollisionMU");
	return 0;
}

//���㶥�������
extern "C" int runcalculateIFMU() {
	int threadNum = 512;
	int blockNum = (triEdgeNum_d + threadNum - 1) / threadNum;

	//printf("spring number: %d\n", triEdgeNum_d);
	calculateIFMU << <blockNum, threadNum >> > (triVertPos_d, triVertForce_d, // vert number
		triEdgeStiffness_d, triEdgeOrgLength_d, triEdgeIndex_d, //spring number
		triVertFixed_d,
		triEdgeNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateIFMU");
	return 0;
}

__global__ void calculateIFMU(float* positions, float* force, 
	float* springStiffness, float* springOrigin, unsigned int* springIndex, 
	float* triVertFixed,
	int springNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("in calculateIFMU\n");
	if (threadid >= springNum) return;

	int vIndex0 = springIndex[threadid * 2 + 0];
	int vIndex1 = springIndex[threadid * 2 + 1];

	//printf("threadid:%d, spring index:%d %d\n", threadid, vIndex0, vIndex1);
	//��ȡ�����������local��
	float pos0x = positions[vIndex0 * 3 + 0];
	float pos0y = positions[vIndex0 * 3 + 1];
	float pos0z = positions[vIndex0 * 3 + 2];
	float pos1x = positions[vIndex1 * 3 + 0];
	float pos1y = positions[vIndex1 * 3 + 1];
	float pos1z = positions[vIndex1 * 3 + 2];

#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculateIFMU p0[%f %f %f] p1[%f %f %f]\n",
			pos0x, pos0y, pos0z, pos1x, pos1y, pos1z);
	}
#endif

	//����local��d
	float dx = pos0x - pos1x;
	float dy = pos0y - pos1y;
	float dz = pos0z - pos1z;

	float length = sqrt(dx * dx + dy * dy + dz * dz);
	if (length < springOrigin[threadid]) return;
	dx = dx * (springOrigin[threadid] / length);
	dy = dy * (springOrigin[threadid] / length);
	dz = dz * (springOrigin[threadid] / length);

	//��Ӧ�������˵������
	//����Ӧ����Ҫԭ�Ӳ���
	float tempx = dx - pos0x + pos1x;
	float tempy = dy - pos0y + pos1y;
	float tempz = dz - pos0z + pos1z;

	// �ѵ��������ڵ������˵Ķ�����
	atomicAdd(force + vIndex0 * 3 + 0, tempx * springStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 1, tempy * springStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 2, tempz * springStiffness[threadid]);

	atomicAdd(force + vIndex1 * 3 + 0, -tempx * springStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 1, -tempy * springStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 2, -tempz * springStiffness[threadid]);
}

__global__ void calculateIFMU(float* positions, float* force, 
	float* springStiffness, float* springOrigin, unsigned int* springIndex, 
	int* sortedSpringIndices, int offset, int activeElementNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("in calculateIFMU\n");
	if (threadid >= activeElementNum) return;

	int springIdx = sortedSpringIndices[offset + threadid];

	int vIndex0 = springIndex[springIdx * 2 + 0];
	int vIndex1 = springIndex[springIdx * 2 + 1];


	//��ȡ�����������local��
	float pos0x = positions[vIndex0 * 3 + 0];
	float pos0y = positions[vIndex0 * 3 + 1];
	float pos0z = positions[vIndex0 * 3 + 2];
	float pos1x = positions[vIndex1 * 3 + 0];
	float pos1y = positions[vIndex1 * 3 + 1];
	float pos1z = positions[vIndex1 * 3 + 2];


#ifdef OUTPUT_INFO
	if (springIdx == LOOK_THREAD)
	{
		printf("calculateIFMU p0[%f %f %f] p1[%f %f %f]\n",
			pos0x, pos0y, pos0z, pos1x, pos1y, pos1z);
	}
	if (threadid == 0)
	{
		printf("springIdx:%d, vert index:%d %d pos1[%f %f %f] pos2[%f %f %f]\n", springIdx, vIndex0, vIndex1,
			pos0x, pos0y, pos0z, pos1x, pos1y, pos1z);
	}
#endif

	//����local��d
	float dx = pos0x - pos1x;
	float dy = pos0y - pos1y;
	float dz = pos0z - pos1z;

	float length = sqrt(dx * dx + dy * dy + dz * dz);
	if (length < springOrigin[springIdx]) return;
	dx = dx * (springOrigin[springIdx] / length);
	dy = dy * (springOrigin[springIdx] / length);
	dz = dz * (springOrigin[springIdx] / length);

	//��Ӧ�������˵������
	//����Ӧ����Ҫԭ�Ӳ���
	float tempx = dx - pos0x + pos1x;
	float tempy = dy - pos0y + pos1y;
	float tempz = dz - pos0z + pos1z;

	//if (vIndex1 == 72990)
	//{
	//	float k = springStiffness[springIdx];
	//	printf("springIdx:%d, vert 72990 spring force[%f %f %f] stiffness:%f\n", springIdx, k * tempx, k * tempy, k * tempz, springStiffness[springIdx]);
	//}
	atomicAdd(force + vIndex0 * 3 + 0, tempx * springStiffness[springIdx]);
	atomicAdd(force + vIndex0 * 3 + 1, tempy * springStiffness[springIdx]);
	atomicAdd(force + vIndex0 * 3 + 2, tempz * springStiffness[springIdx]);

	atomicAdd(force + vIndex1 * 3 + 0, -tempx * springStiffness[springIdx]);
	atomicAdd(force + vIndex1 * 3 + 1, -tempy * springStiffness[springIdx]);
	atomicAdd(force + vIndex1 * 3 + 2, -tempz * springStiffness[springIdx]);
}

int runcalculateRestPosForceWithTetPos(float toolRadius)
{

	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	// ���ݾ����Զ�����restpos stiffness�� ��Ҫ��һ����������
	calculateRestStiffnessWithTet << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d, 
		triVertPos_d, triVertisCollide_d, triVertfromTetStiffness_d, 
		cylinderNum_d, triVertNum_d);
	calculateRestPosWithTetPosMU << <blockNum, threadNum >> > (triVertPos_d, triVert2TetVertMapping_d,
		triVertForce_d, triVertCollisionDiag_d,
		tetVertPos_d, triVertfromTetStiffness_d,
		triVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateRestPosForceWithTetPos");
	return 0;
}

__global__ void calculateRestStiffnessWithTet(float* ballPos, unsigned char* toolCollideFlag, float* positions, unsigned char* isCollide, float* restStiffness, int toolNum, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float base_stiffness = 100;  //���Ƹն�ϵ���Ĵ�С��Χ
	float max_stiffness = 110;

	bool flag = false;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0)
			flag = true;
	}
	if (flag) { //����͹��߷�����ײ 
		switch (isCollide[threadid])
		{
		case 1: {  //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ��㣬���ఴ���Լ�������ȥ���Σ��ն�ϵ����С
			//restStiffness[threadid] = 1.0;
			restStiffness[threadid] = base_stiffness;
#//ifdef OUTPUT_INFO
			if (threadid == LOOK_THREAD)
				printf("mesh reststiffness in thread:%d: %f\n", threadid, restStiffness[LOOK_THREAD]);
#//endif
			break;
		}
		case 2: { //��ȡ�㣬�͹���ֱ����ײ�Ķ��㣬Ϊ���ܸ���ץǯ�ƶ����ն�ϵ���ϴ�
			restStiffness[threadid] = 1 * max_stiffness;
			break;
		}
		case 0: { //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
			float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
			int indexX = threadid * 3 + 0;
			int indexY = threadid * 3 + 1;
			int indexZ = threadid * 3 + 2;
			float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
			for (int i = 0; i < toolNum; i++)
			{
				float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
				float d = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
				if (d < distance) distance = d;
			}

			float k;
			//k = 10.0 / (1 + exp(-distance + 2));
			//k = distance + 3.0;
			//k = 0.6 / (exp2(0.3 * (-distance + 0.3)));  //����ģ��
			k = 0.6 / (exp2(0.3 * (-distance + 0.3)));
			restStiffness[threadid] = k * base_stiffness;
			if (restStiffness[threadid] > max_stiffness) restStiffness[threadid] = max_stiffness;
		}
		default:
			break;
		}
	}
	else {   //���û�к͹��߷�����ײ�������������ƶ����ն�ϵ����Ϊ���
		restStiffness[threadid] = max_stiffness;
	}
}

__global__ void calculateRestStiffnessWithTet(float* ballPos, float toolRadius,
	unsigned char* toolCollideFlag, float* positions,
	unsigned char* isCollide, float* restStiffness,
	int toolNum, int* sortedIndices, int offset, int activeElementNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[offset + threadid];

	float base_stiffness = 0.7;  //���Ƹն�ϵ���Ĵ�С��Χ
	float max_stiffness = 1;

	bool flag = false;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0)
			flag = true;
	}
	if (flag) { //����͹��߷�����ײ 
		switch (isCollide[vertIdx])
		{
		case 1: {  //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ��㣬���ఴ���Լ�������ȥ���Σ��ն�ϵ����С
			//restStiffness[vertIdx] = 1.0;
			restStiffness[vertIdx] = 0.0;
#//ifdef OUTPUT_INFO
			if (vertIdx == LOOK_THREAD)
				printf("mesh reststiffness in thread:%d: %f\n", vertIdx, restStiffness[LOOK_THREAD]);
#//endif
			break;
		}
		case 2: { //��ȡ�㣬�͹���ֱ����ײ�Ķ��㣬Ϊ���ܸ���ץǯ�ƶ����ն�ϵ���ϴ�
			restStiffness[vertIdx] = 1 * max_stiffness;
			break;
		}
		case 0: { //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
			float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
			int indexX = vertIdx * 3 + 0;
			int indexY = vertIdx * 3 + 1;
			int indexZ = vertIdx * 3 + 2;
			float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
			for (int i = 0; i < toolNum; i++)
			{
				float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
				float d = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
				if (d < distance) distance = d;
			}

			float k;
			//k = 10.0 / (1 + exp(-distance + 2));
			//k = distance + 3.0;
			//k = 0.6 / (exp2(0.3 * (-distance + 0.3)));  //����ģ��

			//restStiffness[vertIdx] = max_stiffness;

			//float x = distance - 1*toolRadius;
			//float ratio = 2 / (1 + exp(-x)) - 1;
			//restStiffness[vertIdx] = ratio * max_stiffness;

			//float x = distance - 2 * toolRadius;
			//float influence_r = 2*toolRadius;
			//if (x > influence_r)
			//	x = influence_r;
			//else if (x < 0)
			//	x = 0;

			//float t = x / influence_r;
			//restStiffness[vertIdx] = t*t*t * max_stiffness;

			k = 0.6 / (exp2(0.3 * (-distance + toolRadius)));
			restStiffness[threadid] = k * base_stiffness;
			if (restStiffness[threadid] > max_stiffness) restStiffness[threadid] = max_stiffness;

		}
		default:
			break;
		}
	}
	else {   //���û�к͹��߷�����ײ�������������ƶ����ն�ϵ����Ϊ���
		restStiffness[vertIdx] = max_stiffness;
	}
	restStiffness[vertIdx] = max_stiffness;
}

__global__ void calculateRestPosWithTetPosMU(float* positions, int* skeletonIndex, float* force, float* collisionDiag, 
	float* rest_positions, float* restStiffness, int vertexNum) {
	/*˵����skeletonIndex��СΪ�������Ƕ�������x2
		ÿһ�������������񶥵��Ӧ������������ʾ����󶨵������嶥���±�
		����ڶ�������Ϊ-1�������������ֱ�Ӷ�Ӧ�ŵ�һ��������ʾ�������嶥��
		������������������񶥵����ڱ�������ϸ�ֹ��������ӵĶ��㣬restpos��Ӧ���������嶥����е㡣
	*/
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int tet_idx1 = skeletonIndex[2 * threadid + 0]; // �����嶥��1�±�
	int tet_idx2 = skeletonIndex[2 * threadid + 1]; // �����嶥��2�±�

	//��ȡ��������
	float tri_pos0x = positions[3 * threadid + 0]; // ���涥������
	float tri_pos0y = positions[3 * threadid + 1];
	float tri_pos0z = positions[3 * threadid + 2];
	//��������
	float tempx = (rest_positions[3 * tet_idx1 + 0] + rest_positions[3 * tet_idx2 + 0]) * 0.5f - tri_pos0x;
	float tempy = (rest_positions[3 * tet_idx1 + 1] + rest_positions[3 * tet_idx2 + 1]) * 0.5f - tri_pos0y;
	float tempz = (rest_positions[3 * tet_idx1 + 2] + rest_positions[3 * tet_idx2 + 2]) * 0.5f - tri_pos0z;

#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		float tet0_x = rest_positions[3 * tet_idx1 + 0];
		float tet0_y = rest_positions[3 * tet_idx1 + 1];
		float tet0_z = rest_positions[3 * tet_idx1 + 2];

		float tet1_x = rest_positions[3 * tet_idx2 + 0];
		float tet1_y = rest_positions[3 * tet_idx2 + 1];
		float tet1_z = rest_positions[3 * tet_idx2 + 2];

		//printf("threadid:%d, vertexRestForce:[%f %f %f]\n", threadid, tempx * springStiffness, tempy * springStiffness, tempz * springStiffness);
		printf("calculateIFRestMUDefaultStiffness threadid:%d, springStiffness:%f tetidx0:%d tetidx1:%d temp[%f %f %f]\n", threadid, restStiffness[threadid], tet_idx1, tet_idx2, tempx, tempy, tempz);
		printf("\t tet0[%f %f %f] tet1[%f %f %f] tri[%f %f %f]\n",
			tet0_x, tet0_y, tet0_z,
			tet1_x, tet1_y, tet1_z,
			tri_pos0x, tri_pos0y, tri_pos0z);
	}
#endif
	atomicAdd(force + threadid * 3 + 0, tempx * restStiffness[threadid]);
	atomicAdd(force + threadid * 3 + 1, tempy * restStiffness[threadid]);
	atomicAdd(force + threadid * 3 + 2, tempz * restStiffness[threadid]);

	atomicAdd(collisionDiag + threadid * 3 + 0, restStiffness[threadid]);
	atomicAdd(collisionDiag + threadid * 3 + 1, restStiffness[threadid]);
	atomicAdd(collisionDiag + threadid * 3 + 2, restStiffness[threadid]);
}

__global__ void calculateRestPosWithTetPosMU(float* positions, int* skeletonIndex, float* force, float* collisionDiag,
	float* rest_positions, float* restStiffness, 
	int* sortedIndices, int offset, int activeElementNum) 
{
	/*˵����skeletonIndex��СΪ�������Ƕ�������x2
		ÿһ�������������񶥵��Ӧ������������ʾ����󶨵������嶥���±�
		����ڶ�������Ϊ-1�������������ֱ�Ӷ�Ӧ�ŵ�һ��������ʾ�������嶥��
		������������������񶥵����ڱ�������ϸ�ֹ��������ӵĶ��㣬restpos��Ӧ���������嶥����е㡣
	*/
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[offset + threadid];

	int tet_idx1 = skeletonIndex[2 * vertIdx + 0]; // �����嶥��1�±�
	int tet_idx2 = skeletonIndex[2 * vertIdx + 1]; // �����嶥��2�±�

	//��ȡ��������
	float tri_pos0x = positions[3 * vertIdx + 0]; // ���涥������
	float tri_pos0y = positions[3 * vertIdx + 1];
	float tri_pos0z = positions[3 * vertIdx + 2];
	//��������
	float tempx = (rest_positions[3 * tet_idx1 + 0] + rest_positions[3 * tet_idx2 + 0]) * 0.5f - tri_pos0x;
	float tempy = (rest_positions[3 * tet_idx1 + 1] + rest_positions[3 * tet_idx2 + 1]) * 0.5f - tri_pos0y;
	float tempz = (rest_positions[3 * tet_idx1 + 2] + rest_positions[3 * tet_idx2 + 2]) * 0.5f - tri_pos0z;

#ifdef OUTPUT_INFO
	if (vertIdx == LOOK_THREAD)
	{
		float tet0_x = rest_positions[3 * tet_idx1 + 0];
		float tet0_y = rest_positions[3 * tet_idx1 + 1];
		float tet0_z = rest_positions[3 * tet_idx1 + 2];

		float tet1_x = rest_positions[3 * tet_idx2 + 0];
		float tet1_y = rest_positions[3 * tet_idx2 + 1];
		float tet1_z = rest_positions[3 * tet_idx2 + 2];

		//printf("vertIdx:%d, vertexRestForce:[%f %f %f]\n", vertIdx, tempx * springStiffness, tempy * springStiffness, tempz * springStiffness);
		printf("calculateIFRestMUDefaultStiffness vertIdx:%d, springStiffness:%f tetidx0:%d tetidx1:%d temp[%f %f %f]\n", vertIdx, restStiffness[vertIdx], tet_idx1, tet_idx2, tempx, tempy, tempz);
		printf("\t tet0[%f %f %f] tet1[%f %f %f] tri[%f %f %f]\n",
			tet0_x, tet0_y, tet0_z,
			tet1_x, tet1_y, tet1_z,
			tri_pos0x, tri_pos0y, tri_pos0z);
	}
#endif
	atomicAdd(force + vertIdx * 3 + 0, tempx * restStiffness[vertIdx]);
	atomicAdd(force + vertIdx * 3 + 1, tempy * restStiffness[vertIdx]);
	atomicAdd(force + vertIdx * 3 + 2, tempz * restStiffness[vertIdx]);

	atomicAdd(collisionDiag + vertIdx * 3 + 0, restStiffness[vertIdx]);
	atomicAdd(collisionDiag + vertIdx * 3 + 1, restStiffness[vertIdx]);
	atomicAdd(collisionDiag + vertIdx * 3 + 2, restStiffness[vertIdx]);
}

///����ÿ�������restposԼ��
int runcalculateRestPosMU() {
	int  threadNum = 512;
	int blockNum = (tetNum_d + threadNum - 1) / threadNum;
	calculateRestPosStiffness << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d, tetVertPos_d, tetVertisCollide_d, tetVertRestStiffness_d, 1, tetVertNum_d
		);
	calculateRestPos << <blockNum, threadNum >> > (
		tetVertPos_d, tetVertRestPos_d,
		tetVertCollisionForce_d, tetVertCollisionDiag_d,
		tetVertRestStiffness_d, tetVertNum_d);

	hipDeviceSynchronize();
	printCudaError("runcalculateRestPos");
	return 0;
}

//�б�ѩ�����λ��
extern "C" int runcalculatePosMU(float omega, float dt) {

	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	//���м���
	
	calculatePOSMU << <blockNum, threadNum >> > (triVertPos_d,
		triVertForce_d, triVertFixed_d, triVertMass_d,
		triVertPos_next_d, triVertPos_prev_d, triVertPos_old_d,
		triEdgeDiag_d, triVertCollisionDiag_d, triVertCollisionForce_d,
		triVertNum_d, dt, omega);

	hipDeviceSynchronize();
	printCudaError("runcalculatePosMU");
	return 0;
}


__global__ void calculatePOSMU(float* positions, float* force, float* fixed, float* mass, float* next_positions, float* prev_positions, float* old_positions, float* springDiag, float* collisionDiag, float* collisionForce, int vertexNum, float dt, float omega) {

	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	if (fixed[threadid] > 1e8) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	float diagConstant = (mass[threadid] + fixed[threadid]) / (dt * dt);

	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];

#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculatePOSMU springDiag:%f collisionDiag:[%f %f %f] constantDiag:%f\n",
			springDiag[threadid], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ], diagConstant);
	}
#endif
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (springDiag[threadid] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (springDiag[threadid] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (springDiag[threadid] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];

	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];
}

__global__ void calculatePOSMU(float* positions, float* force, float* fixed, float* mass, float* next_positions, float* prev_positions,
	float* old_positions, float* springDiag, float* collisionDiag, float* collisionForce,
	int* sortedIndices, int offset, int activeElementNum,
	float dt, float omega)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;
	
	int vertIdx = sortedIndices[offset + threadid];
	int indexX = vertIdx * 3 + 0;
	int indexY = vertIdx * 3 + 1;
	int indexZ = vertIdx * 3 + 2;

	float diagConstant = (mass[vertIdx] + fixed[vertIdx]) / (dt * dt);
	//if (vertIdx == LOOK_THREAD)
	//	printf("calculatePOSMU vertIdx:0, diagConstant:%d\n", diagConstant);
	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];
	//if (vertIdx == LOOK_THREAD)
	//	printf("elements:[%f %f %f]\n", elementX, elementY, elementZ);

#ifdef OUTPUT_INFO
	if (vertIdx == LOOK_THREAD)
	{
		printf("calculatePOSMU springDiag:%f collisionDiag:[%f %f %f] constantDiag:%f\n",
			springDiag[vertIdx], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ], diagConstant);
	}
#endif
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (springDiag[vertIdx] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (springDiag[vertIdx] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (springDiag[vertIdx] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];

	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];
}
//�����ٶ�
extern "C" int runcalculateVMU(float dt) {

	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	
	calculateVMU << <blockNum, threadNum >> > (triVertPos_d, triVertVelocity_d, triVertPos_old_d, triVertNum_d, dt);

	printCudaError("runcalculateVMU");
	return 0;
}

__global__ void calculateVMU(float* positions, float* velocity, float* old_positions, int vertexNum, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	velocity[threadid * 3 + 0] += (positions[threadid * 3 + 0] - old_positions[threadid * 3 + 0]) / dt;
	velocity[threadid * 3 + 1] += (positions[threadid * 3 + 1] - old_positions[threadid * 3 + 1]) / dt;
	velocity[threadid * 3 + 2] += (positions[threadid * 3 + 2] - old_positions[threadid * 3 + 2]) / dt;
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculateVMU v[%f %f %f]\n",
			(positions[threadid * 3 + 0] - old_positions[threadid * 3 + 0]) / dt,
			(positions[threadid * 3 + 1] - old_positions[threadid * 3 + 1]) / dt,
			(positions[threadid * 3 + 2] - old_positions[threadid * 3 + 2]) / dt);
	}
#endif // OUTPUT_INFO

}

__global__ void calculateVMU(float* positions, float* velocity, float* old_positions, 
	int* sortedIndices, int offset, int activeElementNum,
	float dt) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[offset + threadid];
	velocity[vertIdx * 3 + 0] += (positions[vertIdx * 3 + 0] - old_positions[vertIdx * 3 + 0]) / dt;
	velocity[vertIdx * 3 + 1] += (positions[vertIdx * 3 + 1] - old_positions[vertIdx * 3 + 1]) / dt;
	velocity[vertIdx * 3 + 2] += (positions[vertIdx * 3 + 2] - old_positions[vertIdx * 3 + 2]) / dt;
#ifdef OUTPUT_INFO
	if (vertIdx == LOOK_THREAD)
	{
		printf("calculateVMU v[%f %f %f]\n",
			(positions[vertIdx * 3 + 0] - old_positions[vertIdx * 3 + 0]) / dt,
			(positions[vertIdx * 3 + 1] - old_positions[vertIdx * 3 + 1]) / dt,
			(positions[vertIdx * 3 + 2] - old_positions[vertIdx * 3 + 2]) / dt);
	}
#endif // OUTPUT_INFO

}

