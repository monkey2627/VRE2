#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"

// triangle
unsigned int* triIndex_d;		  // �������������������������Ӧ�Ķ����±꣬3*triNum_d

__device__ float l2len(float* v0, float* v1)
{
	float d[3] = { v0[0] - v1[0],v0[1] - v1[1] ,v0[2] - v1[2] };
	return sqrt(d[0] * d[0] + d[1] * d[1] + d[2] * d[2]);
}
//����mesh���㷨��
extern "C" int runUpdateMeshNormalMU() {

	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	//������㷨����Ϣ�����¼���
	clearNormalMU << <blockNum, threadNum >> > (triVertNorm_d, triVertNormAccu_d, triVertNum_d);

	threadNum = 512;
	blockNum = (triNum_d + threadNum - 1) / threadNum;
	updateMeshNormalMU << <blockNum, threadNum >> > (triVertPos_d, triVertNorm_d, triVertNormAccu_d, triIndex_d, triNum_d);
	hipDeviceSynchronize();
	printCudaError("updateMeshNormalMU");

	threadNum = 512;
	blockNum = (triVertNum_d + threadNum - 1) / threadNum;
	normalizeMeshtriVertNorm_debug << <blockNum, threadNum >> > (triVertNorm_d, triVertPos_d, triVertNormAccu_d, triVertNum_d);

	hipDeviceSynchronize();
	printCudaError("normalizeMeshtriVertNorm");
	return 0;
}

//������㷨����Ϣ����Ҫ���¼���
__global__ void clearNormalMU(float* meshNormal, float* totAngle, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	meshNormal[3 * threadid + 0] = 0.0f;
	meshNormal[3 * threadid + 1] = 0.0f;
	meshNormal[3 * threadid + 2] = 0.0f;

	totAngle[threadid] = 0.0f;
}

//���ݶ���λ�ú�������Ƭ������Ƭ����
__global__ void updateMeshNormalMU(float* meshPosition, float* meshNormal, float* totAngle, unsigned int* meshTriangle, int meshTriangleNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= meshTriangleNum) return;

	//��ȡmesh�����ε��������������
	unsigned int index0 = meshTriangle[threadid * 3 + 0];
	unsigned int index1 = meshTriangle[threadid * 3 + 1];
	unsigned int index2 = meshTriangle[threadid * 3 + 2];

	float vecAx = meshPosition[index1 * 3 + 0] - meshPosition[index0 * 3 + 0];
	float vecAy = meshPosition[index1 * 3 + 1] - meshPosition[index0 * 3 + 1];
	float vecAz = meshPosition[index1 * 3 + 2] - meshPosition[index0 * 3 + 2];

	float vecBx = meshPosition[index2 * 3 + 0] - meshPosition[index0 * 3 + 0];
	float vecBy = meshPosition[index2 * 3 + 1] - meshPosition[index0 * 3 + 1];
	float vecBz = meshPosition[index2 * 3 + 2] - meshPosition[index0 * 3 + 2];

	//��˼��������η���
	float crossX = vecAy * vecBz - vecBy * vecAz;
	float crossY = vecAz * vecBx - vecBz * vecAx;
	float crossZ = vecAx * vecBy - vecBx * vecAy;

	//���ߵ�λ��
	float product = crossX * crossX + crossY * crossY + crossZ * crossZ;
	product = sqrt(product);
	crossX /= product;
	crossY /= product;
	crossZ /= product;

	float len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	float len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle0 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
	vecAx = meshPosition[index0 * 3 + 0] - meshPosition[index1 * 3 + 0];
	vecAy = meshPosition[index0 * 3 + 1] - meshPosition[index1 * 3 + 1];
	vecAz = meshPosition[index0 * 3 + 2] - meshPosition[index1 * 3 + 2];

	vecBx = meshPosition[index2 * 3 + 0] - meshPosition[index1 * 3 + 0];
	vecBy = meshPosition[index2 * 3 + 1] - meshPosition[index1 * 3 + 1];
	vecBz = meshPosition[index2 * 3 + 2] - meshPosition[index1 * 3 + 2];
	len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle1 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
	vecAx = meshPosition[index0 * 3 + 0] - meshPosition[index2 * 3 + 0];
	vecAy = meshPosition[index0 * 3 + 1] - meshPosition[index2 * 3 + 1];
	vecAz = meshPosition[index0 * 3 + 2] - meshPosition[index2 * 3 + 2];

	vecBx = meshPosition[index1 * 3 + 0] - meshPosition[index2 * 3 + 0];
	vecBy = meshPosition[index1 * 3 + 1] - meshPosition[index2 * 3 + 1];
	vecBz = meshPosition[index1 * 3 + 2] - meshPosition[index2 * 3 + 2];
	len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle2 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("UpdateMeshNormalMU tri index: %d %d %d", index0, index1, index2);
		printf("p0[%f %f %f] p1[%f %f %f] p2[%f %f %f]\n",
			meshPosition[index0 * 3 + 0], meshPosition[index0 * 3 + 1], meshPosition[index0 * 3 + 2],
			meshPosition[index1 * 3 + 0], meshPosition[index1 * 3 + 1], meshPosition[index1 * 3 + 2],
			meshPosition[index2 * 3 + 0], meshPosition[index2 * 3 + 1], meshPosition[index2 * 3 + 2]);
	}
#endif
	//�������ۼӵ�ÿ����������Ƭ�Ķ�����
	atomicAdd(meshNormal + index0 * 3 + 0, crossX * angle0);
	atomicAdd(meshNormal + index0 * 3 + 1, crossY * angle0);
	atomicAdd(meshNormal + index0 * 3 + 2, crossZ * angle0);
	atomicAdd(totAngle + index0, angle0);

	atomicAdd(meshNormal + index1 * 3 + 0, crossX * angle1);
	atomicAdd(meshNormal + index1 * 3 + 1, crossY * angle1);
	atomicAdd(meshNormal + index1 * 3 + 2, crossZ * angle1);
	atomicAdd(totAngle + index1, angle1);

	atomicAdd(meshNormal + index2 * 3 + 0, crossX * angle2);
	atomicAdd(meshNormal + index2 * 3 + 1, crossY * angle2);
	atomicAdd(meshNormal + index2 * 3 + 2, crossZ * angle2);
	atomicAdd(totAngle + index2, angle2);
}

//���ݶ���λ�ú�������Ƭ������Ƭ����
__global__ void updateMeshNormalMU(float* meshPosition, float* meshNormal, 
	float* totAngle, unsigned int* meshTriangle, 
	int* sortedTriIndices, int offset, int activeElementNum) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int triIdx = sortedTriIndices[offset + threadid];
	//��ȡmesh�����ε��������������
	unsigned int index0 = meshTriangle[triIdx * 3 + 0];
	unsigned int index1 = meshTriangle[triIdx * 3 + 1];
	unsigned int index2 = meshTriangle[triIdx * 3 + 2];

	float vecAx = meshPosition[index1 * 3 + 0] - meshPosition[index0 * 3 + 0];
	float vecAy = meshPosition[index1 * 3 + 1] - meshPosition[index0 * 3 + 1];
	float vecAz = meshPosition[index1 * 3 + 2] - meshPosition[index0 * 3 + 2];

	float vecBx = meshPosition[index2 * 3 + 0] - meshPosition[index0 * 3 + 0];
	float vecBy = meshPosition[index2 * 3 + 1] - meshPosition[index0 * 3 + 1];
	float vecBz = meshPosition[index2 * 3 + 2] - meshPosition[index0 * 3 + 2];

	//��˼��������η���
	float crossX = vecAy * vecBz - vecBy * vecAz;
	float crossY = vecAz * vecBx - vecBz * vecAx;
	float crossZ = vecAx * vecBy - vecBx * vecAy;

	//���ߵ�λ��
	float product = crossX * crossX + crossY * crossY + crossZ * crossZ;
	product = sqrt(product);
	crossX /= product;
	crossY /= product;
	crossZ /= product;

	float len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	float len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle0 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
	vecAx = meshPosition[index0 * 3 + 0] - meshPosition[index1 * 3 + 0];
	vecAy = meshPosition[index0 * 3 + 1] - meshPosition[index1 * 3 + 1];
	vecAz = meshPosition[index0 * 3 + 2] - meshPosition[index1 * 3 + 2];

	vecBx = meshPosition[index2 * 3 + 0] - meshPosition[index1 * 3 + 0];
	vecBy = meshPosition[index2 * 3 + 1] - meshPosition[index1 * 3 + 1];
	vecBz = meshPosition[index2 * 3 + 2] - meshPosition[index1 * 3 + 2];
	len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle1 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
	vecAx = meshPosition[index0 * 3 + 0] - meshPosition[index2 * 3 + 0];
	vecAy = meshPosition[index0 * 3 + 1] - meshPosition[index2 * 3 + 1];
	vecAz = meshPosition[index0 * 3 + 2] - meshPosition[index2 * 3 + 2];

	vecBx = meshPosition[index1 * 3 + 0] - meshPosition[index2 * 3 + 0];
	vecBy = meshPosition[index1 * 3 + 1] - meshPosition[index2 * 3 + 1];
	vecBz = meshPosition[index1 * 3 + 2] - meshPosition[index2 * 3 + 2];
	len_A = sqrt(vecAx * vecAx + vecAy * vecAy + vecAz * vecAz);
	len_B = sqrt(vecBx * vecBx + vecBy * vecBy + vecBz * vecBz);
	vecAx /= len_A; vecAy /= len_A; vecAz /= len_A;
	vecBx /= len_B; vecBy /= len_B; vecBz /= len_B;
	float angle2 = acos(vecAx * vecBx + vecAy * vecBy + vecAz * vecBz);
	//------------------------------------------------------------------
#ifdef OUTPUT_INFO
	if (triIdx == LOOK_THREAD)
	{
		printf("UpdateMeshNormalMU tri index: %d %d %d", index0, index1, index2);
		printf("p0[%f %f %f] p1[%f %f %f] p2[%f %f %f]\n",
			meshPosition[index0 * 3 + 0], meshPosition[index0 * 3 + 1], meshPosition[index0 * 3 + 2],
			meshPosition[index1 * 3 + 0], meshPosition[index1 * 3 + 1], meshPosition[index1 * 3 + 2],
			meshPosition[index2 * 3 + 0], meshPosition[index2 * 3 + 1], meshPosition[index2 * 3 + 2]);
	}
#endif
	//�������ۼӵ�ÿ����������Ƭ�Ķ�����
	atomicAdd(meshNormal + index0 * 3 + 0, crossX * angle0);
	atomicAdd(meshNormal + index0 * 3 + 1, crossY * angle0);
	atomicAdd(meshNormal + index0 * 3 + 2, crossZ * angle0);
	atomicAdd(totAngle + index0, angle0);

	atomicAdd(meshNormal + index1 * 3 + 0, crossX * angle1);
	atomicAdd(meshNormal + index1 * 3 + 1, crossY * angle1);
	atomicAdd(meshNormal + index1 * 3 + 2, crossZ * angle1);
	atomicAdd(totAngle + index1, angle1);

	atomicAdd(meshNormal + index2 * 3 + 0, crossX * angle2);
	atomicAdd(meshNormal + index2 * 3 + 1, crossY * angle2);
	atomicAdd(meshNormal + index2 * 3 + 2, crossZ * angle2);
	atomicAdd(totAngle + index2, angle2);
}
//���߹�һ��
__global__ void normalizeMeshtriVertNorm_debug(float* meshNormal, float* meshPosition, float* totAngle, int meshVertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= meshVertexNum) return;

	float normalx = meshNormal[threadid * 3 + 0];
	float normaly = meshNormal[threadid * 3 + 1];
	float normalz = meshNormal[threadid * 3 + 2];

	// �����������нǵĴ�С�Զ��㷨�������м�Ȩ�����øö���������εķ�������ռȨ����ռ�øö���������εĽǶȳ����ȡ�
	float product = totAngle[threadid];
	normalx /= product;
	normaly /= product;
	normalz /= product;
	//���ߵ�λ��
	float sqr_len = normalx * normalx + normaly * normaly + normalz * normalz;
	float len = sqrt(sqr_len);
	normalx /= len;
	normaly /= len;
	normalz /= len;



#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		float px = meshPosition[threadid * 3 + 0];
		float py = meshPosition[threadid * 3 + 1];
		float pz = meshPosition[threadid * 3 + 2];
		printf("normalizeMeshNormalMU threadid %d: n[%f %f %f] p[%f %f %f]\n", threadid, normalx, normaly, normalz, px, py, pz);
		if (product < 1e-6)
		{
			printf("totalAngle product is too small\n");
		}
		if (len < 1e-6)
		{
			printf("length of normal is too small\n");
		}
	}
#endif
	meshNormal[threadid * 3 + 0] = normalx;
	meshNormal[threadid * 3 + 1] = normaly;
	meshNormal[threadid * 3 + 2] = normalz;
}

//���߹�һ��
__global__ void normalizeMeshtriVertNorm_debug(float* meshNormal, float* meshPosition, float* totAngle,
	int* sortedTriVertIndices, int offset, int activeElementNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;
	int triVertIdx = sortedTriVertIndices[offset + threadid];
	float normalx = meshNormal[triVertIdx * 3 + 0];
	float normaly = meshNormal[triVertIdx * 3 + 1];
	float normalz = meshNormal[triVertIdx * 3 + 2];

	// �����������нǵĴ�С�Զ��㷨�������м�Ȩ�����øö���������εķ�������ռȨ����ռ�øö���������εĽǶȳ����ȡ�
	float product = totAngle[triVertIdx];
	normalx /= product;
	normaly /= product;
	normalz /= product;
	//���ߵ�λ��
	float sqr_len = normalx * normalx + normaly * normaly + normalz * normalz;
	float len = sqrt(sqr_len);
	normalx /= len;
	normaly /= len;
	normalz /= len;

#ifdef OUTPUT_INFO
	if (triVertIdx == LOOK_THREAD)
	{
		float px = meshPosition[triVertIdx * 3 + 0];
		float py = meshPosition[triVertIdx * 3 + 1];
		float pz = meshPosition[triVertIdx * 3 + 2];
		printf("normalizeMeshNormalMU triVertIdx %d: n[%f %f %f] p[%f %f %f]\n", triVertIdx, normalx, normaly, normalz, px, py, pz);
		if (product < 1e-6)
		{
			printf("totalAngle product is too small\n");
		}
		if (len < 1e-6)
		{
			printf("length of normal is too small\n");
		}
	}
#endif
	meshNormal[triVertIdx * 3 + 0] = normalx;
	meshNormal[triVertIdx * 3 + 1] = normaly;
	meshNormal[triVertIdx * 3 + 2] = normalz;
}
int setDDirwithNormal()
{
	hipMemcpy(triVertNonPenetrationDir_d, triVertNorm_d, triVertNum_d * 3 * sizeof(float), hipMemcpyDeviceToDevice);
	int threadNum = 512;
	int blockNum = (triVertNum_d + threadNum - 1) / threadNum;

	setNonPenetrationDirWithTriVertNormal << <blockNum, threadNum >> > (triVertNonPenetrationDir_d, triVertNorm_d, triVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runUpdateDirectDirectionMU");
	return 0;
}

__global__ void setNonPenetrationDirWithTriVertNormal(float* nonPenetrationDir, float* normal, int vertexNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	nonPenetrationDir[indexX] = -normal[indexX];
	nonPenetrationDir[indexY] = -normal[indexY];
	nonPenetrationDir[indexZ] = -normal[indexZ];
}

__global__ void updateInnerTetVertDirectDirection(
	float* tetVertPositions,
	int* bindingTetVertIndices, float* bindingWeight, 
	float* directDir, 
	int vertexNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	
	float q[3] = { tetVertPositions[threadid * 3 + 0],tetVertPositions[threadid * 3 + 1] ,tetVertPositions[threadid * 3 + 2] };
	float w0 = bindingWeight[threadid * 3 + 0];
	float w1 = bindingWeight[threadid * 3 + 1];
	float w2 = bindingWeight[threadid * 3 + 2];
	int bindingTetVertIdx0 = bindingTetVertIndices[threadid * 3 + 0];
	int bindingTetVertIdx1 = bindingTetVertIndices[threadid * 3 + 1];
	int bindingTetVertIdx2 = bindingTetVertIndices[threadid * 3 + 2];
	float p0[3] = { tetVertPositions[bindingTetVertIdx0 * 3 + 0], tetVertPositions[bindingTetVertIdx0 * 3 + 1], tetVertPositions[bindingTetVertIdx0 * 3 + 2] };
	float p1[3] = { tetVertPositions[bindingTetVertIdx1 * 3 + 0], tetVertPositions[bindingTetVertIdx1 * 3 + 1], tetVertPositions[bindingTetVertIdx1 * 3 + 2] };
	float p2[3] = { tetVertPositions[bindingTetVertIdx2 * 3 + 0], tetVertPositions[bindingTetVertIdx2 * 3 + 1], tetVertPositions[bindingTetVertIdx2 * 3 + 2] };
	float p[3];
	p[0] = p0[0] * w0 + p1[0] * w1 + p2[0] * w2;
	p[1] = p0[1] * w0 + p1[1] * w1 + p2[1] * w2;
	p[2] = p0[2] * w0 + p1[2] * w1 + p2[2] * w2;
	float dDir[3];
	dDir[0] = q[0] - p[0];
	dDir[1] = q[1] - p[1];
	dDir[2] = q[2] - p[2];
	//if (w2 > 0)
	//{// a inner point
	//	printf("q: %f %f %f\npoint0: %f %f %f point1: %f %f %f point2: %f %f %f\ndis: %f %f %f\n",
	//		q[0], q[1], q[2],
	//		p0[0], p0[1], p0[2],
	//		p1[0], p1[1], p1[2],
	//		p2[0], p2[1], p2[2],
	//		l2len(q, p0),l2len(q, p1), l2len(q, p2));
	//}
	float dDirLen = sqrt(dDir[0] * dDir[0] + dDir[1] * dDir[1] + dDir[2] * dDir[2]);
	//if (dDirLen < 1e-6)
	//	printf("dDir not available. w0:%f w1:%f w2:%f\n", w0, w1, w2);
	directDir[threadid * 3 + 0] = dDir[0] / dDirLen;
	directDir[threadid * 3 + 1] = dDir[1] / dDirLen;
	directDir[threadid * 3 + 2] = dDir[2] / dDirLen;
	// ��ʱ��dDirΪ��Ӧ�ı��涥��ָ�������嶥���������
	// �����ǰ�����嶥����ģ�ͱ��棬dDir��������ġ�
	// ����취������һ�������м����ڱ���������嶥���ָ���������ö�Ӧ�ı����������񶥵�ķ������ķ������壩
	// ���������������֮����ͳһ��׼��
}

__global__ void updateSurfaceTetVertDirectDirection(
	int* onSurfaceTetVertIndices,
	int* TetVertNearestTriVertIndices, float* triVertNorm,
	float* tetVertDDir, float* tetVertPos, float* triVertPos, int surfaceTetVertNum)
{
	//�����ڱ���������嶥��ָ������������Ϊ����ﶨ�ı������񶥵㷨�����ķ�����
	// ����������񶥵�������嶥��֮���нϴ��ƫ�룬�����ָ�������ķ�����ܲ�׼ȷ��
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= surfaceTetVertNum) return;

	int surfaceTetVertIdx = onSurfaceTetVertIndices[threadid];
	int idx0 = surfaceTetVertIdx * 3 + 0;
	int idx1 = surfaceTetVertIdx * 3 + 1;
	int idx2 = surfaceTetVertIdx * 3 + 2;
	int matchingTriVertIdx = TetVertNearestTriVertIndices[surfaceTetVertIdx];
	float n[3] = { triVertNorm[matchingTriVertIdx * 3 + 0],triVertNorm[matchingTriVertIdx * 3 + 1] ,triVertNorm[matchingTriVertIdx * 3 + 2] };
	tetVertDDir[idx0] = -n[0];
	tetVertDDir[idx1] = -n[1];
	tetVertDDir[idx2] = -n[2];
	float tetPos[3] = { tetVertPos[idx0], tetVertPos[idx1], tetVertPos[idx2] };
	float triPos[3] = { triVertPos[matchingTriVertIdx * 3 + 0], triVertPos[matchingTriVertIdx * 3 + 1] ,triVertPos[matchingTriVertIdx * 3 + 2] };
	float d[3] = { tetPos[0] - triPos[0],tetPos[1] - triPos[1] ,tetPos[2] - triPos[2] };
	float dist = sqrt(d[0] * d[0] + d[1] * d[1] + d[2] * d[2]);
	//if (dist > 1e-3)
	//{
	//	printf("threadid:%d tetPoint-surfacepoint matching error! d[%f %f %f]\n", threadid, d[0], d[1], d[2]);
	//}
	//printf("SurfaceTetVertDDir: %f %f %f\n", n[0], n[1], n[2]);
}
void printCudaError() {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "error: %s\n", hipGetErrorString(cudaStatus));
	}
}

void printCudaError(const char* funcName) {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "%s error: %s\n", funcName, hipGetErrorString(cudaStatus));
	}
}