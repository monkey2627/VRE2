#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"

//��ת
__device__ float* cylinderShiftMU;
__device__ float* cylinderLastPosMU;
__device__ float* cylinderPosMU;
__device__ float* cylinderGraphicalPosMU;
__device__ float* cylinderDirZMU;
__device__ float* cylinderDirYMU;
__device__ float* cylinderDirXMU;
__device__ float* cylinderVMU;
__device__ float* relativePositionMU;
__device__ unsigned int* isGrapMU;
__device__ unsigned int* isGrapHalfMU;
__device__ float* adsorbForceMU;
__device__ float* grapperUpDirXMU;
__device__ float* grapperUpDirYMU;
__device__ float* grapperUpDirZMU;
__device__ float* grapperDownDirXMU;
__device__ float* grapperDownDirYMU;
__device__ float* grapperDownDirZMU;
__device__ unsigned int* collideFlag;

extern "C" int runcalculateCollisionCylinderMU(
	float length, float radius,
	float collisionStiffness, float adsorbStiffness,
	int flag)
{

	//ѡȡ�����ֹ���
	int		cylinderButton;
	bool* firstGrab;

	if (flag == 1) {  //Left
		cylinderShiftMU = cylinderShiftLeft_D;
		cylinderLastPosMU = cylinderLastPosLeft_D;
		cylinderPosMU = cylinderPosLeft_D;
		cylinderGraphicalPosMU = cylinderGraphicalPosLeft_D;
		cylinderDirZMU = cylinderDirZLeft_D;
		cylinderDirYMU = cylinderDirYLeft_D;
		cylinderDirXMU = cylinderDirXLeft_D;
		cylinderVMU = cylinderVLeft_D;
		relativePositionMU = relativePositionLeftMU_D;
		isGrapMU = isGrabLeftMU_D;
		isGrapHalfMU = isGrabHalfLeftMU_D;
		cylinderButton = cylinderButtonLeft_D;
		firstGrab = &firstGrabLeftMU_D;
		adsorbForceMU = adsorbForceLeft_D;
		grapperUpDirXMU = tetgrapperUpDirXLeft_D;
		grapperUpDirYMU = tetgrapperUpDirYLeft_D;
		grapperUpDirZMU = tetgrapperUpDirZLeft_D;
		grapperDownDirXMU = tetgrapperDownDirXLeft_D;
		grapperDownDirYMU = tetgrapperDownDirYLeft_D;
		grapperDownDirZMU = tetgrapperDownDirZLeft_D;
		collideFlag = CollideFlagLeftMU_D;
		timer = timerLeft;
		timeTop = timeTopLeft;
	}
	else {  //Right
		cylinderShiftMU = cylinderShiftRight_D;
		cylinderLastPosMU = cylinderLastPosRight_D;
		cylinderPosMU = cylinderPosRight_D;
		cylinderDirZMU = cylinderDirZRight_D;
		cylinderDirYMU = cylinderDirYRight_D;
		cylinderDirXMU = cylinderDirXRight_D;
		cylinderVMU = cylinderVRight_D;
		relativePositionMU = relativePositionRightMU_D;
		isGrapMU = isGrabRigthMU_D;
		isGrapHalfMU = isGrabHalfRightMU_D;
		cylinderButton = cylinderButtonRight_D;
		firstGrab = &firstGrabRightMU_D;
		adsorbForceMU = adsorbForceRight_D;
		grapperUpDirXMU = tetgrapperUpDirXRight_D;
		grapperUpDirYMU = tetgrapperUpDirYRight_D;
		grapperUpDirZMU = tetgrapperUpDirZRight_D;
		grapperDownDirXMU = tetgrapperDownDirXRight_D;
		grapperDownDirYMU = tetgrapperDownDirYRight_D;
		grapperDownDirZMU = tetgrapperDownDirZRight_D;
		collideFlag = CollideFlagRightMU_D;
		timer = timerRight;
		timeTop = timeTopRight;
	}

	//�жϹ���״̬
	switch (cylinderButton)
	{
	case grab: {
		//break;
		int threadNum = 512;
		int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
		//��һ�μ�ȡʱ�жϼ�ȡ����
		if (*firstGrab) {
			//printf("mesh grab\n");
			//���㱻��ȡ������Ķ���																							//���Ƽ�ȡ����Ĵ�С
			calculateGrabCylinderMU << <blockNum, threadNum >> > (cylinderPosMU, cylinderDirZMU, cylinderDirYMU, cylinderDirXMU, 0.5, 1.0, 2.2, triVertPos_d, isGrapMU, isGrapHalfMU, triVertNum_d, relativePositionMU);
			//hipDeviceSynchronize();
			*firstGrab = false;
		}

		if (timer < timeTop) { //��ȡ���м���̣���û�м�ס
			calculateGrabForceMU << <blockNum, threadNum >> > (cylinderPosMU, grapperUpDirZMU, grapperUpDirYMU, grapperUpDirXMU, 0.5, 0.5, 2.2, triVertPos_d, isGrapHalfMU, triVertNum_d, adsorbStiffness, triVertForce_d, triVertCollisionDiag_d, 1);
			//hipDeviceSynchronize();
			calculateGrabForceMU << <blockNum, threadNum >> > (cylinderPosMU, grapperDownDirZMU, grapperDownDirYMU, grapperDownDirXMU, 0.5, 0.5, 2.2, triVertPos_d, isGrapHalfMU, triVertNum_d, adsorbStiffness, triVertForce_d, triVertCollisionDiag_d, 2);
		}
		else { //�����ȫ��ס
			//��������ײ��⣬���Ǳ���֮ǰ����ײ��Ϣ��Լ����λ�ø��Ź����˶�
			calculateAdsorbForceMU << <blockNum, threadNum >> > (cylinderPosMU, cylinderDirXMU, cylinderDirYMU, cylinderDirZMU, triVertPos_d, isGrapMU, triVertForce_d, triVertCollisionDiag_d, relativePositionMU, triVertNum_d, adsorbStiffness);
			//�������ֵ�collide���кϲ�
			mergeCollideMU << <blockNum, threadNum >> > (triVertisCollide_d, collideFlag, isGrapMU, triVertNum_d);
		}

		hipDeviceSynchronize();
		break;
	}
	case normal: {
		int threadNum = 512;
		int blockNum = (triVertNum_d + threadNum - 1) / threadNum;
		*firstGrab = true;
		//������ײ���
		calculateCollisionCylinderAdvanceMU << <blockNum, threadNum >> > (cylinderLastPosMU, cylinderPosMU, cylinderDirZMU,      length, radius, triVertPos_d, triVertForce_d, 	triVertisCollide_d, collideFlag, triVertCollisionDiag_d,  triVertNum_d, collisionStiffness, triVertCollisionForce_d,  directDirectionMU_D, cylinderShiftMU);
		calculateCollisionCylinderAdvanceMU << <blockNum, threadNum >> > (cylinderLastPosMU, cylinderPosMU, grapperUpDirZMU, 2.0, radius * 0.7, triVertPos_d,  triVertForce_d, triVertisCollide_d, collideFlag, triVertCollisionDiag_d, triVertNum_d, collisionStiffness * 4, triVertCollisionForce_d, directDirectionMU_D, cylinderShiftMU);
		calculateCollisionCylinderAdvanceMU << <blockNum, threadNum >> > (cylinderLastPosMU, cylinderPosMU, grapperDownDirZMU, 2.0, radius * 0.7, triVertPos_d,  triVertForce_d, triVertisCollide_d, collideFlag, triVertCollisionDiag_d, triVertNum_d, collisionStiffness * 4, triVertCollisionForce_d, directDirectionMU_D, cylinderShiftMU);

		hipDeviceSynchronize();
	}
	default:
		break;
	}
	return 0;
}


//������Ҫ����ȡ�����������
__global__ void calculateGrabCylinderMU(float* cylinderPos, float* cylinderDirZ, float* cylinderDirY, float* cylinderDirX, float grappleX, float grappleY, float grappleZ, float* positions, unsigned int* isCollide, unsigned int* isCollideHalf, int vertexNum, float* relativePosition) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	isCollide[threadid] = 0;
	isCollideHalf[threadid] = 0;
	float collisionNormal[3];
	float collisionPos[3];
	float t = 0.0;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	bool collisionUp = obbCollisionMU(cylinderPos[0], cylinderPos[1], cylinderPos[2], cylinderDirX[0], cylinderDirX[1], cylinderDirX[2], cylinderDirY[0], cylinderDirY[1], cylinderDirY[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], positions[indexX], positions[indexY], positions[indexZ], grappleX * 1.5, grappleY, grappleZ);
	if (collisionUp) {
		//���ñ�־λ
		isCollide[threadid] = 1;
		//���㶥���ƫ��ֵ
		float vertexPosShift = (positions[indexX] - cylinderPos[0]) * cylinderDirY[0] + (positions[indexY] - cylinderPos[1]) * cylinderDirY[1] + (positions[indexZ] - cylinderPos[2]) * cylinderDirY[2];
		vertexPosShift = abs(vertexPosShift);
		//��¼��ײ��͹��ߵ����λ��
		relativePosition[indexX] = positions[indexX] - cylinderDirY[0] * (vertexPosShift - 0.05) - cylinderPos[0];
		relativePosition[indexY] = positions[indexY] - cylinderDirY[1] * (vertexPosShift - 0.05) - cylinderPos[1];
		relativePosition[indexZ] = positions[indexZ] - cylinderDirY[2] * (vertexPosShift - 0.05) - cylinderPos[2];
	}

	bool collisionDown = obbCollisionMU(cylinderPos[0], cylinderPos[1], cylinderPos[2], cylinderDirX[0], cylinderDirX[1], cylinderDirX[2], -cylinderDirY[0], -cylinderDirY[1], -cylinderDirY[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], positions[indexX], positions[indexY], positions[indexZ], grappleX * 1.5, grappleY, grappleZ);
	if (collisionDown) {
		isCollide[threadid] = 1;
		float vertexPosShift = (positions[indexX] - cylinderPos[0]) * cylinderDirY[0] + (positions[indexY] - cylinderPos[1]) * cylinderDirY[1] + (positions[indexZ] - cylinderPos[2]) * cylinderDirY[2];
		vertexPosShift = abs(vertexPosShift);
		relativePosition[indexX] = positions[indexX] + cylinderDirY[0] * (vertexPosShift - 0.05) - cylinderPos[0];
		relativePosition[indexY] = positions[indexY] + cylinderDirY[1] * (vertexPosShift - 0.05) - cylinderPos[1];
		relativePosition[indexZ] = positions[indexZ] + cylinderDirY[2] * (vertexPosShift - 0.05) - cylinderPos[2];
	}
	//δ��ײ��ֱ���˳�
	if (isCollide[threadid] != 1) return;

	//����ֲ�����
	float x = relativePosition[indexX] * cylinderDirX[0] + relativePosition[indexY] * cylinderDirX[1] + relativePosition[indexZ] * cylinderDirX[2];
	float y = relativePosition[indexX] * cylinderDirY[0] + relativePosition[indexY] * cylinderDirY[1] + relativePosition[indexZ] * cylinderDirY[2];
	float z = relativePosition[indexX] * cylinderDirZ[0] + relativePosition[indexY] * cylinderDirZ[1] + relativePosition[indexZ] * cylinderDirZ[2];
	//��¼�ֲ�����
	relativePosition[indexX] = x;
	relativePosition[indexY] = y;
	relativePosition[indexZ] = z;
}

//���㶥��������--��ȡ��
__global__ void calculateGrabForceMU(float* grapperPos, float* grapperDirZ, float* grapperDirY, float* grapperDirX, float grappleX, float grappleY, float grappleZ, float* positions, unsigned int* isCollide, int vertexNum, float adsorbStiffness, float* force, float* collisionDiag, unsigned int grabFlag) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float relativePos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	if (isCollide[threadid] == 0) {
		bool collisionFlag = obbCollisionMU(grapperPos[0], grapperPos[1], grapperPos[2], grapperDirX[0], grapperDirX[1], grapperDirX[2], grapperDirY[0], grapperDirY[1], grapperDirY[2], grapperDirZ[0], grapperDirZ[1], grapperDirZ[2], positions[indexX], positions[indexY], positions[indexZ], grappleX * 1.5, grappleY, grappleZ);
		if (!collisionFlag) return;
		//���ñ�־λ--���ĸ�ץǯ��ײ
		isCollide[threadid] = grabFlag;
	}

	if (isCollide[threadid] != grabFlag) return;
	//���㶥���ƫ��ֵ
	float vertexPosShift = (positions[indexX] - grapperPos[0]) * grapperDirY[0] + (positions[indexY] - grapperPos[1]) * grapperDirY[1] + (positions[indexZ] - grapperPos[2]) * grapperDirY[2];
	//vertexPosShift = abs(vertexPosShift);
	if (vertexPosShift < 0) vertexPosShift = 0;
	//��¼��ײ��͹��ߵ����λ��
	relativePos[0] = positions[indexX] - grapperDirY[0] * (vertexPosShift - 0.05) - grapperPos[0];
	relativePos[1] = positions[indexY] - grapperDirY[1] * (vertexPosShift - 0.05) - grapperPos[1];
	relativePos[2] = positions[indexZ] - grapperDirY[2] * (vertexPosShift - 0.05) - grapperPos[2];

	//����ֲ�����
	float x = relativePos[0] * grapperDirX[0] + relativePos[1] * grapperDirX[1] + relativePos[2] * grapperDirX[2];
	float y = relativePos[0] * grapperDirY[0] + relativePos[1] * grapperDirY[1] + relativePos[2] * grapperDirY[2];
	float z = relativePos[0] * grapperDirZ[0] + relativePos[1] * grapperDirZ[1] + relativePos[2] * grapperDirZ[2];

	float deltaPos[3];

	//����ƫ������
	float deltax = x * grapperDirX[0] + y * grapperDirY[0] + z * grapperDirZ[0];
	float deltay = x * grapperDirX[1] + y * grapperDirY[1] + z * grapperDirZ[1];
	float deltaz = x * grapperDirX[2] + y * grapperDirY[2] + z * grapperDirZ[2];

	float targetPosx = deltax + grapperPos[0];
	float targetPosy = deltay + grapperPos[1];
	float targetPosz = deltaz + grapperPos[2];

	float distance = calculateCylinderDisMU(grapperPos[0], grapperPos[1], grapperPos[2], grapperDirZ[0], grapperDirZ[1], grapperDirZ[2], targetPosx, targetPosy, targetPosz, 1.5);
	float k;
	//k = 1.0;
	k = 1 / (1 + exp(12 * distance - 5));
	adsorbStiffness = k * adsorbStiffness;

	deltaPos[0] = targetPosx - positions[indexX];
	deltaPos[1] = targetPosy - positions[indexY];
	deltaPos[2] = targetPosz - positions[indexZ];

	//ÿ�ζ������㣬�����ۼ�
	force[indexX] += adsorbStiffness * deltaPos[0];
	force[indexY] += adsorbStiffness * deltaPos[1];
	force[indexZ] += adsorbStiffness * deltaPos[2];

	collisionDiag[indexX] += adsorbStiffness;
	collisionDiag[indexY] += adsorbStiffness;
	collisionDiag[indexZ] += adsorbStiffness;
}

//����ץȡ��--��ȡ���
__global__ void calculateAdsorbForceMU(float* cylinderPos, float* cylinderDirX, float* cylinderDirY, float* cylinderDirZ, float* positions, unsigned int* isCollide, float* force, float* collisionDiag, float* relativePosition, int vertexNum, float adsorbStiffness) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//���������ײ���ֱ������
	if (isCollide[threadid] != 1) return;

	//����ײ��ͼ�����Ҫ���µ�λ�ã��ټ���attachԼ��
	float posx = positions[indexX];
	float posy = positions[indexY];
	float posz = positions[indexZ];
	float deltaPos[3];


	//����ƫ������
	float deltax = relativePosition[indexX] * cylinderDirX[0] + relativePosition[indexY] * cylinderDirY[0] + relativePosition[indexZ] * cylinderDirZ[0];
	float deltay = relativePosition[indexX] * cylinderDirX[1] + relativePosition[indexY] * cylinderDirY[1] + relativePosition[indexZ] * cylinderDirZ[1];
	float deltaz = relativePosition[indexX] * cylinderDirX[2] + relativePosition[indexY] * cylinderDirY[2] + relativePosition[indexZ] * cylinderDirZ[2];



	float targetPosx = deltax + cylinderPos[0];
	float targetPosy = deltay + cylinderPos[1];
	float targetPosz = deltaz + cylinderPos[2];

	float distance = calculateCylinderDisMU(cylinderPos[0], cylinderPos[1], cylinderPos[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], targetPosx, targetPosy, targetPosz, 1.5);
	float k;
	//k = 1.0;
	k = 1 / (1 + exp(12 * distance - 5));
	adsorbStiffness = k * adsorbStiffness;

	deltaPos[0] = targetPosx - posx;
	deltaPos[1] = targetPosy - posy;
	deltaPos[2] = targetPosz - posz;



	//ÿ�ζ������㣬�ۼӿ���
	force[indexX] += adsorbStiffness * deltaPos[0];
	force[indexY] += adsorbStiffness * deltaPos[1];
	force[indexZ] += adsorbStiffness * deltaPos[2];

	//�ᱻ���㣬�����ۼ�
	collisionDiag[indexX] += adsorbStiffness;
	collisionDiag[indexY] += adsorbStiffness;
	collisionDiag[indexZ] += adsorbStiffness;
}


//�ϲ���ȡ����ײ���
__global__ void mergeCollideMU(unsigned char* isCollide, unsigned int* CollideFlag, unsigned int* isGrap, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	if (isGrap[threadid] != 0) {
		isCollide[threadid] = 2;
		CollideFlag[0] = 2;
	}
}

//���㶥�㵽������ľ���
__device__ float calculateCylinderDisMU(float posx, float posy, float posz, float dirx, float diry, float dirz, float vertx, float verty, float vertz, float length) {
	float pos1x = posx + dirx * length;
	float pos1y = posy + diry * length;
	float pos1z = posz + dirz * length;
	float posdx = pos1x - posx;
	float posdy = pos1y - posy;
	float posdz = pos1z - posz;

	float dx = vertx - posx;
	float dy = verty - posy;
	float dz = vertz - posz;

	float t = dirx * dx + diry * dy + dirz * dz;
	t /= length;
	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	dx = vertx - posx - t * posdx;
	dy = verty - posy - t * posdy;
	dz = vertz - posz - t * posdz;
	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	return distance;
}


//���Զ����obb��Χ�н�����ײ���(ģ��ץǯץȡ�ķ�Χ)
__device__ bool obbCollisionMU(float posx, float posy, float posz, float dirXx, float dirXy, float dirXz, float dirYx, float dirYy, float dirYz, float dirZx, float dirZy, float dirZz, float vertx, float verty, float vertz, float width, float length, float height) {
	float x = (vertx - posx) * dirXx + (verty - posy) * dirXy + (vertz - posz) * dirXz;
	float y = (vertx - posx) * dirYx + (verty - posy) * dirYy + (vertz - posz) * dirYz;
	float z = (vertx - posx) * dirZx + (verty - posy) * dirZy + (vertz - posz) * dirZz;

	if (z < 0 || z > height) return false;
	if (y < 0 || y > length) return false;
	if (x < -width || x > width) return false;
	return true;
}

//ʹ����ͶӰ��������ɢ��ײ���
__device__ bool cylinderRayCollisionMU(float* cylinderPos, float* cylinderDir, float vertx, float verty, float vertz, float* moveDir, 
	float length, float radius, float* t, float* sln, float* collisionNormal, float* collisionPos) {

	float cylinder0x, cylinder0y, cylinder0z;
	cylinder0x = cylinderPos[0];
	cylinder0y = cylinderPos[1];
	cylinder0z = cylinderPos[2];
	float cylinder1x, cylinder1y, cylinder1z;
	cylinder1x = cylinderPos[0] + cylinderDir[0] * length;
	cylinder1y = cylinderPos[1] + cylinderDir[1] * length;
	cylinder1z = cylinderPos[2] + cylinderDir[2] * length;

	float cylinderdx = cylinder1x - cylinder0x;
	float cylinderdy = cylinder1y - cylinder0y;
	float cylinderdz = cylinder1z - cylinder0z;
	float dx = vertx - cylinder0x;
	float dy = verty - cylinder0y;
	float dz = vertz - cylinder0z;

	*t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	*t /= length;

	if (*t < 0) {
		*t = 0;
	}
	else if (*t > 1) {
		*t = 1;
	}

	dx = vertx - cylinder0x - (*t) * cylinderdx;
	dy = verty - cylinder0y - (*t) * cylinderdy;
	dz = vertz - cylinder0z - (*t) * cylinderdz;

	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	if (distance > radius) return false;

	//������ײ����ͶӰ�Ľ���
	float moveLength = sqrt(moveDir[0] * moveDir[0] + moveDir[1] * moveDir[1] + moveDir[2] * moveDir[2]);
	moveDir[0] /= moveLength;
	moveDir[1] /= moveLength;
	moveDir[2] /= moveLength;

	//��������
	collisionNormal[0] = moveDir[0];
	collisionNormal[1] = moveDir[1];
	collisionNormal[2] = moveDir[2];

	float projectx = cylinder0x + (*t) * cylinderdx;
	float projecty = cylinder0y + (*t) * cylinderdy;
	float projectz = cylinder0z + (*t) * cylinderdz;

	//����local��,���һ��һԪ���η���
	float solution;
	float SN = (vertx - projectx) * (collisionNormal[0]) + (verty - projecty) * (collisionNormal[1]) + (vertz - projectz) * (collisionNormal[2]);
	float SS = (vertx - projectx) * (vertx - projectx) + (verty - projecty) * (verty - projecty) + (vertz - projectz) * (vertz - projectz);
	solution = -SN + sqrt(SN * SN - SS + radius * radius);//ֻȡ����

	if (solution != solution) return false;

	//���⴫�ݳ�ȥ
	*sln = solution;

	collisionPos[0] = vertx + collisionNormal[0] * solution;
	collisionPos[1] = verty + collisionNormal[1] * solution;
	collisionPos[2] = vertz + collisionNormal[2] * solution;

	//�ٴ���������
	dx = collisionPos[0] - projectx;
	dy = collisionPos[1] - projecty;
	dz = collisionPos[2] - projectz;
	distance = sqrt(dx * dx + dy * dy + dz * dz);
	collisionNormal[0] = dx / distance;
	collisionNormal[1] = dy / distance;
	collisionNormal[2] = dz / distance;

	return true;

}


//ʹ��������ײ���ı�������ײ����㷨
__global__ void calculateCollisionCylinderAdvanceMU(
	float* cylinderLastPos, float* cylinderPos, float* cylinderDir,
	float halfLength, float radius,
	float* positions, float* force,
	unsigned char* isCollide, unsigned int* collideFlag, float* collisionDiag,
	int vertexNum,
	float collisionStiffness, float* collisionForce,
	float* directDir, float* cylinderShift)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}

	//ָ��������ײ���ķ���
	float moveDir[3];
	moveDir[0] = directDir[indexX];
	moveDir[1] = directDir[indexY];
	moveDir[2] = directDir[indexZ];

	//ƫ��һ���뾶���ȣ���������뾶Ϊԭ����������ʵ��ƫ�ĵ�Բ��
	float newPos[3];
	newPos[0] = cylinderPos[0] + cylinderShift[0] * 0.5 * radius;
	newPos[1] = cylinderPos[1] + cylinderShift[1] * 0.5 * radius;
	newPos[2] = cylinderPos[2] + cylinderShift[2] * 0.5 * radius;
	radius *= 1.5;
	//radius *= 2.0;

	float tetPosition[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPos[0], cylinderLastPos[1] - cylinderPos[1], cylinderLastPos[2] - cylinderPos[2] };
	float moveDistance = tettriVertNorm_d(toolMoveDir);

	//if(threadid==5758)
	//	printf("threadid:%d p[%f %f %f]\n", threadid, tetPosition[0], tetPosition[1], tetPosition[2]);

	if (moveDistance > 0.5) {
		//ʹ��������ײ����������������ײ
		bool collisionContinues = cylinderCollisionContinueMU(halfLength, moveDistance, radius, cylinderPos, cylinderLastPos, cylinderDir, 
			toolMoveDir, tetPosition, &t, collisionNormal, collisionPos, moveDir);
		if (!collisionContinues) return;
	}
	else {
		//ʹ��ָ�������������ײ���
		bool collision = cylinderRayCollisionMU(newPos, cylinderDir, positions[indexX], positions[indexY], positions[indexZ], 
			moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
		if (!collision) return;
	}

	float deltaPos[3];
	deltaPos[0] = collisionPos[0] - positions[indexX];
	deltaPos[1] = collisionPos[1] - positions[indexY];
	deltaPos[2] = collisionPos[2] - positions[indexZ];


	//���ñ�־λ
	isCollide[threadid] = 1;
	collideFlag[0] = 1;
}


// ʹ��������ײ������������ײ���ж�
__device__ bool cylinderCollisionContinueMU(float length, float moveDistance, float radius, float* cylinderPos, float* cylinderLastPos, float* cylinderDir, float* moveDir, float* position, float* t, float* collisionNormal, float* collisionPos, float* directDir) {
	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCrossMU_D(cylinderDir, moveDir, normal);
	tettriVertNorm_d(normal);

	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - cylinderPos[0] ,position[1] - cylinderPos[1] ,position[2] - cylinderPos[2] };
	float lineStart0[3] = { cylinderPos[0] ,cylinderPos[1] ,cylinderPos[2] };
	float lineStart1[3] = { cylinderLastPos[0] ,cylinderLastPos[1] ,cylinderLastPos[2] };
	float lineStart2[3] = { cylinderPos[0] + cylinderDir[0] * length ,cylinderPos[1] + cylinderDir[1] * length,cylinderPos[2] + cylinderDir[2] * length };


	//����Ҫ�Ƚ���һ����ײ��⣬�������Ƿ�����ײ��������������Ҫ������ײ���


	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	float x, y, z;
	float det = tetSolveFormulaMU_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float distance = 0.0;
	//2.���������ҵ��������ڵ�����
	if (x > length && y > moveDistance) {
		//����㵽��ľ���
		float basePoint[3] = { cylinderLastPos[0] + length * cylinderDir[0],cylinderLastPos[1] + length * cylinderDir[1] , cylinderLastPos[2] + length * cylinderDir[2] };
		distance = tetPointPointDistanceMU_D(position, basePoint);
	}
	else if (x > length && y < moveDistance && y>0.0) {
		//����㵽�ߵľ���
		distance = tetPointLineDistanceMU_D(lineStart2, moveDir, position);
	}
	else if (x > length && y < 0.0) {
		distance = tetPointPointDistanceMU_D(position, lineStart2);
	}
	else if (x > 0.0 && x < length && y > moveDistance) {
		distance = tetPointLineDistanceMU_D(lineStart1, cylinderDir, position);
	}
	else if (x > 0.0 && x < length && y < moveDistance && y>0.0) {
		//����㵽��ľ���
		distance = abs(z);
	}
	else if (x > 0.0 && x < length && y < 0.0) {
		distance = tetPointLineDistanceMU_D(lineStart0, cylinderDir, position);
	}
	else if (x<0.0 && y > moveDistance) {
		distance = tetPointPointDistanceMU_D(position, cylinderLastPos);
	}
	else if (x < 0.0 && y < moveDistance && y>0.0) {
		distance = tetPointLineDistanceMU_D(lineStart0, moveDir, position);
	}
	else if (x < 0.0 && y < 0.0) {
		distance = tetPointPointDistanceMU_D(position, cylinderPos);
	}


	//3.�жϾ���
	if (distance > radius) return false;


	//printf("x:%f,y:%f,z:%f\n", x, y, z);

	//4. �����������ײ�ų�λ��
	//����Ԫһ�η���,�����������Բ�����м���
	float lineDir[3] = { moveDir[0],moveDir[1], moveDir[2] };


	float v0[3] = { position[0] - lineStart0[0] ,position[1] - lineStart0[1] ,position[2] - lineStart0[2] };
	float v1[3] = { position[0] - lineStart1[0] ,position[1] - lineStart1[1] ,position[2] - lineStart1[2] };
	float v2[3] = { position[0] - lineStart2[0] ,position[1] - lineStart2[1] ,position[2] - lineStart2[2] };


	//��Բ���ཻ
	float solve00, solve01;
	float solve10, solve11;
	tetSolveInsectMU_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
	tetSolveInsectMU_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
	float solve = min(solve11, solve01);


	//�����ཻ
	float solve20, solve21;
	tetSolveInsectSphereMU_D(lineDir, v0, radius, &solve20, &solve21);
	solve = min(solve, solve21);


	if (solve != solve) return false;
	//printf("x:%f,y:%f,z:%f, solve: %f\n",x,y,z, solve);


	//����λ�õõ������ų���λ��
	collisionPos[0] = position[0] - lineDir[0] * solve;
	collisionPos[1] = position[1] - lineDir[1] * solve;
	collisionPos[2] = position[2] - lineDir[2] * solve;

	//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
	float projPos[3] = { collisionPos[0] - cylinderPos[0],collisionPos[1] - cylinderPos[1],collisionPos[2] - cylinderPos[2] };
	float proj = tetDotMU_D(projPos, cylinderDir);
	projPos[0] = collisionPos[0] - cylinderPos[0] - cylinderDir[0] * proj;
	projPos[1] = collisionPos[1] - cylinderPos[1] - cylinderDir[1] * proj;
	projPos[2] = collisionPos[2] - cylinderPos[2] - cylinderDir[2] * proj;

	tettriVertNorm_d(projPos);
	collisionNormal[0] = projPos[0];
	collisionNormal[1] = projPos[1];
	collisionNormal[2] = projPos[2];

	//printf("continue: x:%f,y:%f,z:%f,solve:%f\n", collisionPos[0], collisionPos[1], collisionPos[2],solve);
	//printf("continue: nx:%f,ny:%f,nz:%f\n", collisionNormal[0], collisionNormal[1], collisionNormal[2]);
	return true;
}



__device__ float tettriVertNorm_d(float* vec0) {
	float length = vec0[0] * vec0[0] + vec0[1] * vec0[1] + vec0[2] * vec0[2];
	length = sqrt(length);
	vec0[0] /= length;
	vec0[1] /= length;
	vec0[2] /= length;
	return length;
}


__device__ void tetCrossMU_D(float* a, float* b, float* c) {
	//��˼��������η���
	c[0] = a[1] * b[2] - b[1] * a[2];
	c[1] = a[2] * b[0] - b[2] * a[0];
	c[2] = a[0] * b[1] - b[0] * a[1];
}

__device__ float tetDotMU_D(float* a, float* b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__device__ float tetSolveFormulaMU_D(float* xAxis, float* yAxis, float* zAxis, float* target, float* x, float* y, float* z) {
	//ʹ�ø�˹��Ԫ
	float a[3][3] = {
		{ xAxis[0], yAxis[0], zAxis[0] },
		{ xAxis[1], yAxis[1], zAxis[1] },
		{ xAxis[2], yAxis[2], zAxis[2] },
	};

	float b[3] = { target[0],target[1],target[2] };

	//��һ����Ԫ
	float k = -a[1][0] / a[0][0];
	a[1][0] += a[0][0] * k;
	a[1][1] += a[0][1] * k;
	a[1][2] += a[0][2] * k;
	b[1] += b[0] * k;

	k = -a[2][0] / a[0][0];
	a[2][0] += a[0][0] * k;
	a[2][1] += a[0][1] * k;
	a[2][2] += a[0][2] * k;
	b[2] += b[0] * k;

	//�ڶ�����Ԫ
	k = -a[2][1] / a[1][1];
	a[2][1] += a[1][1] * k;
	a[2][2] += a[1][2] * k;
	b[2] += b[1] * k;

	//������
	*z = b[2] / a[2][2];
	*y = (b[1] - (*z) * a[1][2]) / a[1][1];
	*x = (b[0] - (*y) * a[0][1] - (*z) * a[0][2]) / a[0][0];

	return 0;
}

__device__ float tetPointLineDistanceMU_D(float* lineStart, float* lineDir, float* point) {
	float v[3] = { point[0] - lineStart[0],point[1] - lineStart[1], point[2] - lineStart[2] };

	float d = tetDotMU_D(lineDir, v);

	float projPos[3] = { lineStart[0] + d * lineDir[0],lineStart[1] + d * lineDir[1] ,lineStart[2] + d * lineDir[2] };

	projPos[0] = point[0] - projPos[0];
	projPos[1] = point[1] - projPos[1];
	projPos[2] = point[2] - projPos[2];

	return tettriVertNorm_d(projPos);
}

__device__ float tetPointPointDistanceMU_D(float* start, float* end) {
	float x = start[0] - end[0];
	float y = start[1] - end[1];
	float z = start[2] - end[2];
	return sqrt(x * x + y * y + z * z);
}

//���ߺ�Բ����
__device__ void tetSolveInsectMU_D(float* lineDir, float* toolDir, float* VSubO, float radius, float* solve0, float* solve1) {

	float temp0 = tetDotMU_D(VSubO, toolDir);
	float temp1 = -tetDotMU_D(lineDir, toolDir);

	float Bvector[3] = { VSubO[0] - temp0 * toolDir[0], VSubO[1] - temp0 * toolDir[1],  VSubO[2] - temp0 * toolDir[2] };
	float Avector[3] = { -lineDir[0] - temp1 * toolDir[0],-lineDir[1] - temp1 * toolDir[1],-lineDir[2] - temp1 * toolDir[2] };


	float A = tetDotMU_D(Avector, Avector);
	float B = 2.0 * tetDotMU_D(Avector, Bvector);
	float C = tetDotMU_D(Bvector, Bvector) - radius * radius;
	float delta = B * B - 4 * A * C;
	*solve0 = (-sqrt(delta) - B) / (2.0 * A);
	*solve1 = (sqrt(delta) - B) / (2.0 * A);
	if (*solve1 < 0.0) *solve1 = 0.0;
}

//���ߺ�����
__device__ void tetSolveInsectSphereMU_D(float* lineDir, float* VSubO, float radius, float* solve0, float* solve1) {
	float dir[3] = { -lineDir[0],-lineDir[1] ,-lineDir[2] };

	float distance = tetDotMU_D(VSubO, VSubO);

	float A = 1.0;
	float B = tetDotMU_D(dir, VSubO) * 2.0;
	float C = distance - radius * radius;

	float delta = B * B - 4 * A * C;

	*solve0 = (-sqrt(delta) - B) / (2.0 * A);
	*solve1 = (sqrt(delta) - B) / (2.0 * A);
	if (*solve1 < 0.0) *solve1 = 0.0;
}

__global__ void calculateToolShiftMU(float* cylinderPos, float* cylinderDir, float* directDir, 
	float halfLength, float radius, float* positions, float* cylinderShift, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//����һ�����ߵ���ײ���
	//ָ��������ײ���ķ���
	float moveDir[3];
	moveDir[0] = directDir[indexX];
	moveDir[1] = directDir[indexY];
	moveDir[2] = directDir[indexZ];

	//ʹ��ָ�������������ײ���
	bool collision = cylinderRayCollisionMU(cylinderPos, cylinderDir, positions[indexX], positions[indexY], positions[indexZ], 
		moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
	if (!collision) return;

	//�ۼӵõ�ƫ������
	atomicAdd(cylinderShift + 0, -directDir[indexX]);
	atomicAdd(cylinderShift + 1, -directDir[indexY]);
	atomicAdd(cylinderShift + 2, -directDir[indexZ]);
}



////����Բ������ײ
//int runcalculateCollisionCylinder(float halfLength, float radius, float collisionStiffness, float adsorbStiffness, float frictionStiffness, float forceDirX, float forceDirY, float forceDirZ, int flag)
//{
//	/*hipGraphicsMapResources(1, &tetIndex_OPENGL, 0);
//	size_t size = tetNum_d * 12 * sizeof(unsigned int);
//	hipGraphicsResourceGetMappedPointer((void **)&tetDrawIndex_D, &size, tetIndex_OPENGL);*/
//
//	//ѡȡ�����ֵĹ���
//	int		cylinderButton;
//	bool* firstGrab;
//
//	if (flag == 1) {
//		cylinderShift = cylinderShiftLeft_D;
//		cylinderLastPos = cylinderLastPosLeft_D;
//		cylinderPos = cylinderPosLeft_D;// ����λ�˻�ͼ��λ�ˣ�����ʱ���ƽ�������ֵ��ʲô��
//		cylinderGraphicalPos = cylinderGraphicalPosLeft_D;// ͼ��λ��
//		cylinderDirZ = cylinderDirZLeft_D;
//		cylinderDirY = cylinderDirYLeft_D;
//		cylinderDirX = cylinderDirXLeft_D;
//		cylinderV = cylinderVLeft_D;
//		relativePosition = relativePositionLeft_D;
//		isGrap = isGrapLeft_D;
//		isGrapHalf = isGrapHalfLeft_D;
//		cylinderButton = cylinderButtonLeft_D;
//		firstGrab = &firstGrabLeft_D;
//		adsorbForce = adsorbForceLeft_D;
//		//grapperUpPos = tetgrapperUpPosLeft_D;
//		//grapperDownPos = tetgrapperDownPosLeft_D;
//		grapperUpDirX = tetgrapperUpDirXLeft_D;
//		grapperUpDirY = tetgrapperUpDirYLeft_D;
//		grapperUpDirZ = tetgrapperUpDirZLeft_D;
//		grapperDownDirX = tetgrapperDownDirXLeft_D;
//		grapperDownDirY = tetgrapperDownDirYLeft_D;
//		grapperDownDirZ = tetgrapperDownDirZLeft_D;
//		sphereGrabFlag = grabFlagLeft_D;
//		collideFlag = collideFlagLeft_D;
//		timer = timerLeft;
//		timeTop = timeTopLeft;
//	}
//	else {
//		cylinderShift = cylinderShiftRight_D;
//		cylinderLastPos = cylinderLastPosRight_D;
//		cylinderPos = cylinderPosRight_D;
//		cylinderDirZ = cylinderDirZRight_D;
//		cylinderDirY = cylinderDirYRight_D;
//		cylinderDirX = cylinderDirXRight_D;
//		cylinderV = cylinderVRight_D;
//		relativePosition = relativePositionRight_D;
//		isGrap = isGrapRight_D;
//		isGrapHalf = isGrapHalfRigth_D;
//		cylinderButton = cylinderButtonRight_D;
//		firstGrab = &firstGrabRight_D;
//		adsorbForce = adsorbForceRight_D;
//		//grapperUpPos = tetgrapperUpPosRight_D;
//		//grapperDownPos = tetgrapperDownPosRight_D;
//		grapperUpDirX = tetgrapperUpDirXRight_D;
//		grapperUpDirY = tetgrapperUpDirYRight_D;
//		grapperUpDirZ = tetgrapperUpDirZRight_D;
//		grapperDownDirX = tetgrapperDownDirXRight_D;
//		grapperDownDirY = tetgrapperDownDirYRight_D;
//		grapperDownDirZ = tetgrapperDownDirZRight_D;
//		sphereGrabFlag = grabFlagRight_D;
//		timer = timerRight;
//		timeTop = timeTopRight;
//	}
//
//	//������һ���жϣ��Ƿ��ڼ�ȡ״̬
//	switch (cylinderButton)
//	{
//
//	case grab: {
//		int  threadNum = 512;
//		int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
//		//��ÿ�μ�ȡ�ĵ�һ�ν��м�ȡ������ж�
//		if (*firstGrab) {
//			//printf("grab\n");
//			//���㱻��ȡ������Ķ���																					//���Ƽ�ȡ�����С
//			calculateGrabCylinder << <blockNum, threadNum >> > (cylinderPos, cylinderDirZ, cylinderDirY, cylinderDirX, 0.5, 1.0, 2.2, tetVertPos_d, isGrap, isGrapHalf, tetVertNum_d, relativePosition, directIndex_D, sphereGrabFlag);
//			//hipDeviceSynchronize();
//			*firstGrab = false;
//		}
//		//calculateGrabOBB << <blockNum, threadNum >> > (grapperUpPos, grapperUpDirZ, grapperUpDirY, grapperUpDirX, grapperDownPos, grapperDownDirZ, grapperDownDirY, grapperDownDirX, 0.48, 0.43, 2.0, tetVertPos_d, tetVertNum_d, CollideFlag_D);
//		//��������ײ��⣬���Ǳ���֮ǰ����ײ��Ϣ��Լ����λ�ø��Ź����˶�
//		//hipDeviceSynchronize();
//		//calculateAdsorbForce << <blockNum, threadNum >> > (cylinderPos, cylinderDirX, cylinderDirY, cylinderDirZ, tetVertPos_d, isGrap, tetVertForce_d, tetCollisionDiag_d, relativePosition, tetVertNum_d, adsorbStiffness, CollideFlag_D);
//
//		if (timer < timeTop) { //��ȡ���м���̣���û����ȫ��ס
//			calculateGrabForce << <blockNum, threadNum >> > (cylinderPos, grapperUpDirZ, grapperUpDirY, grapperUpDirX, 0.5, 0.5, 2.2, tetVertPos_d, isGrapHalf, tetVertNum_d, adsorbStiffness, tetVertForce_d, tetVertCollisionDiag_d, 1);
//			//hipDeviceSynchronize();
//			calculateGrabForce << <blockNum, threadNum >> > (cylinderPos, grapperDownDirZ, grapperDownDirY, grapperDownDirX, 0.5, 0.5, 2.2, tetVertPos_d, isGrapHalf, tetVertNum_d, adsorbStiffness, tetVertForce_d, tetVertCollisionDiag_d, 2);
//		}
//		else { //�����ȫ��ס
//			calculateAdsorbForce << <blockNum, threadNum >> > (cylinderPos, cylinderDirX, cylinderDirY, cylinderDirZ, tetVertPos_d, isGrap, tetVertForce_d, tetVertCollisionDiag_d, relativePosition, tetVertNum_d, adsorbStiffness);
//			//�������ֵ�collide���кϲ� from Dou �õ���ʱ���ٺϲ���ش��롣
//			//mergeCollideMU << <blockNum, threadNum >> > (triVertIsCollide_d, collideFlag, isGrapMU, triVertNum_d);
//		}
//
//		//�����������˵����������ݸ���������
//		blockNum = (sphereNum_D + threadNum - 1) / threadNum;
//		//calculateAdsorbForceForHaptic << <blockNum, threadNum >> > (spherePositions_D, sphereConnectStart_D, sphereConnectCount_D, sphereConnect_D, sphereConnectLength_D, sphereGrabFlag, adsorbForce, sphereNum_D);
//		hipDeviceSynchronize();
//		break;
//	}
//	case normal: {
//		int  threadNum = 512;
//		int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
//		*firstGrab = true;
//		//���м�����ײ
//		//calculateCollisionCylinderGraphical << <blockNum, threadNum >> > (
//		//	cylinderGraphicalPos, cylinderDirZ, cylinderV, halfLength, radius, 
//		//	tetVertPos_d, isCollideGraphical_D, 
//		//	tetVertNum_d);
//		//calculateCollisionCylinder << <blockNum, threadNum >> > (cylinderPos, cylinderDirZ, cylinderV, halfLength, radius,
//		//	tetVertPos_d, tetVertVelocity_d, tetVertForce_d, tetIsCollide_d,
//		//	tetCollisionDiag_d, tetVolumeDiag_d, tetVertNum_d, collisionStiffness, frictionStiffness);
//		// -----------------------------------------------------------------------
//#ifdef WITH_DIRECTDIR
//		calculateCollisionCylinderAdvance << <blockNum, threadNum >> > (
//			cylinderLastPos, cylinderPos, cylinderDirZ, cylinderV, halfLength, radius,
//			tetVertPos_d, tetVertVelocity_d, tetVertForce_d, tetIsCollide_d,
//			tetCollisionDiag_d, tetVolumeDiag_d,
//			tetVertNum_d,
//			collisionStiffness,
//			frictionStiffness,
//			tetCollisionForce_d,
//			directDirection_D,
//			cylinderShift);
//#else
//		calculateCollisionCylinderAdvance_without_directDir << <blockNum, threadNum >> > (
//			cylinderLastPos, cylinderPos, cylinderDirZ, cylinderV, halfLength, radius,
//			tetVertPos_d, tetVertVelocity_d, tetVertForce_d, tetIsCollide_d,
//			tetVertCollisionDiag_d, tetVolumeDiag_d,
//			tetVertNum_d,
//			collisionStiffness,
//			frictionStiffness,
//			tetVertCollisionForce_d,
//			cylinderShift);
//#endif // WITH_DIRECTDIR
//		// -----------------------------------------------------------------------
//
//
//		hipDeviceSynchronize();
//
//		break;
//	}
//	default:
//		break;
//	}
//
//	//��ץȡ������
//	if (*firstGrab) {
//		hipMemset(sphereGrabFlag, 0, sizeof(int) * sphereNum_D);
//		//printf("clear\n");
//	}
//
//	return 0;
//}


//����ץȡ��
__global__ void calculateAdsorbForce(float* cylinderPos, float* cylinderDirX, float* cylinderDirY, float* cylinderDirZ, float* positions, unsigned int* isCollide, float* force, float* collisionDiag, float* relativePosition, int vertexNum, float adsorbStiffness) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//���������ײ���ֱ������
	if (isCollide[threadid] == 0) return;

	//����ײ��ͼ�����Ҫ���µ�λ�ã��ټ���attachԼ��
	float posx = positions[indexX];
	float posy = positions[indexY];
	float posz = positions[indexZ];
	float deltaPos[3];


	//����ƫ������
	float deltax = relativePosition[indexX] * cylinderDirX[0] + relativePosition[indexY] * cylinderDirY[0] + relativePosition[indexZ] * cylinderDirZ[0];
	float deltay = relativePosition[indexX] * cylinderDirX[1] + relativePosition[indexY] * cylinderDirY[1] + relativePosition[indexZ] * cylinderDirZ[1];
	float deltaz = relativePosition[indexX] * cylinderDirX[2] + relativePosition[indexY] * cylinderDirY[2] + relativePosition[indexZ] * cylinderDirZ[2];

	float targetPosx = deltax + cylinderPos[0];
	float targetPosy = deltay + cylinderPos[1];
	float targetPosz = deltaz + cylinderPos[2];

	float distance = calculateCylinderDis(cylinderPos[0], cylinderPos[1], cylinderPos[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], targetPosx, targetPosy, targetPosz, 1.5);
	float k;
	//k = 1.0;
	k = 1 / (1 + exp(12 * distance - 5));
	adsorbStiffness = k * adsorbStiffness;

	deltaPos[0] = targetPosx - posx;
	deltaPos[1] = targetPosy - posy;
	deltaPos[2] = targetPosz - posz;

	//ÿ�ζ������㣬�ۼӿ���
	force[indexX] += adsorbStiffness * deltaPos[0];
	force[indexY] += adsorbStiffness * deltaPos[1];
	force[indexZ] += adsorbStiffness * deltaPos[2];

	//�ᱻ���㣬�����ۼ�
	collisionDiag[indexX] += adsorbStiffness;
	collisionDiag[indexY] += adsorbStiffness;
	collisionDiag[indexZ] += adsorbStiffness;


}

//���㶥�㵽������ľ���
__device__ float calculateCylinderDis(float posx, float posy, float posz, float dirx, float diry, float dirz, float vertx, float verty, float vertz, float length) {
	float pos1x = posx + dirx * length;
	float pos1y = posy + diry * length;
	float pos1z = posz + dirz * length;
	float posdx = pos1x - posx;
	float posdy = pos1y - posy;
	float posdz = pos1z - posz;

	float dx = vertx - posx;
	float dy = verty - posy;
	float dz = vertz - posz;

	float t = dirx * dx + diry * dy + dirz * dz;
	t /= length;
	if (t < 0) {
		t = 0;
	}
	else if (t > 1) {
		t = 1;
	}

	dx = vertx - posx - t * posdx;
	dy = verty - posy - t * posdy;
	dz = vertz - posz - t * posdz;
	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	return distance;
}

//ʹ��������ײ���ı�������ײ����㷨
__global__ void calculateCollisionCylinderAdvance_without_directDir(
	float* cylinderLastPos, float* cylinderPos,
	float* cylinderDir, float* cylinderV,
	float halfLength, float radius,
	float* positions, float* velocity, float* force,
	unsigned int* isCollide,
	float* collisionDiag,
	float* volumnDiag,
	int vertexNum, float collisionStiffness, float frictionStiffness,
	float* collisionForce, float* cylinderShift)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float t = 0.0;
	float solution = 0.0;

	float collisionNormal[3];
	float collisionPos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	//��ƫ��������Ϊ��λ����
	float shiftLength = sqrt(cylinderShift[0] * cylinderShift[0] + cylinderShift[1] * cylinderShift[1] + cylinderShift[2] * cylinderShift[2]);
	if (shiftLength > 0.01f) {
		cylinderShift[0] /= shiftLength;
		cylinderShift[1] /= shiftLength;
		cylinderShift[2] /= shiftLength;
	}

	float tetPosition[3] = { positions[indexX] ,positions[indexY] ,positions[indexZ] };
	float toolMoveDir[3] = { cylinderLastPos[0] - cylinderPos[0],cylinderLastPos[1] - cylinderPos[1], cylinderLastPos[2] - cylinderPos[2] };
	float moveDistance = tetNormal_D(toolMoveDir);

	float ratio = 0.0f;
	float newPos[3];
	newPos[0] = cylinderPos[0] + cylinderShift[0] * ratio * radius;
	newPos[1] = cylinderPos[1] + cylinderShift[1] * ratio * radius;
	newPos[2] = cylinderPos[2] + cylinderShift[2] * ratio * radius;
	float w = moveDistance / radius;
	float enlarged_radius = radius * (1.5 - 0.5 / w);

	if (moveDistance > 0.05) {
		//�޸�ʹ��������ײ����������������ײ
		// �޶���ָ��������������ײ���
		bool collisionContinus = cylinderCollisionContinue_without_directDir(halfLength, moveDistance, enlarged_radius, cylinderPos, cylinderLastPos, cylinderDir, toolMoveDir, tetPosition, &t, collisionNormal, collisionPos);
		if (!collisionContinus) return;
		//printf("lianxu\n");
	}
	else {
		//ʹ��ָ�������������ײ���
		////bool collision = cylinderRayCollisionDetection(newPos, cylinderDir, positions[indexX], positions[indexY], positions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
		//bool collision = cylinderRayCollisionDetection(cylinderPos, cylinderDir, positions[indexX], positions[indexY], positions[indexZ], moveDir, halfLength, radius, &t, &solution, collisionNormal, collisionPos);
		float vert[3] = { positions[indexX], positions[indexY], positions[indexZ] };
		bool collision = cylinderCollision(cylinderPos, cylinderDir, vert, halfLength, radius, &t, collisionNormal, collisionPos);
		if (!collision) return;
		//printf("---lisan\n");
	}

	float deltaPos[3];
	deltaPos[0] = collisionPos[0] - positions[indexX];
	deltaPos[1] = collisionPos[1] - positions[indexY];
	deltaPos[2] = collisionPos[2] - positions[indexZ];

	// ������ײ����Ӵ�����
	float temp[3];
	temp[0] = collisionStiffness * (collisionNormal[0] * collisionNormal[0] * deltaPos[0] + collisionNormal[0] * collisionNormal[1] * deltaPos[1] + collisionNormal[0] * collisionNormal[2] * deltaPos[2]);
	temp[1] = collisionStiffness * (collisionNormal[1] * collisionNormal[0] * deltaPos[0] + collisionNormal[1] * collisionNormal[1] * deltaPos[1] + collisionNormal[1] * collisionNormal[2] * deltaPos[2]);
	temp[2] = collisionStiffness * (collisionNormal[2] * collisionNormal[0] * deltaPos[0] + collisionNormal[2] * collisionNormal[1] * deltaPos[1] + collisionNormal[2] * collisionNormal[2] * deltaPos[2]);
	collisionForce[indexX] += temp[0];
	collisionForce[indexY] += temp[1];
	collisionForce[indexZ] += temp[2];


	//����Ħ����
	float friction[3];
	friction[0] = 0.0;
	friction[1] = 0.0;
	friction[2] = 0.0;
	float frictionT[3];	//����Ħ����
	float frictionN[3];	//����Ħ����
						//��������˶����ٶȣ���������ٶ�������
	float v[3];
	v[0] = velocity[indexX] - cylinderV[0];
	v[1] = velocity[indexY] - cylinderV[1];
	v[2] = velocity[indexZ] - cylinderV[2];
	float c[3];
	c[0] = collisionStiffness * collisionNormal[0] * collisionNormal[0];
	c[1] = collisionStiffness * collisionNormal[1] * collisionNormal[1];
	c[2] = collisionStiffness * collisionNormal[2] * collisionNormal[2];
	c[0] += volumnDiag[threadid];
	c[1] += volumnDiag[threadid];
	c[2] += volumnDiag[threadid];
	//b-c(x-st)������໥������
	float relatedForce[3];
	relatedForce[0] = force[indexX] + c[0] * v[0] * 0.0009;
	relatedForce[1] = force[indexY] + c[1] * v[1] * 0.0009;
	relatedForce[2] = force[indexZ] + c[2] * v[2] * 0.0009;


	//�������������Ħ����
	float product = relatedForce[0] * collisionNormal[0]\
		+ relatedForce[1] * collisionNormal[1] \
		+ relatedForce[2] * collisionNormal[2];
	//if (product < 0) {
	//	//���㴹ֱ����
	//	frictionN[0] = -product*collisionNormal[0];
	//	frictionN[1] = -product*collisionNormal[1];
	//	frictionN[2] = -product*collisionNormal[2];

	//	//�����������
	//	relatedForce[0] -= product*collisionNormal[0];
	//	relatedForce[1] -= product*collisionNormal[1];
	//	relatedForce[2] -= product*collisionNormal[2];

	//	//�Ƿ񳬹���Ħ����ֵ
	//	float relatedlength = sqrt(relatedForce[0]* relatedForce[0]+ relatedForce[1]* relatedForce[1]+ relatedForce[2]* relatedForce[2]);
	//	
	//	float frictionNlength = sqrt(frictionN[0]* frictionN[0]+ frictionN[1]* frictionN[1]+ frictionN[2]* frictionN[2]);
	//	if (relatedlength <= frictionNlength*frictionStiffness) {
	//		
	//		frictionT[0] = -relatedForce[0];
	//		frictionT[1] = -relatedForce[1];
	//		frictionT[2] = -relatedForce[2];
	//
	//	}
	//	else {
	//		frictionT[0] = -frictionStiffness*frictionNlength*(relatedForce[0]/relatedlength);
	//		frictionT[1] = -frictionStiffness*frictionNlength*(relatedForce[1]/relatedlength);
	//		frictionT[2] = -frictionStiffness*frictionNlength*(relatedForce[2]/relatedlength);
	//	}

	//	friction[0] = frictionT[0];
	//	friction[1] = frictionT[1];
	//	friction[2] = frictionT[2];
	//}

	//������
	force[indexX] += temp[0] + friction[0];
	force[indexY] += temp[1] + friction[1];
	force[indexZ] += temp[2] + friction[2];
	//force[indexX] += friction[0];
	//force[indexY] += friction[1];
	//force[indexZ] += friction[2];


	//����Խ�Ԫ�ض�Ӧ��ֵ
	collisionDiag[indexX] += collisionStiffness * collisionNormal[0] * collisionNormal[0];
	collisionDiag[indexY] += collisionStiffness * collisionNormal[1] * collisionNormal[1];
	collisionDiag[indexZ] += collisionStiffness * collisionNormal[2] * collisionNormal[2];

	//���ñ�־λ
	isCollide[threadid] = 1;
}

//ר����Բ������ײ��������һ����װ
__device__ bool cylinderCollision(float* pos, float* dir, float* vert, float length, float radius, float* t, float* collisionNormal, float* collisionPos) {
	float cylinder0x, cylinder0y, cylinder0z;
	cylinder0x = pos[0];
	cylinder0y = pos[1];
	cylinder0z = pos[2];
	float cylinder1x, cylinder1y, cylinder1z;
	cylinder1x = pos[0] + dir[0] * length;
	cylinder1y = pos[1] + dir[1] * length;
	cylinder1z = pos[2] + dir[2] * length;

	float cylinderdx = cylinder1x - cylinder0x;
	float cylinderdy = cylinder1y - cylinder0y;
	float cylinderdz = cylinder1z - cylinder0z;
	float dx = vert[0] - cylinder0x;
	float dy = vert[1] - cylinder0y;
	float dz = vert[2] - cylinder0z;
	*t = dir[0] * dx + dir[1] * dy + dir[2] * dz;

	*t /= length;

	if (*t < 0) {
		*t = 0;
	}
	else if (*t > 1) {
		*t = 1;
	}

	dx = vert[0] - cylinder0x - (*t) * cylinderdx;
	dy = vert[1] - cylinder0y - (*t) * cylinderdy;
	dz = vert[2] - cylinder0z - (*t) * cylinderdz;

	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	if (distance > radius) return false;
	if (distance < 1e-5)// ���������������������ײ��Ӧ������ά��ԭ״��
	{
		collisionNormal[0] = 0;
		collisionNormal[1] = 0;
		collisionNormal[2] = 0;
		collisionPos[0] = vert[0];
		collisionPos[1] = vert[1];
		collisionPos[2] = vert[2];
	}
	else
	{
		collisionNormal[0] = dx / distance;
		collisionNormal[1] = dy / distance;
		collisionNormal[2] = dz / distance;
		collisionPos[0] = vert[0] + collisionNormal[0] * (radius - distance);
		collisionPos[1] = vert[1] + collisionNormal[1] * (radius - distance);
		collisionPos[2] = vert[2] + collisionNormal[2] * (radius - distance);
	}

	return true;
}

__device__ bool cylinderCollision_withDepth(float * pose, float* vert, float length, float radius, float* t, float* depth, float* dist, float* collisionNormal, float* collisionPos)
{
	float cylinder0x, cylinder0y, cylinder0z;
	cylinder0x = pose[0];
	cylinder0y = pose[1];
	cylinder0z = pose[2];
	float dir[3] = { pose[3], pose[4], pose[5] };
	float cylinder1x, cylinder1y, cylinder1z;
	cylinder1x = pose[0] + dir[0] * length;
	cylinder1y = pose[1] + dir[1] * length;
	cylinder1z = pose[2] + dir[2] * length;

	float cylinderdx = cylinder1x - cylinder0x;
	float cylinderdy = cylinder1y - cylinder0y;
	float cylinderdz = cylinder1z - cylinder0z;
	float dx = vert[0] - cylinder0x;
	float dy = vert[1] - cylinder0y;
	float dz = vert[2] - cylinder0z;
	*t = dir[0] * dx + dir[1] * dy + dir[2] * dz;

	*t /= length;
	/*printf("tool len: %f\n", length);*/
	if (*t < 0) {
		*t = 0;
		*depth = 0;
		// ע�������return, ��Բ�����������Բ�򡣽�����
		//return false;
	}
	else if (*t > 1) {
		*t = 1;
		*depth = 0;
		// ע�������return, ��Բ�����������Բ�򡣽�����
		//return false;
	}

	dx = vert[0] - cylinder0x - (*t) * cylinderdx;
	dy = vert[1] - cylinder0y - (*t) * cylinderdy;
	dz = vert[2] - cylinder0z - (*t) * cylinderdz;

	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	*dist = distance;

	if (distance > radius) return false;
	if (distance < 1e-5)// ���������������������ײ��Ӧ������ά��ԭ״��
	{
		collisionNormal[0] = 0;
		collisionNormal[1] = 0;
		collisionNormal[2] = 0;
		collisionPos[0] = vert[0];
		collisionPos[1] = vert[1];
		collisionPos[2] = vert[2];
		*depth = 0;
	}
	else
	{
		collisionNormal[0] = dx / distance;
		collisionNormal[1] = dy / distance;
		collisionNormal[2] = dz / distance;
		float d = radius - distance;
		collisionPos[0] = vert[0] + collisionNormal[0] * d;
		collisionPos[1] = vert[1] + collisionNormal[1] * d;
		collisionPos[2] = vert[2] + collisionNormal[2] * d;
		//printf("collided, vert: %f %f %f, toolpos: %f %f %f dir: %f %f %f\nd: %f t%%: %f\n", vert[0], vert[1], vert[2], 
		//	pose[0], pose[1], pose[2], dir[0], dir[1], dir[2], d, *t);
		*depth = d;
	}

	return true;
}


//������Ҫ����ȡ�����������
__global__ void calculateGrabCylinder(float* cylinderPos, float* cylinderDirZ, float* cylinderDirY, float* cylinderDirX, float grappleX, float grappleY, float grappleZ, float* positions, unsigned int* isCollide, unsigned int* isCollideHalf, int vertexNum, float* relativePosition, int* directIndex, int* sphereGrabFlag) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	isCollide[threadid] = 0;
	isCollideHalf[threadid] = 0;
	float collisionNormal[3];
	float collisionPos[3];
	float t = 0.0;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	bool collisionUp = obbCollision(cylinderPos[0], cylinderPos[1], cylinderPos[2], cylinderDirX[0], cylinderDirX[1], cylinderDirX[2], cylinderDirY[0], cylinderDirY[1], cylinderDirY[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], positions[indexX], positions[indexY], positions[indexZ], 1.5 * grappleX, grappleY, grappleZ);
	if (collisionUp) {
		//���ñ�־λ
		isCollide[threadid] = 1;
		//���㶥���ƫ��ֵ
		float vertexPosShift = (positions[indexX] - cylinderPos[0]) * cylinderDirY[0] + (positions[indexY] - cylinderPos[1]) * cylinderDirY[1] + (positions[indexZ] - cylinderPos[2]) * cylinderDirY[2];
		vertexPosShift = abs(vertexPosShift);
		//��¼��ײ��͹��ߵ����λ��
		relativePosition[indexX] = positions[indexX] - cylinderDirY[0] * (vertexPosShift - 0.05) - cylinderPos[0];
		relativePosition[indexY] = positions[indexY] - cylinderDirY[1] * (vertexPosShift - 0.05) - cylinderPos[1];
		relativePosition[indexZ] = positions[indexZ] - cylinderDirY[2] * (vertexPosShift - 0.05) - cylinderPos[2];
	}

	bool collisionDown = obbCollision(cylinderPos[0], cylinderPos[1], cylinderPos[2], cylinderDirX[0], cylinderDirX[1], cylinderDirX[2], -cylinderDirY[0], -cylinderDirY[1], -cylinderDirY[2], -cylinderDirZ[0], -cylinderDirZ[1], -cylinderDirZ[2], positions[indexX], positions[indexY], positions[indexZ], 1.5 * grappleX, grappleY, grappleZ);
	if (collisionDown) {
		isCollide[threadid] = 1;
		float vertexPosShift = (positions[indexX] - cylinderPos[0]) * cylinderDirY[0] + (positions[indexY] - cylinderPos[1]) * cylinderDirY[1] + (positions[indexZ] - cylinderPos[2]) * cylinderDirY[2];
		vertexPosShift = abs(vertexPosShift);
		relativePosition[indexX] = positions[indexX] + cylinderDirY[0] * (vertexPosShift - 0.05) - cylinderPos[0];
		relativePosition[indexY] = positions[indexY] + cylinderDirY[1] * (vertexPosShift - 0.05) - cylinderPos[1];
		relativePosition[indexZ] = positions[indexZ] + cylinderDirY[2] * (vertexPosShift - 0.05) - cylinderPos[2];
	}

	//δ��ײֱ���˳�
	if (isCollide[threadid] != 1) return;


	//����ֲ�����
	float x = relativePosition[indexX] * cylinderDirX[0] + relativePosition[indexY] * cylinderDirX[1] + relativePosition[indexZ] * cylinderDirX[2];
	float y = relativePosition[indexX] * cylinderDirY[0] + relativePosition[indexY] * cylinderDirY[1] + relativePosition[indexZ] * cylinderDirY[2];
	float z = relativePosition[indexX] * cylinderDirZ[0] + relativePosition[indexY] * cylinderDirZ[1] + relativePosition[indexZ] * cylinderDirZ[2];
	//��¼�ֲ�����
	relativePosition[indexX] = x;
	relativePosition[indexY] = y;
	relativePosition[indexZ] = z;

	//�����������˵�ץȡ��
	//1. ��ȡ�ͱ�ץȡ���ӵ������ڵ�
	int sphereIndex = directIndex[threadid];

	//2. ����ץȡ��־��Ϊ1
	sphereGrabFlag[sphereIndex] = 1;

}

//�����ȡ��2.0
__global__ void calculateGrabForce(float* grapperPos, float* grapperDirZ, float* grapperDirY, float* grapperDirX, float grappleX, float grappleY, float grappleZ, float* positions, unsigned int* isCollide, int vertexNum, float adsorbStiffness, float* force, float* collisionDiag, unsigned int grabFlag) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float relativePos[3];
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	if (isCollide[threadid] == 0) {
		bool collisionFlag = obbCollision(grapperPos[0], grapperPos[1], grapperPos[2], grapperDirX[0], grapperDirX[1], grapperDirX[2], grapperDirY[0], grapperDirY[1], grapperDirY[2], grapperDirZ[0], grapperDirZ[1], grapperDirZ[2], positions[indexX], positions[indexY], positions[indexZ], grappleX * 1.5, grappleY, grappleZ);
		if (!collisionFlag) return;
		//���ñ�־λ--���ĸ�ץǯ��ײ
		isCollide[threadid] = grabFlag;
	}

	if (isCollide[threadid] != grabFlag) return;
	//���㶥���ƫ��ֵ
	float vertexPosShift = (positions[indexX] - grapperPos[0]) * grapperDirY[0] + (positions[indexY] - grapperPos[1]) * grapperDirY[1] + (positions[indexZ] - grapperPos[2]) * grapperDirY[2];
	//vertexPosShift = abs(vertexPosShift);
	if (vertexPosShift < 0) vertexPosShift = 0;
	//��¼��ײ��͹��ߵ����λ��
	relativePos[0] = positions[indexX] - grapperDirY[0] * (vertexPosShift - 0.05) - grapperPos[0];
	relativePos[1] = positions[indexY] - grapperDirY[1] * (vertexPosShift - 0.05) - grapperPos[1];
	relativePos[2] = positions[indexZ] - grapperDirY[2] * (vertexPosShift - 0.05) - grapperPos[2];

	//����ֲ�����
	float x = relativePos[0] * grapperDirX[0] + relativePos[1] * grapperDirX[1] + relativePos[2] * grapperDirX[2];
	float y = relativePos[0] * grapperDirY[0] + relativePos[1] * grapperDirY[1] + relativePos[2] * grapperDirY[2];
	float z = relativePos[0] * grapperDirZ[0] + relativePos[1] * grapperDirZ[1] + relativePos[2] * grapperDirZ[2];

	float deltaPos[3];

	//����ƫ������
	float deltax = x * grapperDirX[0] + y * grapperDirY[0] + z * grapperDirZ[0];
	float deltay = x * grapperDirX[1] + y * grapperDirY[1] + z * grapperDirZ[1];
	float deltaz = x * grapperDirX[2] + y * grapperDirY[2] + z * grapperDirZ[2];

	float targetPosx = deltax + grapperPos[0];
	float targetPosy = deltay + grapperPos[1];
	float targetPosz = deltaz + grapperPos[2];

	float distance = calculateCylinderDis(grapperPos[0], grapperPos[1], grapperPos[2], grapperDirZ[0], grapperDirZ[1], grapperDirZ[2], targetPosx, targetPosy, targetPosz, 1.5);
	float k;
	//k = 1.0;
	k = 1 / (1 + exp(12 * distance - 5));
	adsorbStiffness = k * adsorbStiffness;

	deltaPos[0] = targetPosx - positions[indexX];
	deltaPos[1] = targetPosy - positions[indexY];
	deltaPos[2] = targetPosz - positions[indexZ];

	//ÿ�ζ������㣬�����ۼ�
	force[indexX] += adsorbStiffness * deltaPos[0];
	force[indexY] += adsorbStiffness * deltaPos[1];
	force[indexZ] += adsorbStiffness * deltaPos[2];

	collisionDiag[indexX] += adsorbStiffness;
	collisionDiag[indexY] += adsorbStiffness;
	collisionDiag[indexZ] += adsorbStiffness;

}


//���Զ����obb��Χ�н�����ײ��⣨ģ��ץǯץȡ�ķ�Χ��
__device__ bool obbCollision(float posx, float posy, float posz, float dirXx, float dirXy, float dirXz, float dirYx, float dirYy, float dirYz, float dirZx, float dirZy, float dirZz, float vertx, float verty, float vertz, float width, float length, float height) {
	float x = (vertx - posx) * dirXx + (verty - posy) * dirXy + (vertz - posz) * dirXz;
	float y = (vertx - posx) * dirYx + (verty - posy) * dirYy + (vertz - posz) * dirYz;
	float z = (vertx - posx) * dirZx + (verty - posy) * dirZy + (vertz - posz) * dirZz;

	if (z<0 || z>height) return false;
	if (y<0 || y>length) return false;
	if (x<-width || x>width) return false;

	return true;
}

//ʹ��������ײ������������ײ���ж�
__device__ bool cylinderCollisionContinue(
	float length, float moveDistance, float radius,
	float* cylinderPos, float* cylinderLastPos,
	float* cylinderDir,
	float* moveDir, float* position,
	float* t, float* collisionNormal,
	float* collisionPos,
	float* directDir)
{
	float tt, sol;
	float collisionN[3], collisionP[3];
	// ��ͼ��λ�˵�Բ������ײ��⡣
	//bool collision = cylinderCollision(cylinderPos, cylinderDir, position, length, radius, &tt, collisionN, collisionP);
	bool collision = cylinderRayCollisionDetection(cylinderLastPos, cylinderDir,
		position[0], position[1], position[2],
		directDir, // �����ָ���������Ͱ󶨵������ڵ�λ����ء�
		length, radius,
		&tt, &sol,
		collisionN, collisionP);
	float toolMoveVec[3] = { cylinderPos[0] - cylinderLastPos[0],cylinderPos[1] - cylinderLastPos[1], cylinderPos[2] - cylinderLastPos[2] };
	tetNormal_D(toolMoveVec);
	tetNormal_D(directDir);
	float cos_val = tetDot_D(toolMoveVec, directDir);
	if (collision)
	{
		collisionNormal[0] = collisionN[0];
		collisionNormal[1] = collisionN[1];
		collisionNormal[2] = collisionN[2];

		collisionPos[0] = collisionP[0];
		collisionPos[1] = collisionP[1];
		collisionPos[2] = collisionP[2];
		return true;
	}

	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCross_D(cylinderDir, moveDir, normal);
	tetNormal_D(normal);

	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - cylinderPos[0] ,position[1] - cylinderPos[1] ,position[2] - cylinderPos[2] };//���߼��ָ�������ײ�������
	float lineStart0[3] = { cylinderPos[0] ,cylinderPos[1] ,cylinderPos[2] };// ��ǰ���߼�ˣ��������߼�ˣ�
	float lineStart1[3] = { cylinderLastPos[0] ,cylinderLastPos[1] ,cylinderLastPos[2] };// ��һ֡���߼��(��ͼ�ι��߼��)
	float lineStart2[3] = { cylinderPos[0] + cylinderDir[0] * length ,cylinderPos[1] + cylinderDir[1] * length,cylinderPos[2] + cylinderDir[2] * length };// ��ǰ֡����β������������β����


	//����Ҫ�Ƚ���һ����ײ��⣬�������Ƿ�����ײ��������������Ҫ������ײ���


	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	// �����᣺ ���߷���cylinderDir���˶�����moveDir�����߷������˶������ųɵ�ƽ��ķ�����normal
	// ������ײ��������������ɵľֲ�����ϵ����[x, y, z] �ô������������������ϵ�£�x������ײ���ڹ�����ͶӰ��λ�ã�y�������˶������ϵ��˶�����
	// ��˹��Ԫ��[A|I] ֻʹ����֮��ļӼ���A���I��I����A�������
	float x, y, z;
	float det = tetSolveFormula_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float distance = 0.0;
	bool flag = false;
	//2.���������ҵ��������ڵ�����
	if (x > length && y > moveDistance) {
		//����㵽��ľ���
		float basePoint[3] = { cylinderLastPos[0] + length * cylinderDir[0],cylinderLastPos[1] + length * cylinderDir[1] , cylinderLastPos[2] + length * cylinderDir[2] };
		distance = tetPointPointDistance_D(position, basePoint);
		flag = true;
	}
	else if (x > length && y < moveDistance && y>0.0) {
		//����㵽�ߵľ���
		distance = tetPointLineDistance_D(lineStart2, moveDir, position);
	}
	else if (x > length && y < 0.0) {
		distance = tetPointPointDistance_D(position, lineStart2);
	}
	else if (x > 0.0 && x < length && y > moveDistance) {
		distance = tetPointLineDistance_D(lineStart1, cylinderDir, position);
		flag = true;
	}
	else if (x > 0.0 && x < length && y < moveDistance && y>0.0) {
		//����㵽��ľ���
		distance = abs(z);
	}
	else if (x > 0.0 && x < length && y < 0.0) {
		distance = tetPointLineDistance_D(lineStart0, cylinderDir, position);
	}
	else if (x<0.0 && y > moveDistance) {
		distance = tetPointPointDistance_D(position, cylinderLastPos);
		flag = true;
	}
	else if (x < 0.0 && y < moveDistance && y>0.0) {
		distance = tetPointLineDistance_D(lineStart0, moveDir, position);
	}
	else if (x < 0.0 && y < 0.0) {
		distance = tetPointPointDistance_D(position, cylinderPos);
	}


	//3.�жϾ���
	if (distance > radius) return false;
	//if (flag) return false;

	//printf("x:%f,y:%f,z:%f\n", x, y, z);

	//4. �����������ײ�ų�λ��
	//����Ԫһ�η���,�����������Բ�����м���
	float lineDir[3] = { moveDir[0],moveDir[1], moveDir[2] };


	float v0[3] = { position[0] - lineStart0[0] ,position[1] - lineStart0[1] ,position[2] - lineStart0[2] };
	float v1[3] = { position[0] - lineStart1[0] ,position[1] - lineStart1[1] ,position[2] - lineStart1[2] };
	float v2[3] = { position[0] - lineStart2[0] ,position[1] - lineStart2[1] ,position[2] - lineStart2[2] };


	//��Բ���ཻ
	float solve00, solve01;
	float solve10, solve11;
	tetSolveInsect_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
	tetSolveInsect_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
	float solve = min(solve11, solve01);
	//tetSolveInsect_D(lineDir, cylinderDir, v1, radius, &solve00, &solve01);
	//solve = min(solve, solve01);
	//tetSolveInsect_D(lineDir, moveDir, v2, radius, &solve00, &solve01);
	//solve = min(solve, solve01);


	//�����ཻ
	float solve20, solve21;
	tetSolveInsectSphere_D(lineDir, v0, radius, &solve20, &solve21);
	solve = min(solve, solve21);
	//printf("%f\n", solve);
	//tetSolveInsectSphere_D(lineDir, v1, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, v2, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, VSubO, radius, &solve10, &solve11);
	//solve = min(solve, solve11);

	if (solve != solve) return false;
	//printf("x:%f,y:%f,z:%f, solve: %f\n",x,y,z, solve);

	//����λ�õõ������ų���λ��
	collisionPos[0] = position[0] - lineDir[0] * solve;
	collisionPos[1] = position[1] - lineDir[1] * solve;
	collisionPos[2] = position[2] - lineDir[2] * solve;

	//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
	float projPos[3] = { collisionPos[0] - cylinderPos[0],collisionPos[1] - cylinderPos[1],collisionPos[2] - cylinderPos[2] };
	float proj = tetDot_D(projPos, cylinderDir);
	projPos[0] = collisionPos[0] - cylinderPos[0] - cylinderDir[0] * proj;
	projPos[1] = collisionPos[1] - cylinderPos[1] - cylinderDir[1] * proj;
	projPos[2] = collisionPos[2] - cylinderPos[2] - cylinderDir[2] * proj;

	tetNormal_D(projPos);
	collisionNormal[0] = projPos[0];
	collisionNormal[1] = projPos[1];
	collisionNormal[2] = projPos[2];

	return true;
}

__device__ bool cylinderRayCollisionDetection(
	float* cylinderPos, float* cylinderDir,
	float vertx, float verty, float vertz,
	float* moveDir, // ָ����������ײ���*���嶥��*���˶�����
	float length, float radius,
	float* t, float* sln,
	float* collisionNormal, float* collisionPos) {

	float cylinder0x, cylinder0y, cylinder0z; // ���߼��
	cylinder0x = cylinderPos[0];
	cylinder0y = cylinderPos[1];
	cylinder0z = cylinderPos[2];
	float cylinder1x, cylinder1y, cylinder1z; // ����β��
	cylinder1x = cylinderPos[0] + cylinderDir[0] * length;
	cylinder1y = cylinderPos[1] + cylinderDir[1] * length;
	cylinder1z = cylinderPos[2] + cylinderDir[2] * length;

	float cylinderdx = cylinder1x - cylinder0x;
	float cylinderdy = cylinder1y - cylinder0y;
	float cylinderdz = cylinder1z - cylinder0z;
	float dx = vertx - cylinder0x;
	float dy = verty - cylinder0y;
	float dz = vertz - cylinder0z;

	*t = cylinderDir[0] * dx + cylinderDir[1] * dy + cylinderDir[2] * dz;

	*t /= length;

	if (*t < 0) {
		*t = 0;
	}
	else if (*t > 1) {
		*t = 1;
	}

	//�ӹ���ָ��Ӵ���ġ���ֱ�ڹ��ߵ�����
	dx = vertx - cylinder0x - (*t) * cylinderdx;
	dy = verty - cylinder0y - (*t) * cylinderdy;
	dz = vertz - cylinder0z - (*t) * cylinderdz;

	float distance = sqrt(dx * dx + dy * dy + dz * dz);
	if (distance > radius) return false;// �빤��������ľ���

	//������ײ����ͶӰ�Ľ���
	float moveLength = sqrt(moveDir[0] * moveDir[0] + moveDir[1] * moveDir[1] + moveDir[2] * moveDir[2]);
	moveDir[0] /= moveLength;
	moveDir[1] /= moveLength;
	moveDir[2] /= moveLength;

	//���������������moveDir�������ָ�������ķ���directDirection_D���ӵ�ǰ����ָ����󶨵�������ģ�
	// �༴�䱻ѹ�ȵ��˶����򡣶��㳯�˷����˶����Խ��Ͷ���Ƕ�빤�ߵ���ȣ�
	collisionNormal[0] = moveDir[0];
	collisionNormal[1] = moveDir[1];
	collisionNormal[2] = moveDir[2];

	//�����������ϵ���ײ��ͶӰ
	float projectx = cylinder0x + (*t) * cylinderdx;
	float projecty = cylinder0y + (*t) * cylinderdy;
	float projectz = cylinder0z + (*t) * cylinderdz;

	//����local��,���һ��һԪ���η���
	float solution;
	float SN = (vertx - projectx) * (collisionNormal[0]) + (verty - projecty) * (collisionNormal[1]) + (vertz - projectz) * (collisionNormal[2]);
	float SS = (vertx - projectx) * (vertx - projectx) + (verty - projecty) * (verty - projecty) + (vertz - projectz) * (vertz - projectz);
	solution = -SN + sqrt(SN * SN - SS + radius * radius);//ֻȡ����

	if (isnan(solution)) return false;

	//���⴫�ݳ�ȥ �����������������ǣ���������Ϊ�������߷���ͶӰ�����㵽���߱���ľ��롣��sqrt�Ĳ���Ϊ����ʱ��û�н��㣬����nan�����
	*sln = solution;

	collisionPos[0] = vertx + collisionNormal[0] * solution;
	collisionPos[1] = verty + collisionNormal[1] * solution;
	collisionPos[2] = vertz + collisionNormal[2] * solution;

	//�ٴ��������򣬸�Ϊ�ӹ�������ָ��Ӵ����ѹ����
	dx = collisionPos[0] - projectx;
	dy = collisionPos[1] - projecty;
	dz = collisionPos[2] - projectz;
	distance = sqrt(dx * dx + dy * dy + dz * dz);
	collisionNormal[0] = dx / distance;
	collisionNormal[1] = dy / distance;
	collisionNormal[2] = dz / distance;

	return true;
}


__device__ void tetCross_D(float* a, float* b, float* c) {
	//��˼��������η���
	c[0] = a[1] * b[2] - b[1] * a[2];
	c[1] = a[2] * b[0] - b[2] * a[0];
	c[2] = a[0] * b[1] - b[0] * a[1];
}

__device__ float tetDot_D(float* a, float* b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__device__ float tetNormal_D(float* vec0) {
	float length = vec0[0] * vec0[0] + vec0[1] * vec0[1] + vec0[2] * vec0[2];
	length = sqrt(length);
	vec0[0] /= length;
	vec0[1] /= length;
	vec0[2] /= length;
	return length;
}

__device__ float tetSolveFormula_D(float* xAxis, float* yAxis, float* zAxis, float* target, float* x, float* y, float* z) {
	//ʹ�ø�˹��Ԫ
	float a[3][3] = {
		{ xAxis[0], yAxis[0], zAxis[0] },
		{ xAxis[1], yAxis[1], zAxis[1] },
		{ xAxis[2], yAxis[2], zAxis[2] },
	};

	float b[3] = { target[0],target[1],target[2] };

	//��һ����Ԫ
	float k = -a[1][0] / a[0][0];
	a[1][0] += a[0][0] * k;
	a[1][1] += a[0][1] * k;
	a[1][2] += a[0][2] * k;
	b[1] += b[0] * k;

	k = -a[2][0] / a[0][0];
	a[2][0] += a[0][0] * k;
	a[2][1] += a[0][1] * k;
	a[2][2] += a[0][2] * k;
	b[2] += b[0] * k;

	//�ڶ�����Ԫ
	k = -a[2][1] / a[1][1];
	a[2][1] += a[1][1] * k;
	a[2][2] += a[1][2] * k;
	b[2] += b[1] * k;

	//������
	*z = b[2] / a[2][2];
	*y = (b[1] - (*z) * a[1][2]) / a[1][1];
	*x = (b[0] - (*y) * a[0][1] - (*z) * a[0][2]) / a[0][0];

	return 0;
}

__device__ float tetPointLineDistance_D(float* lineStart, float* lineDir, float* point) {
	float v[3] = { point[0] - lineStart[0],point[1] - lineStart[1], point[2] - lineStart[2] };

	float d = tetDot_D(lineDir, v);

	float projPos[3] = { lineStart[0] + d * lineDir[0],lineStart[1] + d * lineDir[1] ,lineStart[2] + d * lineDir[2] };

	projPos[0] = point[0] - projPos[0];
	projPos[1] = point[1] - projPos[1];
	projPos[2] = point[2] - projPos[2];

	return tetNormal_D(projPos);
}

__device__ float tetPointPointDistance_D(float* start, float* end) {
	float x = start[0] - end[0];
	float y = start[1] - end[1];
	float z = start[2] - end[2];
	return sqrt(x * x + y * y + z * z);
}

//���ߺ�Բ����
__device__ void tetSolveInsect_D(float* lineDir, float* toolDir, float* VSubO, float radius, float* solve0, float* solve1) {

	float temp0 = tetDot_D(VSubO, toolDir);
	float temp1 = -tetDot_D(lineDir, toolDir);

	float Bvector[3] = { VSubO[0] - temp0 * toolDir[0], VSubO[1] - temp0 * toolDir[1],  VSubO[2] - temp0 * toolDir[2] };
	float Avector[3] = { -lineDir[0] - temp1 * toolDir[0],-lineDir[1] - temp1 * toolDir[1],-lineDir[2] - temp1 * toolDir[2] };


	float A = tetDot_D(Avector, Avector);
	float B = 2.0 * tetDot_D(Avector, Bvector);
	float C = tetDot_D(Bvector, Bvector) - radius * radius;
	float delta = B * B - 4 * A * C;
	*solve0 = (-sqrt(delta) - B) / (2.0 * A);
	*solve1 = (sqrt(delta) - B) / (2.0 * A);
	if (*solve1 < 0.0) *solve1 = 0.0;
}

//���ߺ�����
__device__ void tetSolveInsectSphere_D(float* lineDir, float* VSubO, float radius, float* solve0, float* solve1) {
	float dir[3] = { -lineDir[0],-lineDir[1] ,-lineDir[2] };

	float distance = tetDot_D(VSubO, VSubO);

	float A = 1.0;
	float B = tetDot_D(dir, VSubO) * 2.0;
	float C = distance - radius * radius;

	float delta = B * B - 4 * A * C;

	*solve0 = (-sqrt(delta) - B) / (2.0 * A);
	*solve1 = (sqrt(delta) - B) / (2.0 * A);
	if (*solve1 < 0.0) *solve1 = 0.0;
}

//ʹ��������ײ������������ײ���ж�
__device__ bool cylinderCollisionContinue_without_directDir(
	float length, float moveDistance, float radius,
	float* cylinderPos, float* cylinderLastPos,
	float* cylinderDir,
	float* moveDir, float* position,
	float* t, float* collisionNormal,
	float* collisionPos)
{
	float tt, sol;
	float collisionN[3], collisionP[3];
	// ��ͼ��λ�˵�Բ������ײ��⡣
	bool collision = cylinderCollision(cylinderPos, cylinderDir, position, length, radius, &tt, collisionN, collisionP);
	//bool collision = cylinderRayCollisionDetection(cylinderLastPos, cylinderDir,
	//	position[0], position[1], position[2],
	//	directDir, // �����ָ���������Ͱ󶨵�����ڵ�λ����ء�
	//	length, radius,
	//	&tt, &sol,
	//	collisionN, collisionP);
	float toolMoveVec[3] = { cylinderPos[0] - cylinderLastPos[0],cylinderPos[1] - cylinderLastPos[1], cylinderPos[2] - cylinderLastPos[2] };
	tetNormal_D(toolMoveVec);
	//tetNormal_D(directDir);
	//float cos_val = tetDot_D(toolMoveVec, directDir);
	if (collision)
	{
		collisionNormal[0] = collisionN[0];
		collisionNormal[1] = collisionN[1];
		collisionNormal[2] = collisionN[2];

		collisionPos[0] = collisionP[0];
		collisionPos[1] = collisionP[1];
		collisionPos[2] = collisionP[2];
		return true;
	}

	//���ȼ�����˶�ƽ��ķ�������
	float normal[3];
	tetCross_D(cylinderDir, moveDir, normal);
	tetNormal_D(normal);

	//���������Ҫ�ı���
	float VSubO[3] = { position[0] - cylinderPos[0] ,position[1] - cylinderPos[1] ,position[2] - cylinderPos[2] };//���߼��ָ�������ײ�������
	float lineStart0[3] = { cylinderPos[0] ,cylinderPos[1] ,cylinderPos[2] };// ��ǰ���߼�ˣ��������߼�ˣ�
	float lineStart1[3] = { cylinderLastPos[0] ,cylinderLastPos[1] ,cylinderLastPos[2] };// ��һ֡���߼��(��ͼ�ι��߼��)
	float lineStart2[3] = { cylinderPos[0] + cylinderDir[0] * length ,cylinderPos[1] + cylinderDir[1] * length,cylinderPos[2] + cylinderDir[2] * length };// ��ǰ֡����β������������β����


	//����Ҫ�Ƚ���һ����ײ��⣬�������Ƿ�����ײ��������������Ҫ������ײ���


	//1.�����ھֲ�����ϵ�е����꣬��������ֲ����겻�������ģ����Բ��ܺ�����е����ʹ�ø�˹��Ԫ
	// �����᣺ ���߷���cylinderDir���˶�����moveDir�����߷������˶������ųɵ�ƽ��ķ�����normal
	// ������ײ��������������ɵľֲ�����ϵ����[x, y, z] �ô������������������ϵ�£�x������ײ���ڹ�����ͶӰ��λ�ã�y�������˶������ϵ��˶�����
	// ��˹��Ԫ��[A|I] ֻʹ����֮��ļӼ���A���I��I����A�������
	float x, y, z;
	float det = tetSolveFormula_D(cylinderDir, moveDir, normal, VSubO, &x, &y, &z);

	if (x != x || y != y || z != z) return false;

	float distance = 0.0;
	bool flag = false;
	//2.���������ҵ��������ڵ�����
	if (x > length && y > moveDistance) {
		//����㵽��ľ���
		float basePoint[3] = { cylinderLastPos[0] + length * cylinderDir[0],cylinderLastPos[1] + length * cylinderDir[1] , cylinderLastPos[2] + length * cylinderDir[2] };
		distance = tetPointPointDistance_D(position, basePoint);
		flag = true;
	}
	else if (x > length && y < moveDistance && y>0.0) {
		//����㵽�ߵľ���
		distance = tetPointLineDistance_D(lineStart2, moveDir, position);
	}
	else if (x > length && y < 0.0) {
		distance = tetPointPointDistance_D(position, lineStart2);
	}
	else if (x > 0.0 && x < length && y > moveDistance) {
		distance = tetPointLineDistance_D(lineStart1, cylinderDir, position);
		flag = true;
	}
	else if (x > 0.0 && x < length && y < moveDistance && y>0.0) {
		//����㵽��ľ���
		distance = abs(z);
	}
	else if (x > 0.0 && x < length && y < 0.0) {
		distance = tetPointLineDistance_D(lineStart0, cylinderDir, position);
	}
	else if (x<0.0 && y > moveDistance) {
		distance = tetPointPointDistance_D(position, cylinderLastPos);
		flag = true;
	}
	else if (x < 0.0 && y < moveDistance && y>0.0) {
		distance = tetPointLineDistance_D(lineStart0, moveDir, position);
	}
	else if (x < 0.0 && y < 0.0) {
		distance = tetPointPointDistance_D(position, cylinderPos);
	}


	//3.�жϾ���
	if (distance > radius) return false;
	//if (flag) return false;

	//printf("x:%f,y:%f,z:%f\n", x, y, z);

	//4. �����������ײ�ų�λ��
	//����Ԫһ�η���,�����������Բ�����м���
	float lineDir[3] = { moveDir[0],moveDir[1], moveDir[2] };


	float v0[3] = { position[0] - lineStart0[0] ,position[1] - lineStart0[1] ,position[2] - lineStart0[2] };
	float v1[3] = { position[0] - lineStart1[0] ,position[1] - lineStart1[1] ,position[2] - lineStart1[2] };
	float v2[3] = { position[0] - lineStart2[0] ,position[1] - lineStart2[1] ,position[2] - lineStart2[2] };


	//��Բ���ཻ
	float solve00, solve01;
	float solve10, solve11;
	tetSolveInsect_D(lineDir, cylinderDir, v0, radius, &solve00, &solve01);
	tetSolveInsect_D(lineDir, moveDir, v0, radius, &solve10, &solve11);
	float solve = min(solve11, solve01);
	//tetSolveInsect_D(lineDir, cylinderDir, v1, radius, &solve00, &solve01);
	//solve = min(solve, solve01);
	//tetSolveInsect_D(lineDir, moveDir, v2, radius, &solve00, &solve01);
	//solve = min(solve, solve01);


	//�����ཻ
	float solve20, solve21;
	tetSolveInsectSphere_D(lineDir, v0, radius, &solve20, &solve21);
	solve = min(solve, solve21);
	//printf("%f\n", solve);
	//tetSolveInsectSphere_D(lineDir, v1, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, v2, radius, &solve10, &solve11);
	//solve = min(solve, solve11);
	//tetSolveInsectSphere_D(lineDir, VSubO, radius, &solve10, &solve11);
	//solve = min(solve, solve11);

	if (solve != solve) return false;
	//printf("x:%f,y:%f,z:%f, solve: %f\n",x,y,z, solve);

	//����λ�õõ������ų���λ��
	collisionPos[0] = position[0] - lineDir[0] * solve;
	collisionPos[1] = position[1] - lineDir[1] * solve;
	collisionPos[2] = position[2] - lineDir[2] * solve;

	//���¶������ײ���ߣ��򹤾������Ͻ���ͶӰ
	float projPos[3] = { collisionPos[0] - cylinderPos[0],collisionPos[1] - cylinderPos[1],collisionPos[2] - cylinderPos[2] };
	float proj = tetDot_D(projPos, cylinderDir);
	projPos[0] = collisionPos[0] - cylinderPos[0] - cylinderDir[0] * proj;
	projPos[1] = collisionPos[1] - cylinderPos[1] - cylinderDir[1] * proj;
	projPos[2] = collisionPos[2] - cylinderPos[2] - cylinderDir[2] * proj;

	tetNormal_D(projPos);
	collisionNormal[0] = projPos[0];
	collisionNormal[1] = projPos[1];
	collisionNormal[2] = projPos[2];

	return true;
}

