#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"


__global__ void calculateVec3Len(float* vec, float* len, int vecNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vecNum) return;

	float x = vec[threadid * 3 + 0];
	float y = vec[threadid * 3 + 1];
	float z = vec[threadid * 3 + 2];
	float vecLen = sqrt(x * x + y * y + z * z);
	//if (vecLen > 0)
	//{
	//	printf("len>0 %f\n", vecLen);
	//}
	len[threadid] = vecLen;
}

//�����ʼ״̬
int runcalculateST(float damping, float dt) {
	//ÿ��block�е��߳���
	int  threadNum = 512;
	//ÿ��grid�е�block��(Ϊ�˱�֤)
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	//���м���
	calculateST << <blockNum, threadNum >> > (tetVertPos_d, tetVertVelocity_d, 
		tetVertExternForce_d, 
		tetVertPos_old_d, tetVertPos_prev_d, tetVertPos_last_d, 
		tetVertFixed_d, 
		tetVertNum_d, gravityX_d, gravityY_d, gravityZ_d, damping, dt);
	hipDeviceSynchronize();//cuda�к˺�����ִ�ж����첽�ģ�������һ����֤�˺�����ȫִ�У����߼���memcpy(cudamemcpy��ͬ����)
	printCudaError("runcalculateST");
	return 0;
}

__global__ void calculateST(float* positions, float* velocity, float* externForce,
	float* old_positions, float* prev_positions, float* last_Positions, float* fixed,
	int vertexNum, float gravityX, float gravityY, float gravityZ, float damping, float dt)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("calculateST vertNum:%d\n", vertexNum);
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;


	last_Positions[indexX] = positions[indexX];
	last_Positions[indexY] = positions[indexY];
	last_Positions[indexZ] = positions[indexZ];

	float fixflag = fixed[threadid] > 1e8 ? 0 : 1;
	//�˶�������
	velocity[indexX] *= damping * fixflag;
	velocity[indexY] *= damping * fixflag;
	velocity[indexZ] *= damping * fixflag;
	//ʩ������
	velocity[indexX] += gravityX * dt * fixflag;
	velocity[indexY] += gravityY * dt * fixflag;
	velocity[indexZ] += gravityZ * dt * fixflag;
		
	//ʩ����������
	velocity[indexX] += externForce[indexX] * dt * fixflag;
	velocity[indexY] += externForce[indexY] * dt * fixflag;
	velocity[indexZ] += externForce[indexZ] * dt * fixflag;

	positions[indexX] += velocity[indexX] * dt * fixflag;
	positions[indexY] += velocity[indexY] * dt * fixflag;
	positions[indexZ] += velocity[indexZ] * dt * fixflag;


	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("%s error: %s\n", "velocity and position", hipGetErrorString(cudaStatus));
	}
	//st
	old_positions[indexX] = positions[indexX];
	old_positions[indexY] = positions[indexY];
	old_positions[indexZ] = positions[indexZ];
	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("%s error: %s\n", "prev and old position", hipGetErrorString(cudaStatus));
	}
	//��������
	externForce[indexX] = 0.0;
	externForce[indexY] = 0.0;
	externForce[indexZ] = 0.0;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("%s error: %s\n", "externForce", hipGetErrorString(cudaStatus));
	}
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
		printf("calculateST p[%f %f %f] v[%f %f %f] extForce[%f %f %f]\n",
			positions[indexX], positions[indexY], positions[indexZ],
			velocity[indexX], velocity[indexY], velocity[indexZ],
			externForce[indexX], externForce[indexY], externForce[indexZ]);
#endif
}

//�����ײ��ǣ�����ײ��ĶԽ�Ԫ��
int runClearCollision() {
	hipMemset(tetVertisCollide_d, 0, tetVertNum_d * sizeof(unsigned char));
	
	hipMemset(tetVertCollisionForce_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	hipMemset(tetVertCollisionDiag_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	hipMemset(tetVertInsertionDepth_d, 0.0f, tetVertNum_d * sizeof(float));

	printCudaError("runClearCollision");
	return 0;
}

int runClearForce()
{
	hipMemset(tetVertForce_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	printCudaError("runClearForce");
	return 0;
}

int runCalculateTetEdgeSpringConstraint()
{
	int threadNum = 512;
	int blockNum = (tetSpringNum_d + threadNum - 1) / threadNum;
	//printf("tetSpringNum_d:%d\n", tetSpringNum_d);
	calculateTetEdgeSpringConstraint << <blockNum, threadNum >> > (
		tetVertPos_d,
		tetVertForce_d,
		tetSpringStiffness_d, tetSpringOrgLen_d, tetSpringIndex_d,
		tetSpringNum_d);
	hipDeviceSynchronize();
	printCudaError("runCalculateTetEdgeSpringConstraint");
	return 0;
}

__global__ void calculateTetEdgeSpringConstraint(
	float* positions, 
	float* force, 
	float* springStiffness, float* springOrigin, int * springIndex, 
	int springNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("in calculateIFMU\n");
	if (threadid >= springNum) return;

	int vIndex0 = springIndex[threadid * 2 + 0];
	int vIndex1 = springIndex[threadid * 2 + 1];


	//printf("threadid:%d, spring index:%d %d\n", threadid, vIndex0, vIndex1);
	//��ȡ�����������local��
	float pos0x = positions[vIndex0 * 3 + 0];
	float pos0y = positions[vIndex0 * 3 + 1];
	float pos0z = positions[vIndex0 * 3 + 2];
	float pos1x = positions[vIndex1 * 3 + 0];
	float pos1y = positions[vIndex1 * 3 + 1];
	float pos1z = positions[vIndex1 * 3 + 2];

	//bool outputFlag = vIndex1 == 93;
	bool outputFlag = false;
	if (outputFlag)
		printf("vIndex0=%d vIndex1=%d pos1[%f %f %f]\n", vIndex0, vIndex1, pos1x, pos1y, pos1z);

#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculateIFMU p0[%f %f %f] p1[%f %f %f]\n",
			pos0x, pos0y, pos0z, pos1x, pos1y, pos1z);
	}
#endif

	//����local��d
	float dx = pos0x - pos1x;
	float dy = pos0y - pos1y;
	float dz = pos0z - pos1z;

	float length = sqrt(dx * dx + dy * dy + dz * dz);
	if (length < springOrigin[threadid]) return;
	dx = dx * (springOrigin[threadid] / length);
	dy = dy * (springOrigin[threadid] / length);
	dz = dz * (springOrigin[threadid] / length);

	if (outputFlag)
	{
		printf("dxyz:[%f %f %f] pos1[%f %f %f] springOrg:%f currentLen:%f\n", dx, dy, dz,
			pos1x, pos1y, pos1z,
			springOrigin[threadid], length);
	}
	//��Ӧ�������˵������
	//����Ӧ����Ҫԭ�Ӳ���
	float tempx = dx - pos0x + pos1x;
	float tempy = dy - pos0y + pos1y;
	float tempz = dz - pos0z + pos1z;

	//if (tempx>0.001)
	//{
	//	float k = springStiffness[threadid];
	//	printf("threadid:%d, vert [%d %d] \n\tpos0[%f %f %f] pos1[%f %f %f]\n\tspring force[%f %f %f] stiffness:%f \n", threadid, vIndex0, vIndex1,
	//		pos0x, pos0y, pos0z, pos1x, pos1y, pos1z,
	//		k * tempx, k * tempy, k * tempz, k);
	//}
	atomicAdd(force + vIndex0 * 3 + 0, tempx * springStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 1, tempy * springStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 2, tempz * springStiffness[threadid]);

	atomicAdd(force + vIndex1 * 3 + 0, -tempx * springStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 1, -tempy * springStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 2, -tempz * springStiffness[threadid]);
}

int runcalculateIF() {

	int  threadNum = 512;
	int blockNum = (tetNum_d + threadNum - 1) / threadNum;
	//���м���
	calculateIF << <blockNum, threadNum >> > (tetVertPos_d, tetIndex_d,
		tetInvD3x3_d, tetInvD3x4_d,
		tetVertForce_d, tetVolume_d, tetActive_d,
		tetNum_d, tetStiffness_d);
	threadNum = 512;
	blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	calculateVec3Len << <blockNum, threadNum >> > (tetVertForce_d, tetVertForceLen_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateIF");
	return 0;
}

///����ÿ�������restposԼ��
int runcalculateRestPos() {
	int  threadNum = 512;
	int blockNum = (tetNum_d + threadNum - 1) / threadNum;
	calculateRestPosStiffness << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d, tetVertPos_d, tetVertisCollide_d, tetVertRestStiffness_d, 1, tetVertNum_d
		);
	calculateRestPos << <blockNum, threadNum >> > (
		tetVertPos_d, tetVertRestPos_d, 
		tetVertCollisionForce_d, tetVertCollisionDiag_d, 
		tetVertRestStiffness_d, tetVertNum_d);

	hipDeviceSynchronize();
	printCudaError("runcalculateRestPos");
	return 0;
 }

__global__ void calculateRestPosStiffness(float* ballPos, unsigned char* toolCollideFlag, float* positions, unsigned char* isCollide, float* reststiffness, int toolNum, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//�����빤�ߵľ��������ײ��Ϣ������restpos�ն�ϵ��
	if (threadid >= vertexNum) return;

	float maxStiffness = 200.0;
	float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	for (int i = 0; i < toolNum; i++)
	{
		float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
		float d = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
		if (d < distance) distance = d;
	}
	if (isCollide[threadid]) {  //�빤�߷�����ײ
		switch (isCollide[threadid])
		{
		case 1: {  //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ���
			reststiffness[threadid] = 0;
			break;
		}
		case 2: { //��ȡ�㣬�͹���ֱ����ײ�Ķ���
			reststiffness[threadid] = 0.0;
			break;
		}
		case 0: { //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
			reststiffness[threadid] = 0.5*maxStiffness*(distance-0.5);
		}
		default:
			break;
		}
	}
	else {   //���û�к͹��߷�����ײ�����ϲ������������Լ����
		reststiffness[threadid] = maxStiffness;
	}
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
		printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}

__global__ void calculateRestPosStiffnessWithMesh_part(
	float* ballPos, float  ballRadius,
	unsigned char* toolCollideFlag, float* positions,
	unsigned char* isCollide, float* meshStiffness,
	int toolNum, int* sortedTetVertIndices, int startIdx, int activeElementNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//�����빤�ߵľ��������ײ��Ϣ������restpos�ն�ϵ��
	if (threadid >= activeElementNum) return;
	int tetVertIdx = sortedTetVertIndices[startIdx + threadid];

	float maxStiffness = 1000;
	bool flag = false;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0)
			flag = true;
	}
	float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
	int indexX = tetVertIdx * 3 + 0;
	int indexY = tetVertIdx * 3 + 1;
	int indexZ = tetVertIdx * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	for (int i = 0; i < toolNum; i++)
	{
		float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
		float d = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
		if (d < distance) distance = d;
	}
	if (flag) {  //�빤�߷�����ײ
		switch (isCollide[tetVertIdx])
		{
		case 1: {  //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ���
			meshStiffness[tetVertIdx] = maxStiffness;
#ifdef OUTPUT_INFO
			if (tetVertIdx == LOOK_THREAD)
				printf("tet reststiffness with mesh in thread:%d: %f\n", meshStiffness[LOOK_THREAD]);
#endif
			break;
		}
		case 2: { //��ȡ�㣬�͹���ֱ����ײ�Ķ���
			meshStiffness[tetVertIdx] = 0.0;
			break;
		}
		case 0: { //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
			//float x = distance - 2*ballRadius;
			//float influence_r = 3*ballRadius;
			//if (x > influence_r)
			//	x = influence_r;
			//else if (x < 0)
			//	x = 0;
			//float t = 1 - x / influence_r;
			//meshStiffness[tetVertIdx] = t*t*t * maxStiffness;

			float k = 1 / (1 + exp(10 * distance - 5));
			meshStiffness[threadid] = k * maxStiffness;
			break;
		}
		default:
			break;
		}
	}
	else {   //���û�к͹��߷�����ײ�����ϲ������������Լ����
		meshStiffness[tetVertIdx] = 0.0;
	}
	//meshStiffness[tetVertIdx] = 0;
#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
		//printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}

__global__ void calculateRestPosStiffnessWithMesh(float* ballPos, unsigned char* toolCollideFlag, float* positions, unsigned char* isCollide, float* meshStiffness, int toolNum, int vertexNum) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
//�����빤�ߵľ��������ײ��Ϣ������restpos�ն�ϵ��
	if (threadid >= vertexNum) return;

	float maxStiffness = 50000;
	bool flag = false;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0)
			flag = true;
	}
	float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	for (int i = 0; i < toolNum; i++)
	{
		float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
		float d = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
		if (d < distance) distance = d;
	}
	if (flag) {  //�빤�߷�����ײ
		switch (isCollide[threadid])
		{
		case 1: {  //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ���
			meshStiffness[threadid] = maxStiffness;				
//ifdef OUTPUT_INFO
			if (threadid == LOOK_THREAD)
				printf("tet reststiffness with mesh in thread:%d: %f\n", meshStiffness[LOOK_THREAD]);
#//endif
			break;
		}
		case 2: { //��ȡ�㣬�͹���ֱ����ײ�Ķ���
			meshStiffness[threadid] = 0.0;
			break;
		}
		case 0: { //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
			float k = 1 / (1 + exp(10 * distance - 5));
			meshStiffness[threadid] = k * maxStiffness;
		}
		default:
			break;
		}
	}
	else {   //���û�к͹��߷�����ײ�����ϲ������������Լ����
		meshStiffness[threadid] = 0.0;
	}
#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
		//printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}

__global__ void calculateRestPos(float* positions, float* rest_positions, float* force, float* collisionDiag, float* restStiffness, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid > vertexNum) return;

	//��������
	float tempx = rest_positions[3 * threadid + 0] - positions[3 * threadid + 0];
	float tempy = rest_positions[3 * threadid + 1] - positions[3 * threadid + 1];
	float tempz = rest_positions[3 * threadid + 2] - positions[3 * threadid + 2];

	atomicAdd(force + threadid * 3 + 0, tempx * restStiffness[threadid]);
	atomicAdd(force + threadid * 3 + 1, tempy * restStiffness[threadid]);
	atomicAdd(force + threadid * 3 + 2, tempz * restStiffness[threadid]);

	atomicAdd(collisionDiag + threadid * 3 + 0, restStiffness[threadid]);
	atomicAdd(collisionDiag + threadid * 3 + 1, restStiffness[threadid]);
	atomicAdd(collisionDiag + threadid * 3 + 2, restStiffness[threadid]);
}

__global__ void calculateRestPos_part(float* positions, float* rest_positions, float* force, float* collisionDiag, float* restStiffness, 
	int* sortedTetVertIndices, int offset, int activeElement)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid > activeElement) return;

	int tetVertIdx = sortedTetVertIndices[threadid+offset];

	//��������
	float tempx = rest_positions[3 * tetVertIdx + 0] - positions[3 * tetVertIdx + 0];
	float tempy = rest_positions[3 * tetVertIdx + 1] - positions[3 * tetVertIdx + 1];
	float tempz = rest_positions[3 * tetVertIdx + 2] - positions[3 * tetVertIdx + 2];

	atomicAdd(force + tetVertIdx * 3 + 0, tempx * restStiffness[tetVertIdx]);
	atomicAdd(force + tetVertIdx * 3 + 1, tempy * restStiffness[tetVertIdx]);
	atomicAdd(force + tetVertIdx * 3 + 2, tempz * restStiffness[tetVertIdx]);

	atomicAdd(collisionDiag + tetVertIdx * 3 + 0, restStiffness[tetVertIdx]);
	atomicAdd(collisionDiag + tetVertIdx * 3 + 1, restStiffness[tetVertIdx]);
	atomicAdd(collisionDiag + tetVertIdx * 3 + 2, restStiffness[tetVertIdx]);
}

__device__ void MatrixSubstract_3_D(float* A, float* B, float* R)						//R=A-B
{
	for (int i = 0; i < 9; i++)	R[i] = A[i] - B[i];
}
__device__ void MatrixProduct_3_D(const float* A, const float* B, float* R)				//R=A*B
{
	R[0] = A[0] * B[0] + A[1] * B[3] + A[2] * B[6];
	R[1] = A[0] * B[1] + A[1] * B[4] + A[2] * B[7];
	R[2] = A[0] * B[2] + A[1] * B[5] + A[2] * B[8];
	R[3] = A[3] * B[0] + A[4] * B[3] + A[5] * B[6];
	R[4] = A[3] * B[1] + A[4] * B[4] + A[5] * B[7];
	R[5] = A[3] * B[2] + A[4] * B[5] + A[5] * B[8];
	R[6] = A[6] * B[0] + A[7] * B[3] + A[8] * B[6];
	R[7] = A[6] * B[1] + A[7] * B[4] + A[8] * B[7];
	R[8] = A[6] * B[2] + A[7] * B[5] + A[8] * B[8];
}
__device__ void MatrixProduct_D(float* A, float* B, float* R, int nx, int ny, int nz)	//R=A*B
{
	memset(R, 0, sizeof(float) * nx * nz);
	for (int i = 0; i < nx; i++)
		for (int j = 0; j < nz; j++)
			for (int k = 0; k < ny; k++)
				R[i * nz + j] += A[i * ny + k] * B[k * nz + j];
}

__global__ void calculateIF(float* positions, int* tetIndex,
	float* tetInvD3x3, float* tetInvD3x4,
	float* force, float* tetVolumn, bool* active,
	int tetNum, float* volumnStiffness) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= tetNum) return;

	if (!active[threadid]) {
		return;
	}

	//��ȡ��ǰ������ı���ϵ��
	//volumnStiffness = tetStiffness_d[threadid];

	//����ÿ���������ʼ����shape�������
	int vIndex0 = tetIndex[threadid * 4 + 0];
	int vIndex1 = tetIndex[threadid * 4 + 1];
	int vIndex2 = tetIndex[threadid * 4 + 2];
	int vIndex3 = tetIndex[threadid * 4 + 3];

	//�ȼ���shape����
	float D[9];
	D[0] = positions[vIndex1 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[1] = positions[vIndex2 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[2] = positions[vIndex3 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[3] = positions[vIndex1 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[4] = positions[vIndex2 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[5] = positions[vIndex3 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[6] = positions[vIndex1 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[7] = positions[vIndex2 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[8] = positions[vIndex3 * 3 + 2] - positions[vIndex0 * 3 + 2];

	//�����α��ݶ�F
	float F[9];
	float* B = &tetInvD3x3[threadid * 9];
	MatrixProduct_3_D(D, &tetInvD3x3[threadid * 9], F);

	//��F�зֽ��R��ֱ�Ӱ��ˣ�����㷨̫�����ˣ�
	float R[9];
	GetRotation_D((float(*)[3])F, (float(*)[3])R);//ת��Ϊ����ָ�룬����Ӧ��ά������β�Ҫ��

	MatrixSubstract_3_D(R, F, R);
	//for (int i = 0; i < 9; i++)	
	//	R[i] = R[i] - F[i];

	float temp[12];
	memset(temp, 0, sizeof(float) *12);
	MatrixProduct_D(R, &tetInvD3x4[threadid * 12], temp, 3, 3, 4);

	//��Ӧ���ĸ����xyz����
	//����Ӧ����Ҫԭ�Ӳ���
	atomicAdd(force + vIndex0 * 3 + 0, temp[0] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 1, temp[4] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex0 * 3 + 2, temp[8] * tetVolumn[threadid] * volumnStiffness[threadid]);

	atomicAdd(force + vIndex1 * 3 + 0, temp[1] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 1, temp[5] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex1 * 3 + 2, temp[9] * tetVolumn[threadid] * volumnStiffness[threadid]);

	atomicAdd(force + vIndex2 * 3 + 0, temp[2] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex2 * 3 + 1, temp[6] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex2 * 3 + 2, temp[10] * tetVolumn[threadid] * volumnStiffness[threadid]);

	atomicAdd(force + vIndex3 * 3 + 0, temp[3] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex3 * 3 + 1, temp[7] * tetVolumn[threadid] * volumnStiffness[threadid]);
	atomicAdd(force + vIndex3 * 3 + 2, temp[11] * tetVolumn[threadid] * volumnStiffness[threadid]);
#ifdef OUTPUT_INFO
	if (vIndex0 == LOOK_THREAD)
		printf("calculateIF tetVertForce_d in calculateIF: %f %f %f\n", force[vIndex0 * 3 + 0], force[vIndex0 * 3 + 1], force[vIndex0 * 3 + 2]);
	//if (vIndex0 == 0)
	//{
	//	printf("calculateIF threadid: %d v0_temp[%f %f %f]\n",
	//		threadid, temp[0], temp[4], temp[8]);
	//	if (isnan(temp[0]) || isnan(temp[4]) || isnan(temp[8]))
	//	{
	//		unsigned int t = threadid * 12;
	//		printf("threadid: %d\n nan occured in v0, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			threadid,
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//		printf("threadid: %d\ntetInvD3x4[ %f %f %f %f \n\t%f %f %f %f\n\t %f %f %f %f]\n",
	//			threadid,
	//			tetInvD3x4[t + 0], tetInvD3x4[t + 1], tetInvD3x4[t + 2], tetInvD3x4[t + 3],
	//			tetInvD3x4[t + 4], tetInvD3x4[t + 5], tetInvD3x4[t + 6], tetInvD3x4[t + 7],
	//			tetInvD3x4[t + 8], tetInvD3x4[t + 9], tetInvD3x4[t + 10], tetInvD3x4[t + 11]);
	//	}
	//}
	//if (vIndex1 == 0)
	//{
	//	printf("calculateIF v1_temp[%f %f %f]\n",
	//		temp[1], temp[5], temp[9]);
	//	if (isnan(temp[1]) || isnan(temp[5]) || isnan(temp[9]))
	//	{
	//		printf("nan occured in v1, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex2 == 0)
	//{
	//	printf("calculateIF v2_temp[%f %f %f]\n",
	//		temp[2], temp[6], temp[10]);
	//	if (isnan(temp[2]) || isnan(temp[6]) || isnan(temp[10]))
	//	{
	//		printf("nan occured in v2, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex3 == 0)
	//{
	//	printf("calculateIF v3_temp[%f %f %f]\n",
	//		temp[3], temp[7], temp[11]);
	//	if (isnan(temp[3]) || isnan(temp[7]) || isnan(temp[11]))
	//	{
	//		printf("nan occured in v3, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
#endif
}

__global__ void calculateIF_part(float* positions, int* tetIndex,
	float* tetInvD3x3, float* tetInvD3x4,
	float* force, float* tetVolumn, float* volumnStiffness, 
	int * sortedTetIdx, int offset, int activeElementNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	//��ȡ��ǰ������ı���ϵ��
	//volumnStiffness = tetStiffness_d[threadid];
#ifdef OUTPUT_INFO
	if (threadid == 0)
		printf("calculateIF startIdx:%d, activeTetNum:%d\n", threadid + offset, activeElementNum);
#endif
	unsigned int tetIdx = sortedTetIdx[threadid+offset];
	//����ÿ���������ʼ����shape�������
	int vIndex0 = tetIndex[tetIdx * 4 + 0];
	int vIndex1 = tetIndex[tetIdx * 4 + 1];
	int vIndex2 = tetIndex[tetIdx * 4 + 2];
	int vIndex3 = tetIndex[tetIdx * 4 + 3];

	//�ȼ���shape����
	float D[9];
	D[0] = positions[vIndex1 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[1] = positions[vIndex2 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[2] = positions[vIndex3 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[3] = positions[vIndex1 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[4] = positions[vIndex2 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[5] = positions[vIndex3 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[6] = positions[vIndex1 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[7] = positions[vIndex2 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[8] = positions[vIndex3 * 3 + 2] - positions[vIndex0 * 3 + 2];

	//�����α��ݶ�F
	float F[9];
	float* B = &tetInvD3x3[tetIdx * 9];
	MatrixProduct_3_D(D, &tetInvD3x3[tetIdx * 9], F);

	//��F�зֽ��R��ֱ�Ӱ��ˣ�����㷨̫�����ˣ�
	float R[9];
	GetRotation_D((float(*)[3])F, (float(*)[3])R);//ת��Ϊ����ָ�룬����Ӧ��ά������β�Ҫ��

	MatrixSubstract_3_D(R, F, R);
	//for (int i = 0; i < 9; i++)	
	//	R[i] = R[i] - F[i];

	float temp[12];
	memset(temp, 0, sizeof(float) * 12);
	MatrixProduct_D(R, &tetInvD3x4[tetIdx * 12], temp, 3, 3, 4);

	//��Ӧ���ĸ����xyz����
	//����Ӧ����Ҫԭ�Ӳ���
	atomicAdd(force + vIndex0 * 3 + 0, temp[0] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex0 * 3 + 1, temp[4] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex0 * 3 + 2, temp[8] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex1 * 3 + 0, temp[1] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex1 * 3 + 1, temp[5] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex1 * 3 + 2, temp[9] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex2 * 3 + 0, temp[2] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex2 * 3 + 1, temp[6] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex2 * 3 + 2, temp[10] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex3 * 3 + 0, temp[3] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex3 * 3 + 1, temp[7] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex3 * 3 + 2, temp[11] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
#ifdef OUTPUT_INFO
	if (vIndex0 == LOOK_THREAD)
		printf("calculateIF tetVertForce_d in calculateIF: %f %f %f\n", force[vIndex0 * 3 + 0], force[vIndex0 * 3 + 1], force[vIndex0 * 3 + 2]);
	//if (vIndex0 == 0)
	//{
	//	printf("calculateIF threadid: %d v0_temp[%f %f %f]\n",
	//		threadid, temp[0], temp[4], temp[8]);
	//	if (isnan(temp[0]) || isnan(temp[4]) || isnan(temp[8]))
	//	{
	//		unsigned int t = threadid * 12;
	//		printf("threadid: %d\n nan occured in v0, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			threadid,
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//		printf("threadid: %d\ntetInvD3x4[ %f %f %f %f \n\t%f %f %f %f\n\t %f %f %f %f]\n",
	//			threadid,
	//			tetInvD3x4[t + 0], tetInvD3x4[t + 1], tetInvD3x4[t + 2], tetInvD3x4[t + 3],
	//			tetInvD3x4[t + 4], tetInvD3x4[t + 5], tetInvD3x4[t + 6], tetInvD3x4[t + 7],
	//			tetInvD3x4[t + 8], tetInvD3x4[t + 9], tetInvD3x4[t + 10], tetInvD3x4[t + 11]);
	//	}
	//}
	//if (vIndex1 == 0)
	//{
	//	printf("calculateIF v1_temp[%f %f %f]\n",
	//		temp[1], temp[5], temp[9]);
	//	if (isnan(temp[1]) || isnan(temp[5]) || isnan(temp[9]))
	//	{
	//		printf("nan occured in v1, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex2 == 0)
	//{
	//	printf("calculateIF v2_temp[%f %f %f]\n",
	//		temp[2], temp[6], temp[10]);
	//	if (isnan(temp[2]) || isnan(temp[6]) || isnan(temp[10]))
	//	{
	//		printf("nan occured in v2, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex3 == 0)
	//{
	//	printf("calculateIF v3_temp[%f %f %f]\n",
	//		temp[3], temp[7], temp[11]);
	//	if (isnan(temp[3]) || isnan(temp[7]) || isnan(temp[11]))
	//	{
	//		printf("nan occured in v3, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
#endif
}
int runcalculateRestPosForceWithMeshPos(float toolRadius)
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	calculateRestPosStiffnessWithMesh << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d,
		tetVertPos_d, tetVertisCollide_d, 
		tetVertfromTriStiffness_d, cylinderNum_d, tetVertNum_d);

	calculateRestPos << <blockNum, threadNum >> > (
		tetVertPos_d, tetVertRestPos_d,
		tetVertCollisionForce_d, tetVertCollisionDiag_d,
		tetVertfromTriStiffness_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateRestPosForceWithMeshPos");
	return 0;
}

__global__ void calculateRestPosForceWithMeshPos(
	float* positions, int* skeletonMesh,
	float* force, float* collisionDiag,
	float* meshPositions, unsigned char* isCollide,
	float* meshStiffness, int vertexNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int tri_Idx = skeletonMesh[threadid];
	if (tri_Idx == -1) return; // û���������嶥��󶨵ı��沼�ϵ㲻������㡣
	if (isCollide[threadid] == 0) return;

	float deltaPos[3];
	deltaPos[0] = meshPositions[3 * tri_Idx + 0] - positions[3 * threadid + 0];
	deltaPos[1] = meshPositions[3 * tri_Idx + 1] - positions[3 * threadid + 1];
	deltaPos[2] = meshPositions[3 * tri_Idx + 2] - positions[3 * threadid + 2];
	float d = sqrt(deltaPos[0] * deltaPos[0] + deltaPos[1] * deltaPos[1] + deltaPos[2] * deltaPos[2]);
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculateRestPosForceWithMeshPos thread:%d deltaPos[%f %f %f]\n", threadid, deltaPos[0], deltaPos[1], deltaPos[2]);
	}
#endif
	if (d < 1e-9)
		return;
	float dir[3] = { deltaPos[0] / d, deltaPos[1] / d,deltaPos[2] / d };
	

	float forcex = deltaPos[0] * meshStiffness[threadid];
	float forcey = deltaPos[1] * meshStiffness[threadid];
	float forcez = deltaPos[2] * meshStiffness[threadid];
	force[threadid * 3 + 0] += forcex;
	force[threadid * 3 + 1] += forcey;
	force[threadid * 3 + 2] += forcez;

	collisionDiag[threadid * 3 + 0] += meshStiffness[threadid];
	collisionDiag[threadid * 3 + 1] += meshStiffness[threadid];
	collisionDiag[threadid * 3 + 2] += meshStiffness[threadid];

}

//����position
__global__ void calculatePOS(float* positions, float* force, float* fixed, float* mass,
	float* next_positions, float* prev_positions, float* old_positions,
	float* volumnDiag, float* collisionDiag, float* collisionForce,
	int vertexNum, float dt, float omega)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	float diagConstant = (mass[threadid] + fixed[threadid]) / (dt * dt);
	float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);

	//����ÿ�����shape match������Լ�����֣���Ϊ֮ǰ�ǰ���ÿ�����������ģ�����Ҫ̯��ÿ��������
	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS force[%f,%f,%f] collisionForce[%f,%f,%f]\n",
	//		force[indexX], force[indexY], force[indexZ],
	//		collisionForce[indexX], collisionForce[indexY], collisionForce[indexZ]);
	//}
#ifdef OUTPUT_INFO


	if (threadid == LOOK_THREAD)
	{
		printf("calculatePOS constantDiag:%f volumeDiag:%f collisionDiag:[%f, %f %f]\n", diagConstant, volumnDiag[threadid], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	}
	//if (collisionDiag[indexX] > 0)
	//{
	//	printf("threadid:%d collisionDiag[%f %f %f]\n", threadid, collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
#endif
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (volumnDiag[threadid] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (volumnDiag[threadid] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (volumnDiag[threadid] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];


	//if (threadid== 6000) {
	//	printf("*********************\n");
	//	printf("%d:����Խ�Ԫ��:%f,%f,%f\n", threadid, elementX, elementY, elementZ);
	//}
	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	// omega���壺omega = 4 / (4 - rho*rho*omega);
	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];

	float deltax = positions[indexX] - prev_positions[indexX];
	float deltay = positions[indexY] - prev_positions[indexY];
	float deltaz = positions[indexZ] - prev_positions[indexZ];

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("point delta x:%f %f %f\n", deltax, deltay, deltaz);
	//}
	//if (isnan(positions[indexX]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}
	//if (isnan(positions[indexZ]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}
	//if (isnan(positions[indexY]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}

	if (forceLen > 2)
	{
		float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
		//printf("%d-tetVertForce_d in calculatePOS:%f %f %f\nmovement:%f constantDiag:%f\n", threadid, force[indexX], force[indexY], force[indexZ], movement, diagConstant);

	}

	//float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
	//if(movement>1e-5)
	//	printf("thread %d movement: %f\n", threadid, movement);
}

//�������λ��
int runcalculatePOS(float omega, float dt) {
	int  threadNum = 512;

	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	//���м���
	calculatePOS << <blockNum, threadNum >> > (tetVertPos_d, tetVertForce_d,
		tetVertFixed_d, tetVertMass_d,
		tetVertPos_next_d, tetVertPos_prev_d, tetVertPos_old_d,
		tetVolumeDiag_d, tetVertCollisionDiag_d, tetVertCollisionForce_d,
		tetVertNum_d, dt, omega);
	hipDeviceSynchronize();
	printCudaError("runcalculatePOS");
	return 0;
}

//����position
__global__ void calculatePOS(float* positions, float* force, float* fixed, float* mass,
	float* next_positions, float* prev_positions, float* old_positions,
	float* volumnDiag, float* collisionDiag, float* collisionForce,
	int* sortedIndices, int offset, int activeElementNum, float dt, float omega)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;
	int vertIdx = sortedIndices[offset + threadid];

	if (vertIdx == GRABED_TETIDX)
		return;

	int indexX = vertIdx * 3 + 0;
	int indexY = vertIdx * 3 + 1;
	int indexZ = vertIdx * 3 + 2;

	float diagConstant = (mass[vertIdx] + fixed[vertIdx]) / (dt * dt);
	float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);

	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];

//#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS force[%f,%f,%f] collisionForce[%f,%f,%f]\n",
	//		force[indexX], force[indexY], force[indexZ],
	//		collisionForce[indexX], collisionForce[indexY], collisionForce[indexZ]);
	//}

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS mass:%f constantDiag:%f volumeDiag:%f collisionDiag:[%f, %f %f]\n", mass[vertIdx], diagConstant, volumnDiag[vertIdx], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
	//if (collisionDiag[indexX] > 0)
	//{
	//	printf("vertIdx:%d collisionDiag[%f %f %f]\n", vertIdx, collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
//#endif
// 
// 
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (volumnDiag[vertIdx] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (volumnDiag[vertIdx] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (volumnDiag[vertIdx] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];


	//if (vertIdx== 6000) {
	//	printf("*********************\n");
	//	printf("%d:����Խ�Ԫ��:%f,%f,%f\n", vertIdx, elementX, elementY, elementZ);
	//}
	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	// omega���壺omega = 4 / (4 - rho*rho*omega);
	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];

	float deltax = positions[indexX] - prev_positions[indexX];
	float deltay = positions[indexY] - prev_positions[indexY];
	float deltaz = positions[indexZ] - prev_positions[indexZ];

	//if (isnan(positions[indexX]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}
	//if (isnan(positions[indexZ]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}
	//if (isnan(positions[indexY]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}

	if (forceLen > 2)
	{
		float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
		//printf("%d-tetVertForce_d in calculatePOS:%f %f %f\nmovement:%f constantDiag:%f\n", vertIdx, force[indexX], force[indexY], force[indexZ], movement, diagConstant);

	}

	//float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
	//if(movement>1e-5)
	//	printf("thread %d movement: %f\n", vertIdx, movement);
}

int runcalculateV(float dt) {
	int  threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	//���м���
	calculateV << <blockNum, threadNum >> > (tetVertPos_d, tetVertVelocity_d, tetVertPos_last_d, tetVertNum_d, dt);

	hipDeviceSynchronize();
	printCudaError("runcalculateV");
	return 0;

}

//�����ٶȸ���
__global__ void calculateV(float* positions, float* velocity, float* last_positions, int vertexNum, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	velocity[threadid * 3 + 0] = (positions[threadid * 3 + 0] - last_positions[threadid * 3 + 0]) / dt;
	velocity[threadid * 3 + 1] = (positions[threadid * 3 + 1] - last_positions[threadid * 3 + 1]) / dt;
	velocity[threadid * 3 + 2] = (positions[threadid * 3 + 2] - last_positions[threadid * 3 + 2]) / dt;
}

__global__ void calculateV(float* positions, float* velocity, float* last_positions, int* sortedIndices, int offset, int activeElementNum, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[threadid + offset];
	velocity[vertIdx * 3 + 0] = (positions[vertIdx * 3 + 0] - last_positions[vertIdx * 3 + 0]) / dt;
	velocity[vertIdx * 3 + 1] = (positions[vertIdx * 3 + 1] - last_positions[vertIdx * 3 + 1]) / dt;
	velocity[vertIdx * 3 + 2] = (positions[vertIdx * 3 + 2] - last_positions[vertIdx * 3 + 2]) / dt;
}
int runUpdateInnerTetVertDDir()
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	updateInnerTetVertDirectDirection << <blockNum, threadNum >> > (tetVertPos_d,
		tetVertBindingTetVertIndices_d, tetVertBindingTetVertWeight_d,
		tetVertNonPenetrationDir_d, tetVertNum_d);
	// no need to sychronize
	printCudaError("updateInnerTetVertDDir");
	return 0;
}
int runUpdateSurfaceTetVertDDir()
{
	int threadNum = 512;
	int blockNum = (triVertOrgNum_d + threadNum - 1) / threadNum;
	updateSurfaceTetVertDirectDirection << <blockNum, threadNum >> > (
		onSurfaceTetVertIndices_d,
		tetVert2TriVertMapping_d, triVertNorm_d,
		tetVertNonPenetrationDir_d,
		tetVertPos_d, triVertPos_d,
		triVertOrgNum_d);
	printCudaError("updateSurfaceTetVertDDir");
	return 0;
}
int runNormalizeDDir()
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	normalizeDDir << <blockNum, threadNum >> > (tetVertNonPenetrationDir_d, tetVertNum_d);
	printCudaError("NormalizeTetVertDDir");
	return 0;
}
int runUpdateTetVertDirectDirection()
{
	runUpdateInnerTetVertDDir();
	runUpdateSurfaceTetVertDDir();
	hipDeviceSynchronize();

	runNormalizeDDir();
	hipDeviceSynchronize();
	printCudaError("UpdateTetVertDirectDirection");
	return 0;
}

__global__ void normalizeDDir(float* dDir, int pointNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= pointNum) return;
	
	int idxX = threadid * 3 + 0;
	int idxY = threadid * 3 + 1;
	int idxZ = threadid * 3 + 2;
	float l = sqrt(dDir[idxX] * dDir[idxX] + dDir[idxY] * dDir[idxY] + dDir[idxZ] * dDir[idxZ]);
	if(l<1e-7)
	{
		//printf("threadid %d, dDirLen=0\n", threadid);
		dDir[idxX] = 1;
		dDir[idxY] = 0;
		dDir[idxZ] = 0;
	}
	else
	{
		dDir[idxX] /= l;
		dDir[idxY] /= l;
		dDir[idxZ] /= l;
	}
	//if (threadid < 10)
	//{
	//	printf("threadid %d, DDir [%f %f %f]\n", threadid, dDir[idxX], dDir[idxY], dDir[idxZ]);
	//}
}
__device__ void GetRotation_D(float F[3][3], float R[3][3])
{
	float C[3][3];
	memset(&C[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C[i][j] += F[k][i] * F[k][j];

	float C2[3][3];
	memset(&C2[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C2[i][j] += C[i][k] * C[j][k];

	float det = F[0][0] * F[1][1] * F[2][2] +
		F[0][1] * F[1][2] * F[2][0] +
		F[1][0] * F[2][1] * F[0][2] -
		F[0][2] * F[1][1] * F[2][0] -
		F[0][1] * F[1][0] * F[2][2] -
		F[0][0] * F[1][2] * F[2][1];

	float I_c = C[0][0] + C[1][1] + C[2][2];
	float I_c2 = I_c * I_c;
	float II_c = 0.5 * (I_c2 - C2[0][0] - C2[1][1] - C2[2][2]);
	float III_c = det * det;
	float k = I_c2 - 3 * II_c;

	float inv_U[3][3];
	if (k < 1e-10f)
	{
		float inv_lambda = 1 / sqrt(I_c / 3);
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = inv_lambda;
		inv_U[1][1] = inv_lambda;
		inv_U[2][2] = inv_lambda;
	}
	else
	{
		float l = I_c * (I_c * I_c - 4.5 * II_c) + 13.5 * III_c;
		float k_root = sqrt(k);
		float value = l / (k * k_root);
		if (value < -1.0) value = -1.0;
		if (value > 1.0) value = 1.0;
		float phi = acos(value);
		float lambda2 = (I_c + 2 * k_root * cos(phi / 3)) / 3.0;
		float lambda = sqrt(lambda2);

		float III_u = sqrt(III_c);
		if (det < 0)   III_u = -III_u;
		float I_u = lambda + sqrt(-lambda2 + I_c + 2 * III_u / lambda);
		float II_u = (I_u * I_u - I_c) * 0.5;

		float U[3][3];
		float inv_rate, factor;

		inv_rate = 1 / (I_u * II_u - III_u);
		factor = I_u * III_u * inv_rate;

		memset(U, 0, sizeof(float) * 9);
		U[0][0] = factor;
		U[1][1] = factor;
		U[2][2] = factor;

		factor = (I_u * I_u - II_u) * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				U[i][j] += factor * C[i][j] - inv_rate * C2[i][j];

		inv_rate = 1 / III_u;
		factor = II_u * inv_rate;
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = factor;
		inv_U[1][1] = factor;
		inv_U[2][2] = factor;





		factor = -I_u * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				inv_U[i][j] += factor * U[i][j] + inv_rate * C[i][j];
	}




	memset(&R[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				R[i][j] += F[i][k] * inv_U[k][j];

	//��飬����invert
	if (det <= 0) {
		R[0][0] = 1;
		R[0][1] = 0;
		R[0][2] = 0;
		R[1][0] = 0;
		R[1][1] = 1;
		R[1][2] = 0;
		R[2][0] = 0;
		R[2][1] = 0;
		R[2][2] = 1;
	}
}
